#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <thrust/device_vector.h>
#include <thrust/host_vector.h>
#include <thrust/scan.h>
#include "common.h"
#include "thrust.h"

namespace StreamCompaction {
    namespace Thrust {
        using StreamCompaction::Common::PerformanceTimer;
        PerformanceTimer& timer()
        {
            static PerformanceTimer timer;
            return timer;
        }
        /**
         * Performs prefix-sum (aka scan) on idata, storing the result into odata.
         */
        void scan(int n, int *odata, const int *idata) {
            // TODO use `thrust::exclusive_scan`
            // example: for device_vectors dv_in and dv_out:
            // thrust::exclusive_scan(dv_in.begin(), dv_in.end(), dv_out.begin());

			thrust::host_vector<int> host_in(idata, idata + n);
			thrust::device_vector<int> dv_in = host_in;
			thrust::device_vector<int> dv_out(n);

			timer().startGpuTimer();
			thrust::exclusive_scan(dv_in.begin(), dv_in.end(), dv_out.begin());
			timer().endGpuTimer();

			thrust::copy(dv_out.begin(), dv_out.end(), odata);
        }
    }
}
