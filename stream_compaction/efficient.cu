#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include "common.h"
#include "efficient.h"



/*! Block size used for CUDA kernel launch. */
#define blockSize 128

namespace StreamCompaction {
    namespace Efficient {
        using StreamCompaction::Common::PerformanceTimer;
        PerformanceTimer& timer()
        {
            static PerformanceTimer timer;
            return timer;
        }
		
		__global__ void kernUpSweep(int n, int p, int *idata) {
			int index = threadIdx.x + (blockIdx.x * blockDim.x);
			if (index >= n) {
				return;
			}

			if (index % (2 * p) == 0) {
				idata[index + 2 * p - 1] += idata[index + p - 1];
			}
			
		}

		__global__ void kernDownSweep(int n, int p, int *idata) {
			int index = threadIdx.x + (blockIdx.x * blockDim.x);
			if (index >= n) {
				return;
			}

			if (index % (2 * p) == 0) {
				int t = idata[index + p - 1];
				idata[index + p - 1] = idata[index + (2 * p) - 1];
				idata[index + (2 * p) - 1] += t;
			}
		}

		void workEfficientScan(int n, int *dev_idata, dim3 &threadsPerBlock, dim3 &fullBlocksPerGrid) {
			//perform upsweep parallel reduction
			for (int d = 0; d < ilog2ceil(n); d++) {
				int p = 1 << d;
				kernUpSweep << <fullBlocksPerGrid, threadsPerBlock >> > (n, p, dev_idata);
				checkCUDAError("kernel kernUpSweep failed!");
			}

			//set root to 0
			hipMemset(dev_idata + n - 1, 0, sizeof(int));

			//perform down sweep as binary tree
			for (int d = ilog2ceil(n)-1 ; d >= 0; d--) {
				int p = 1 << d;
				kernDownSweep << <fullBlocksPerGrid, threadsPerBlock >> > (n, p, dev_idata);
				checkCUDAError("kernel kernDownSweep failed!");
			}

		}

        /**
         * Performs prefix-sum (aka scan) on idata, storing the result into odata.
         */
        void scan(int n, int *odata, const int *idata) {
			//set up variable, allocate space on gpu, and copy over data
			int npad =  1 << ilog2ceil(n); //pads adds padding if needed for arrays of not power of 2 length
			int *dev_idata;

			hipMalloc((void**)&dev_idata, npad * sizeof(int));
			checkCUDAError("hipMalloc dev_idata failed!");

			hipMemcpy(dev_idata, idata, sizeof(int) * n, hipMemcpyHostToDevice);
			checkCUDAError("Memcpy idata failed!");

			dim3 threadsPerBlock(blockSize);
			dim3 fullBlocksPerGrid((npad + blockSize - 1) / blockSize);

            timer().startGpuTimer();

			//call work efficient scan helper
			workEfficientScan(npad, dev_idata, threadsPerBlock, fullBlocksPerGrid);

            timer().endGpuTimer();

			hipMemcpy(odata, dev_idata, sizeof(int) * n, hipMemcpyDeviceToHost);
			checkCUDAError("Memcpy odata failed!");

			hipFree(dev_idata);
        }

        /**
         * Performs stream compaction on idata, storing the result into odata.
         * All zeroes are discarded.
         *
         * @param n      The number of elements in idata.
         * @param odata  The array into which to store elements.
         * @param idata  The array of elements to compact.
         * @returns      The number of elements remaining after compaction.
         */
        int compact(int n, int *odata, const int *idata) {
			//set up variable, allocate space on gpu, and copy over data
			int npad = 1 << ilog2ceil(n);
			int *dev_idata;
			int *dev_indices;
			int *dev_odata;
			int *dev_bools;

			hipMalloc((void**)&dev_idata, npad * sizeof(int));
			checkCUDAError("hipMalloc dev_idata failed!");

			hipMalloc((void**)&dev_odata, npad * sizeof(int));
			checkCUDAError("hipMalloc dev_odata failed!");

			hipMalloc((void**)&dev_indices, npad * sizeof(int));
			checkCUDAError("hipMalloc dev_indices failed!");

			hipMalloc((void**)&dev_bools, npad * sizeof(int));
			checkCUDAError("hipMalloc dev_bools failed!");

			hipMemcpy(dev_idata, idata, sizeof(int) * n, hipMemcpyHostToDevice);
			checkCUDAError("Memcpy idata failed!");

			dim3 threadsPerBlock(blockSize);
			dim3 fullBlocksPerGrid((npad + blockSize - 1) / blockSize);

            timer().startGpuTimer();

			// map data to boolean
			StreamCompaction::Common::kernMapToBoolean<< <fullBlocksPerGrid, threadsPerBlock >> > (npad, dev_bools, dev_idata);
			checkCUDAError("Memcpy kernMapToBoolean failed!");

			//copy to indices to call on workEfficientScan inplace
			hipMemcpy(dev_indices, dev_bools, sizeof(int) * npad, hipMemcpyDeviceToDevice);
			checkCUDAError("Memcpy odata failed!");

			//call work efficient exclusive scan helper to edit dev_indices inplace
			workEfficientScan(npad, dev_indices, threadsPerBlock, fullBlocksPerGrid);

			//scatter results from scan
			StreamCompaction::Common::kernScatter << <fullBlocksPerGrid, threadsPerBlock >> > (npad, dev_odata, dev_idata, dev_bools, dev_indices);
			checkCUDAError("Memcpy kernScatter failed!");

            timer().endGpuTimer();

			//copy back output
			hipMemcpy(odata, dev_odata, sizeof(int) * n, hipMemcpyDeviceToHost);
			checkCUDAError("Memcpy odata failed!");

			// find length of output array as final element of indices array
			int *k = new int;
			hipMemcpy(k, dev_indices + npad - 1, sizeof(int), hipMemcpyDeviceToHost);
			checkCUDAError("Memcpy dev_indices failed!");

			hipFree(dev_idata);
			hipFree(dev_indices);
			hipFree(dev_odata);
			hipFree(dev_bools);

            return *k;
        }
    }
}
