#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include "common.h"
#include ""
#include "efficient.h"

namespace StreamCompaction {
    namespace Efficient {
        using StreamCompaction::Common::PerformanceTimer;
        PerformanceTimer& timer()
        {
            static PerformanceTimer timer;
            return timer;
        }

		__global__ void kernelUpSweepStep(int n, int d, int* cdata) {
			int k = (blockIdx.x * blockDim.x) + threadIdx.x;
			if (k > n)
				return;
			int prev_step_size = 1 << d;
			int cur_step_size = 2 * prev_step_size;
			if (k % cur_step_size == 0)
				cdata[k + cur_step_size - 1] += cdata[k + prev_step_size - 1];
		}

		__global__ void kernelUpSweepStepEfficient(int n, int d, int* cdata) {
			int k = (blockIdx.x * blockDim.x) + threadIdx.x;
			if (k >= n)
				return;
			int prev_step_size = 1 << d;
			int cur_step_size = 2 * prev_step_size;
			int new_offset = k * cur_step_size;
			cdata[new_offset + cur_step_size - 1] += cdata[new_offset + prev_step_size - 1];
		}

		__global__ void kernelDownSweepStep(int n, int d, int* cdata) {
			int k = (blockIdx.x * blockDim.x) + threadIdx.x;
			if (k > n)
				return;
			int left_step = 1 << d;
			int cur_step = 2 * left_step;

			if (k % cur_step == 0) {
				int temp = cdata[k + left_step - 1];
				cdata[k + left_step - 1] = cdata[k + cur_step - 1];
				cdata[k + cur_step - 1] += temp;
			}
		}

		__global__ void kernelDownSweepStepEfficient(int n, int d, int* cdata) {
			int k = (blockIdx.x * blockDim.x) + threadIdx.x;
			if (k >= n)
				return;

			int prev_step_size = 1 << d;
			int cur_step_size = 2 * prev_step_size;
			int new_offset = k * cur_step_size;
			
			int temp = cdata[new_offset + prev_step_size - 1];
			cdata[new_offset + prev_step_size - 1] = cdata[new_offset + cur_step_size - 1];
			cdata[new_offset + cur_step_size - 1] += temp;
		}

		void printArray(int n, int *a, bool abridged = false) {
			printf("    [ ");
			for (int i = 0; i < n; i++) {
				if (abridged && i + 2 == 15 && n > 16) {
					i = n - 2;
					printf("... ");
				}
				printf("%3d ", a[i]);
			}
			printf("]\n");
		}

		void printCudaArray(int size, int* data) {
			int *d_data = new int[size];
			hipMemcpy(d_data, data, size * sizeof(int), hipMemcpyDeviceToHost);
			printArray(size, d_data, true);
		}

        /**
         * Performs prefix-sum (aka scan) on idata, storing the result into odata.
         */
        void scanEfficient(int n, int *odata, const int *idata, int blockSize) {
			// Memory Allocation and Copying
			int power_size = pow(2, ilog2ceil(n));
			int *cdata;
			hipMalloc((void**)&cdata, power_size * sizeof(int));
			checkCUDAErrorFn("hipMalloc adata failed!");
			hipMemset(cdata, 0, power_size * sizeof(int));
			hipMemcpy(cdata, idata, n * sizeof(int), hipMemcpyHostToDevice);

			bool started_timer = true;
			try {
				timer().startGpuTimer();
			}
			catch (const std::exception& e) {
				started_timer = false;
			}

			int numThreads;
			//Up Sweep
			for (int d = 0; d <= ilog2ceil(power_size) - 1 ; d++) {
				numThreads = pow(2, (ilog2ceil(power_size) - 1 - d));
				dim3 fullBlocks((numThreads + blockSize - 1) / blockSize);
				kernelUpSweepStepEfficient <<<fullBlocks, blockSize>>> (numThreads, d, cdata);
			}

			//Down Sweep
			hipMemset(cdata + power_size - 1, 0, sizeof(int));
			for (int d = ilog2(power_size) - 1; d >= 0; d--) {
				numThreads = pow(2, (ilog2ceil(power_size) - 1 - d));
				dim3 fullBlocks((numThreads + blockSize - 1) / blockSize);
				kernelDownSweepStepEfficient <<<fullBlocks, blockSize>>> (numThreads, d, cdata);
			}

			if (started_timer)
				timer().endGpuTimer();

			// Copy Back and Free Memory
			hipMemcpy(odata, cdata, sizeof(int) * n, hipMemcpyDeviceToHost);
			hipFree(cdata);
        }

		/**
		 * Performs prefix-sum (aka scan) on idata, storing the result into odata.
		 */
		 void scanEfficientCUDA(int n, int *odata, const int *idata, int blockSize) {
			 // Memory Allocation and Copying
			 int power_size = pow(2, ilog2ceil(n));
			 int *cdata;
			 hipMalloc((void**)&cdata, power_size * sizeof(int));
			 checkCUDAErrorFn("hipMalloc adata failed!");
			 hipMemset(cdata, 0, power_size * sizeof(int));
			 hipMemcpy(cdata, idata, n * sizeof(int), hipMemcpyDeviceToDevice);

			 bool started_timer = true;
			 try {
				 timer().startGpuTimer();
			 }
			 catch (const std::exception& e) {
				 started_timer = false;
			 }

			 int numThreads;
			 //Up Sweep
			 for (int d = 0; d <= ilog2ceil(power_size) - 1; d++) {
				 numThreads = pow(2, (ilog2ceil(power_size) - 1 - d));
				 dim3 fullBlocks((numThreads + blockSize - 1) / blockSize);
				 kernelUpSweepStepEfficient << <fullBlocks, blockSize >> > (numThreads, d, cdata);
			 }

			 //Down Sweep
			 hipMemset(cdata + power_size - 1, 0, sizeof(int));
			 for (int d = ilog2(power_size) - 1; d >= 0; d--) {
				 numThreads = pow(2, (ilog2ceil(power_size) - 1 - d));
				 dim3 fullBlocks((numThreads + blockSize - 1) / blockSize);
				 kernelDownSweepStepEfficient << <fullBlocks, blockSize >> > (numThreads, d, cdata);
			 }

			 if (started_timer)
				 timer().endGpuTimer();

			 // Copy Back and Free Memory
			 hipMemcpy(odata, cdata, sizeof(int) * n, hipMemcpyDeviceToDevice);
			 hipFree(cdata);
		}

		/**
		 * Performs prefix-sum (aka scan) on idata, storing the result into odata.
		 */
		void scan(int n, int *odata, const int *idata, int blockSize) {
			// Memory Allocation and Copying
			int power_size = pow(2, ilog2ceil(n));
			int *cdata;
			hipMalloc((void**)&cdata, power_size * sizeof(int));
			checkCUDAErrorFn("hipMalloc adata failed!");
			hipMemset(cdata, 0, power_size * sizeof(int));
			hipMemcpy(cdata, idata, n * sizeof(int), hipMemcpyHostToDevice);

			bool started_timer = true;
			try {
				timer().startGpuTimer();
			}
			catch (const std::exception& e) {
				started_timer = false;
			}
			dim3 fullBlocksPerGrid((power_size + blockSize - 1) / blockSize);

			//Up Sweep
			for (int d = 0; d < ilog2ceil(power_size); d++) {
				kernelUpSweepStep << <fullBlocksPerGrid, blockSize >> > (power_size, d, cdata);
			}

			//Down Sweep
			hipMemset(cdata + power_size - 1, 0, sizeof(int));

			for (int d = ilog2(power_size) - 1; d >= 0; d--) {
				kernelDownSweepStep << <fullBlocksPerGrid, blockSize >> > (power_size, d, cdata);
			}
			if (started_timer)
				timer().endGpuTimer();

			// Copy Back and Free Memory
			hipMemcpy(odata, cdata, sizeof(int) * n, hipMemcpyDeviceToHost);
			hipFree(cdata);
		}

        /**
         * Performs stream compaction on idata, storing the result into odata.
         * All zeroes are discarded.
         *
         * @param n      The number of elements in idata.
         * @param odata  The array into which to store elements.
         * @param idata  The array of elements to compact.
         * @returns      The number of elements remaining after compaction.
         */
        int compact(int n, int *odata, const int *idata, bool efficient, int blockSize) {
			// Memory Allocation and Copying
			int *bools = new int[n];
			int *indices = new int[n];
			int *dev_bools;
			int *dev_indices;
			int *dev_idata;
			int *dev_odata;
			hipMalloc((void**)&dev_bools, n * sizeof(int));
			checkCUDAErrorFn("hipMalloc dev_bools failed!");
			hipMalloc((void**)&dev_indices, n * sizeof(int));
			checkCUDAErrorFn("hipMalloc dev_indices failed!");
			hipMalloc((void**)&dev_idata, n * sizeof(int));
			checkCUDAErrorFn("hipMalloc dev_idata failed!");
			hipMalloc((void**)&dev_odata, n * sizeof(int));
			checkCUDAErrorFn("hipMalloc dev_odata failed!");
			hipMemcpy(dev_idata, idata, sizeof(int) * n, hipMemcpyHostToDevice);

			timer().startGpuTimer();
			dim3 fullBlocksPerGrid((n + blockSize - 1) / blockSize);
			StreamCompaction::Common::kernMapToBoolean << <fullBlocksPerGrid, blockSize >> > (n, dev_bools, dev_idata);
			hipMemcpy(bools, dev_bools, sizeof(int) * n, hipMemcpyDeviceToHost);
			if(efficient)
				scanEfficient(n, indices, bools, blockSize);
			else
				scan(n, indices, bools, blockSize);
			hipMemcpy(dev_indices, indices, sizeof(int) * n, hipMemcpyHostToDevice);
			StreamCompaction::Common::kernScatter << <fullBlocksPerGrid, blockSize >> > (n, dev_odata, dev_idata, dev_bools, dev_indices);			
			timer().endGpuTimer();

			// Copy Back and Free Memory
			hipMemcpy(odata, dev_odata, sizeof(int) * n, hipMemcpyDeviceToHost);
			hipFree(dev_bools);
			hipFree(dev_indices);
			hipFree(dev_idata);
			hipFree(dev_odata);
            return indices[n - 1] + bools[n - 1];;
        }
    }
}
