#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include "common.h"
#include "shared.h"
#include "efficient.h"
#include ""


#define checkCUDAErrorWithLine(msg) checkCUDAError(msg, __LINE__)

//#define blockSize 128

int* dev_idata_shared;
int* padded_idata_shared;

namespace StreamCompaction {
	namespace Shared {
		using StreamCompaction::Common::PerformanceTimer;
		PerformanceTimer& timer()
		{
			static PerformanceTimer timer;
			return timer;
		}

		void printArray(int n, int *a, bool abridged = false) {
			printf("    [ ");
			for (int i = 0; i < n; i++) {
				if (abridged && i + 2 == 15 && n > 16) {
					i = n - 2;
					printf("... ");
				}
				printf("%3d ", a[i]);
			}
			printf("]\n");
		}

		__global__ void upSweep(int n, int d, int* A) {
			int index = threadIdx.x + (blockIdx.x * blockDim.x);
			if (index >= n) {
				return;
			}
			int stride = 1 << (d + 1);
			int other_index = 1 << d;
			if ((index) % stride == 0) {
				A[index + stride - 1] += A[index + other_index - 1];
			}
		}

		__global__ void downSweep(int n, int d, int* A) {
			int index = threadIdx.x + (blockIdx.x * blockDim.x);
			if (index >= n) {
				return;
			}
			int left_index = 1 << (d);
			int right_index = 1 << (d + 1);
			if (index % right_index == 0) {
				int temp = A[index + left_index - 1];
				A[index + left_index - 1] = A[index + right_index - 1];
				A[index + right_index - 1] += temp;
			}
		}


		__global__ void scan_array(int n, int* A, int* B, int* intermediate) {
			int index = threadIdx.x + (blockIdx.x * blockDim.x);

			if (index >= n) {
				return;
			}
			int BLOCKSIZE = blockDim.x;
			__shared__ int s[1024];

			int tid = threadIdx.x;
			//Copy Global Memory to Shared
			s[tid] = A[threadIdx.x + (blockIdx.x * BLOCKSIZE)];

			int iterations = ilog2ceil(BLOCKSIZE);

			//Up Sweep
			for (int d = 0; d < ilog2(BLOCKSIZE); d++) {
				__syncthreads();
				int stride = 1 << (d + 1);
				int other_index = 1 << d;
				if ((tid) % stride == 0) {
					s[tid + stride - 1] += s[tid + other_index - 1];
				}
			}
			__syncthreads();

			if (threadIdx.x == 0) {
				//Add last element of block (upsweep) to intermediate array
				intermediate[blockIdx.x] = s[BLOCKSIZE - 1];
				s[BLOCKSIZE - 1] = 0;
			}
			
			//__syncthreads();
			
			//Down Sweep
			for (int d = iterations - 1; d >= 0; d--) {
				__syncthreads();
				int left_index = 1 << (d);
				int right_index = 1 << (d + 1);
				if (tid % right_index == 0) {
					int temp = s[tid + left_index - 1];
					s[tid + left_index - 1] = s[tid + right_index - 1];
					s[tid + right_index - 1] += temp;
				}
			}

			//Copy Result Back to Global Memory
			 B[threadIdx.x + (blockIdx.x * BLOCKSIZE)] = s[threadIdx.x];
			
		}

		__global__ void merge(int n, int* A, int* intermediate) {
			int index = threadIdx.x + (blockIdx.x * blockDim.x);

			if (index >= n) {
				return;
			}
			if(blockIdx.x > 0)
				A[index] = A[index] + intermediate[blockIdx.x];
		}

		/**
		 * Performs prefix-sum (aka scan) on idata, storing the result into odata.
		 */
		void scan(int n, int *odata, const int *idata, int blockSize) {
			int* dev_odata;
			int* dev_intermediate;
			int* dev_intermediate_scan;

			int padded_size = 1 << (ilog2ceil(n));
			int numberOfBlocks = (padded_size + blockSize - 1) / blockSize;

			hipMalloc((void**)&padded_idata_shared, padded_size * sizeof(int));
			checkCUDAErrorWithLine("hipMalloc padded_idata_shared failed!");

			hipMalloc((void**)&dev_odata, padded_size * sizeof(int));
			checkCUDAErrorWithLine("hipMalloc padded_idata_shared failed!");

			hipMalloc((void**)&dev_intermediate, numberOfBlocks * sizeof(int));
			checkCUDAErrorWithLine("hipMalloc padded_idata_shared failed!");

			hipMalloc((void**)&dev_intermediate_scan, numberOfBlocks * sizeof(int));
			checkCUDAErrorWithLine("hipMalloc padded_idata_shared failed!");


			hipMemset(padded_idata_shared, 0, padded_size * sizeof(int));
			hipMemcpy(padded_idata_shared, idata, sizeof(int) * n, hipMemcpyHostToDevice);

			bool caught = false;
			try {
				timer().startGpuTimer();
			}
			catch (const std::exception& e) {
				caught = true;
			}

			dim3 fullBlocksPerGrid(numberOfBlocks);

			printf("Inside scanning \n");
			scan_array << <fullBlocksPerGrid, blockSize >> > (padded_size, padded_idata_shared, dev_odata, dev_intermediate);
			printf("Inside scanning done \n");
			//Scan the intermediate array (without shared memory in GPU)
			StreamCompaction::Efficient::scan_device(numberOfBlocks, dev_intermediate_scan, dev_intermediate, blockSize);
			printf("Scanning intermediate done \n");
			//Add the elements of dev_intermediate to dev_odata
			merge << <fullBlocksPerGrid, blockSize >> > (padded_size, dev_odata, dev_intermediate_scan);
			printf("Merge done \n");
			if (!caught) {
				timer().endGpuTimer();
			}

			hipMemcpy(odata, dev_odata, sizeof(int) * n, hipMemcpyDeviceToHost);

			hipFree(padded_idata_shared);
			hipFree(dev_odata);
			hipFree(dev_intermediate);
			hipFree(dev_intermediate_scan);
		}
		

		/**
		 * Performs stream compaction on idata, storing the result into odata.
		 * All zeroes are discarded.
		 *
		 * @param n      The number of elements in idata.
		 * @param odata  The array into which to store elements.
		 * @param idata  The array of elements to compact.
		 * @returns      The number of elements remaining after compaction.
		 */
		int compact(int n, int *idata, int blockSize) {

			//hipMemcpy(dev_idata_shared, idata, sizeof(int) * n, hipMemcpyHostToDevice);
			printf("N: %d\n", n);
			int* dev_bools;
			hipMalloc((void**)&dev_bools, n * sizeof(int));
			checkCUDAErrorWithLine("hipMalloc dev_bools failed!");
			int* bools;
			bools = (int*)malloc(n * sizeof(int));
			int* indices;
			indices = (int*)malloc(n * sizeof(int));
			int *dev_indices;
			hipMalloc((void**)&dev_indices, n * sizeof(int));
			checkCUDAErrorWithLine("hipMalloc dev_bools failed!");

			timer().startGpuTimer();
			dim3 fullBlocksPerGrid((n + blockSize - 1) / blockSize);

			printf("bOOL \n");
			StreamCompaction::Common::kernMapToBoolean << <fullBlocksPerGrid, blockSize >> > (n, dev_bools, idata);


			hipMemcpy(bools, dev_bools, sizeof(int) * n, hipMemcpyDeviceToHost);

			printf("Scanning \n");
			scan(n, indices, bools, blockSize);
			printf("Scanning done \n");

			int output_length = indices[n - 1] + bools[n - 1];
			printf("Output length: %d", output_length);

			hipMemcpy(dev_indices, indices, sizeof(int) * n, hipMemcpyHostToDevice);

			int *dev_odata;
			hipMalloc((void**)&dev_odata, n * sizeof(int));
			checkCUDAErrorWithLine("hipMalloc dev_odata failed!");

			printf("Scattering \n");
			StreamCompaction::Common::kernScatter << <fullBlocksPerGrid, blockSize >> > (n, dev_odata, idata, dev_bools, dev_indices);
			printf("Scattering done \n");
			hipMemcpy(idata, dev_odata, sizeof(int) * output_length, hipMemcpyDeviceToDevice);

			timer().endGpuTimer();

			hipFree(dev_bools);
			hipFree(dev_indices);
			hipFree(dev_odata);
			return output_length;
		}
	}
}
