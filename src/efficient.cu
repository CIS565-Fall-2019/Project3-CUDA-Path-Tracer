#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include "common.h"
#include "efficient.h"
//#include "thrust.h"
#include <thrust/scan.h>
#include <thrust/device_vector.h>

#define checkCUDAErrorWithLine(msg) checkCUDAError(msg)

namespace StreamCompaction {
    namespace Efficient {
        using StreamCompaction::Common::PerformanceTimer;
        PerformanceTimer& timer()
        {
            static PerformanceTimer timer;
            return timer;
        }

		void printxxx(int n, const int *a) {
			for (int i = 0; i < n; i++) {
				printf("%d ", a[i]);
			}
			printf("\n\n\n");
		}

		__global__ void resetZeros(int n, int *a) {
			int index = (blockDim.x*blockIdx.x) + threadIdx.x;
			if (index >= n) return;
			a[index] = 0;
		}



		__global__ void kernscanBlock(int n, int *odata, int* out_last, const int *idata) {
			
			extern __shared__ int temp[];

			int idx = threadIdx.x;
			int tid = (blockDim.x*blockIdx.x) + threadIdx.x;
			int numPerBlock = 2 * blockDim.x;

			if (tid >= n) return;

			// copy the data this idx boi has to work with to shared memory
			temp[2*idx] = idata[2*tid];
			temp[2*idx + 1] = idata[2*tid + 1];

			int offset = 1;
			for (int d = numPerBlock>> 1; d > 0; d >>=1) {
				__syncthreads();

				if (idx < d) {

					int k1 = offset * (2 * idx + 1) - 1;
					int k2 = offset * (2 * idx + 2) - 1;
					temp[k2] += temp[k1];
				}

				offset = 2 * offset;
			}

			if (idx == 0) { temp[numPerBlock - 1] = 0; }

			for (int d = 1; d < numPerBlock; d *= 2) {
				offset >>= 1;
				__syncthreads();
				if (idx < d) {

					int k1 = offset * (2 * idx + 1) - 1;
					int k2 = offset * (2 * idx + 2) - 1;

					int tmp = temp[k1];
					temp[k1] = temp[k2];
					temp[k2] += tmp;
				}
			}

			__syncthreads();
			
			odata[2 * tid] = temp[2 * idx]; // has to updated with block number
			odata[2 * tid + 1] = temp[2 * idx + 1];

			if (idx == 0) {
				int last = numPerBlock * blockIdx.x + numPerBlock - 1;
				out_last[blockIdx.x] = temp[numPerBlock - 1] + idata[last];
			}
		}

		__global__ void copyLastElements(int n, int blockSize, int *odata, const int *idata) {
			int tid = (blockDim.x*blockIdx.x) + threadIdx.x;
			if (tid >= n) return;

			odata[tid] = idata[tid*blockSize + blockSize - 1];
		}

		__global__ void addLastElement(int n, int blockSize, int *odata, const int *scanSum, const int *idata) {
			int tid = (blockDim.x*blockIdx.x) + threadIdx.x;
			if (tid >= n) return;

			odata[tid] = scanSum[tid] + idata[tid*blockSize + blockSize - 1];
			//odata[tid] = scanSum[tid];
		}

		__global__ void addScanMain(int n, int *odata, const int *scanSum, const int *scanSumBlock) {
			int tid = (blockDim.x*blockIdx.x) + threadIdx.x;
			if (tid >= n) return;

			odata[tid] = scanSumBlock[tid] + scanSum[blockIdx.x];
		}


		void scanShared(int n, int *odata, const int *idata) {
			bool exception = false;

			int *dev_idata, *dev_scanSumBlock, *dev_addLastElements, *dev_scanSum, *dev_odata;

			int d_max = ilog2ceil(n);

			int twoPowN = 1 << d_max;
			if (n != twoPowN) {

				int diff = twoPowN - n;

				hipMalloc((void **)&dev_idata, (n + diff) * sizeof(int));
				checkCUDAErrorWithLine("hipMalloc dev_odata1 failed!");

				int threadsPerBlock = 1024;
				int blocksToLaunch = (n + diff + threadsPerBlock - 1) / threadsPerBlock;
				resetZeros << <blocksToLaunch, threadsPerBlock >> > (n + diff, dev_idata);

				hipMemcpy(dev_idata, idata, n * sizeof(int), hipMemcpyHostToDevice);
				n = n + diff;
			}
			else {
				hipMalloc((void **)&dev_idata, n * sizeof(int));
				checkCUDAErrorWithLine("hipMalloc dev_idata failed!");

				hipMemcpy(dev_idata, idata, n * sizeof(int), hipMemcpyHostToDevice);
			}

			int blockSize = 1024;
			int numBlocks = (n + blockSize - 1) / blockSize;
			int numElements = numBlocks;

			hipMalloc((void **)&dev_scanSumBlock, n * sizeof(int));
			hipMalloc((void **)&dev_addLastElements, numElements * sizeof(int));
			hipMalloc((void **)&dev_scanSum, numElements * sizeof(int));
			hipMalloc((void **)&dev_odata, n * sizeof(int));
			checkCUDAErrorWithLine("hipMalloc failed!");

			hipMemcpy(dev_idata, idata, n * sizeof(int), hipMemcpyHostToDevice);

			//thrust::device_ptr<int> dev_idataItr(dev_addLastElements);
			//thrust::device_ptr<int> dev_odataItr(dev_scanSum);

			try {
				timer().startGpuTimer();
			}
			catch (const std::runtime_error& ex) {
				exception = true;
			}

			kernscanBlock << <numBlocks, blockSize/2, (blockSize) * sizeof(int) >> > (n, dev_scanSumBlock, dev_addLastElements, dev_idata);

			scanCompact(numElements, dev_scanSum, dev_addLastElements);
			//thrust::exclusive_scan(dev_idataItr, dev_idataItr + numElements, dev_odataItr);

			addScanMain<<<numBlocks, blockSize >>>(n, dev_odata, dev_scanSum, dev_scanSumBlock);

			if (!exception)
				timer().endGpuTimer();

			hipMemcpy(odata, dev_odata, n * sizeof(int), hipMemcpyDeviceToHost);

			hipFree(dev_idata);
			hipFree(dev_scanSum);
			hipFree(dev_scanSumBlock);
			hipFree(dev_addLastElements);
			hipFree(dev_odata);
		}

		void scanSharedGPU(int n, int *dev_odata, const int *idata) {
			bool exception = false;

			int *dev_idata, *dev_scanSumBlock, *dev_addLastElements, *dev_scanSum;

			int d_max = ilog2ceil(n);

			int twoPowN = 1 << d_max;
			if (n != twoPowN) {

				int diff = twoPowN - n;

				hipMalloc((void **)&dev_idata, (n + diff) * sizeof(int));
				checkCUDAErrorWithLine("hipMalloc dev_odata1 failed!");

				int threadsPerBlock = 1024;
				int blocksToLaunch = (n + diff + threadsPerBlock - 1) / threadsPerBlock;
				resetZeros << <blocksToLaunch, threadsPerBlock >> > (n + diff, dev_idata);

				hipMemcpy(dev_idata, idata, n * sizeof(int), hipMemcpyDeviceToDevice);
				n = n + diff;
			}
			else {
				hipMalloc((void **)&dev_idata, n * sizeof(int));
				checkCUDAErrorWithLine("hipMalloc dev_idata failed!");

				hipMemcpy(dev_idata, idata, n * sizeof(int), hipMemcpyDeviceToDevice);
			}

			int blockSize = 512;
			int numBlocks = (n + blockSize - 1) / blockSize;
			int numElements = numBlocks;

			hipMalloc((void **)&dev_scanSumBlock, n * sizeof(int));
			hipMalloc((void **)&dev_addLastElements, numElements * sizeof(int));
			hipMalloc((void **)&dev_scanSum, numElements * sizeof(int));
			checkCUDAErrorWithLine("hipMalloc failed!");

			//thrust::device_ptr<int> dev_idataItr(dev_addLastElements);
			//thrust::device_ptr<int> dev_odataItr(dev_scanSum);

			try {
				timer().startGpuTimer();
			}
			catch (const std::runtime_error& ex) {
				exception = true;
			}

			kernscanBlock << <numBlocks, blockSize / 2, (blockSize) * sizeof(int) >> > (n, dev_scanSumBlock, dev_addLastElements, dev_idata);

			//int *a = new int[n];
			//hipMemcpy(a, dev_scanSumBlock, n * sizeof(int), hipMemcpyDeviceToHost);
			//printxxx(n, a);

			scanCompact(numElements, dev_scanSum, dev_addLastElements);
			//thrust::exclusive_scan(dev_idataItr, dev_idataItr + numElements, dev_odataItr);

			addScanMain << <numBlocks, blockSize >> > (n, dev_odata, dev_scanSum, dev_scanSumBlock);

			if (!exception)
				timer().endGpuTimer();

			hipFree(dev_idata);
			hipFree(dev_scanSum);
			hipFree(dev_scanSumBlock);
			hipFree(dev_addLastElements);
		}

		__global__ void upSweep(int n, int d, int *idata) {
			int index = (blockDim.x*blockIdx.x) + threadIdx.x;

			int twoPowd1 = 1 << (d + 1);
			int twoPowd = 1 << d;


			if ((index % twoPowd1 != twoPowd1-1) || index >= n) return;

			int k = index - twoPowd1 + 1;
			idata[index] += idata[k + twoPowd - 1];
		}

		__global__ void downSweep(int n, int d, int *idata) {
			int index = (blockDim.x*blockIdx.x) + threadIdx.x;

			int twoPowd1 = 1 << (d + 1);
			int twoPowd = 1 << d;


			if ((index % twoPowd1 != twoPowd1 - 1) || index >= n) return;

			int k = index - twoPowd1 + 1;
			int t = idata[k + twoPowd - 1];
			idata[k + twoPowd - 1] = idata[index];
			idata[index] += t;
		}


        /**
         * Performs prefix-sum (aka scan) on idata, storing the result into odata.
         */
        void scan(int n, int *odata, const int *idata) {
			bool exception = false;

			int *dev_idata;

			int numThreads = 128;
			int numBlocks = (n + numThreads - 1) / numThreads;

			int d_max = ilog2ceil(n);

			int twoPowN = 1 << d_max;
			if (n != twoPowN) {

				int diff = twoPowN - n;

				hipMalloc((void **)&dev_idata, (n + diff) * sizeof(int));
				checkCUDAErrorWithLine("hipMalloc dev_odata1 failed!");

				resetZeros << <numBlocks, numThreads >> > (n + diff, dev_idata);

				hipMemcpy(dev_idata, idata, n * sizeof(int), hipMemcpyHostToDevice);
				n = n + diff;
			} else {
				hipMalloc((void **)&dev_idata, n * sizeof(int));
				checkCUDAErrorWithLine("hipMalloc dev_idata failed!");

				hipMemcpy(dev_idata, idata, n * sizeof(int), hipMemcpyHostToDevice);
			}

			try {
				timer().startGpuTimer();
			}
			catch (const std::runtime_error& ex) {
				exception = true;
			}


			for (int d = 0; d < d_max; d++) {
				upSweep<<<numBlocks, numThreads>>>(n, d, dev_idata);
			}

			// reset last element to zero
			//int* zero = new int[1];
			//zero[0] = 0;
			//hipMemcpy(dev_idata + n - 1, zero, sizeof(int), hipMemcpyHostToDevice);
			hipMemset(dev_idata + n - 1, 0, sizeof(int));

			
			for(int d = d_max-1; d >= 0; d--) {
				downSweep << <numBlocks, numThreads >> > (n, d, dev_idata);
			}


			if (!exception)
				timer().endGpuTimer();


			hipMemcpy(odata, dev_idata, n * sizeof(int), hipMemcpyDeviceToHost);


			
			hipFree(dev_idata);


        }

		void scanCompact(int n, int *odata, const int *idata) {
			bool exception = false;

			int *dev_idata;

			int numThreads = 128;
			int numBlocks = (n + numThreads - 1) / numThreads;

			int d_max = ilog2ceil(n);

			int twoPowN = 1 << d_max;
			if (n != twoPowN) {

				int diff = twoPowN - n;

				hipMalloc((void **)&dev_idata, (n + diff) * sizeof(int));
				checkCUDAErrorWithLine("hipMalloc dev_odata1 failed!");

				resetZeros << <numBlocks, numThreads >> > (n + diff, dev_idata);

				hipMemcpy(dev_idata, idata, n * sizeof(int), hipMemcpyDeviceToDevice);
				n = n + diff;
			}
			else {
				hipMalloc((void **)&dev_idata, n * sizeof(int));
				checkCUDAErrorWithLine("hipMalloc dev_idata failed!");

				hipMemcpy(dev_idata, idata, n * sizeof(int), hipMemcpyDeviceToDevice);
			}

			try {
				timer().startGpuTimer();
			}
			catch (const std::runtime_error& ex) {
				exception = true;
			}


			for (int d = 0; d < d_max; d++) {
				upSweep << <numBlocks, numThreads >> > (n, d, dev_idata);
			}

			// reset last element to zero
			//int* zero = new int[1];
			//zero[0] = 0;
			//hipMemcpy(dev_idata + n - 1, zero, sizeof(int), hipMemcpyHostToDevice);
			hipMemset(dev_idata + n - 1, 0, sizeof(int));


			for (int d = d_max - 1; d >= 0; d--) {
				downSweep << <numBlocks, numThreads >> > (n, d, dev_idata);
			}

			if (!exception)
				timer().endGpuTimer();

			hipMemcpy(odata, dev_idata, n * sizeof(int), hipMemcpyDeviceToDevice);



			hipFree(dev_idata);
		}


        /**
         * Performs stream compaction on idata, storing the result into odata.
         * All zeroes are discarded.
         *
         * @param n      The number of elements in idata.
         * @param odata  The array into which to store elements.
         * @param idata  The array of elements to compact.
         * @returns      The number of elements remaining after compaction.
         */
        int compact(int n, int *odata, const int *idata) {
            

			int numThreads = 128;
			int numBlocks = (n + numThreads - 1) / numThreads;

			int *dev_checkZeros, *dev_sumIndices, *dev_odata, *dev_idata;

			hipMalloc((void **) &dev_checkZeros, n * sizeof(int));
			checkCUDAErrorWithLine("hipMalloc dev_checkZeros failed!");
			hipMalloc((void **) &dev_sumIndices, n * sizeof(int));
			checkCUDAErrorWithLine("hipMalloc dev_sumIndices failed!");
			hipMalloc((void **)&dev_odata, n * sizeof(int));
			checkCUDAErrorWithLine("hipMalloc dev_odata failed!");
			hipMalloc((void **)&dev_idata, n * sizeof(int));
			checkCUDAErrorWithLine("hipMalloc dev_idata failed!");

			hipMemcpy(dev_idata, idata, n * sizeof(int), hipMemcpyHostToDevice);

			timer().startGpuTimer();

			StreamCompaction::Common::kernMapToBoolean<<<numBlocks, numThreads>>>(n, dev_checkZeros, dev_idata);
			
			int *checkZeros = new int[n];
			hipMemcpy(checkZeros, dev_checkZeros, n * sizeof(int), hipMemcpyDeviceToHost);


			int *sumIndices = new int[n];
			scan(n, sumIndices, checkZeros);

			hipMemcpy(dev_sumIndices, sumIndices , n * sizeof(int), hipMemcpyHostToDevice);

			StreamCompaction::Common::kernScatter<<<numBlocks, numThreads>>>(n, dev_odata, dev_idata, dev_checkZeros, dev_sumIndices);

			
			timer().endGpuTimer();

			hipMemcpy(odata, dev_odata, n * sizeof(int), hipMemcpyDeviceToHost);

			

			int count = checkZeros[n - 1] == 0 ? sumIndices[n - 1] : sumIndices[n - 1] + 1;

			//delete[] checkZeros;
			//delete[] sumIndices;

			//printf("hey\n");

			hipFree(dev_idata);
			hipFree(dev_odata);
			hipFree(dev_checkZeros);
			hipFree(dev_sumIndices);

           
            return count;
        }


		int compactShared(int n, int *dev_idata) {


			int numThreads = 128;
			int numBlocks = (n + numThreads - 1) / numThreads;

			int *dev_checkZeros, *dev_sumIndices, *dev_odata;

			hipMalloc((void **)&dev_checkZeros, n * sizeof(int));
			hipMalloc((void **)&dev_sumIndices, n * sizeof(int));
			hipMalloc((void **)&dev_odata, n * sizeof(int));
			checkCUDAErrorWithLine("hipMalloc dev_odata failed!");

			//hipMemcpy(dev_idata, idata, n * sizeof(int), hipMemcpyHostToDevice);

			timer().startGpuTimer();

			StreamCompaction::Common::kernMapToBoolean << <numBlocks, numThreads >> > (n, dev_checkZeros, dev_idata);

			//int *a = new int[n];
			//hipMemcpy(a, dev_checkZeros, n * sizeof(int), hipMemcpyDeviceToHost);
			//printxxx(n, a);

			scanSharedGPU(n, dev_sumIndices, dev_checkZeros);
			//thrust::device_ptr<int> i1(dev_checkZeros);
			//thrust::device_ptr<int> o1(dev_sumIndices);
			//thrust::exclusive_scan(i1, i1 + n, o1);

			//int *a = new int[n];
			//hipMemcpy(a, dev_sumIndices, n * sizeof(int), hipMemcpyDeviceToHost);
			//printxxx(n, a);

			StreamCompaction::Common::kernScatter << <numBlocks, numThreads >> > (n, dev_odata, dev_idata, dev_checkZeros, dev_sumIndices);

			timer().endGpuTimer();

			hipMemcpy(dev_idata, dev_odata, n * sizeof(int), hipMemcpyDeviceToDevice);

			int *sumIndices = new int[1];
			hipMemcpy(sumIndices, dev_sumIndices + n-1, 1 * sizeof(int), hipMemcpyDeviceToHost);
			int *checkZeros = new int[1];
			hipMemcpy(checkZeros, dev_checkZeros + n-1, 1 * sizeof(int), hipMemcpyDeviceToHost);

			int count = checkZeros[0] == 0 ? sumIndices[0] : sumIndices[0] + 1;

			delete[] checkZeros;
			delete[] sumIndices;

			hipFree(dev_odata);
			hipFree(dev_checkZeros);
			hipFree(dev_sumIndices);

			return count;
		}


		//int compact(int n, int *odata, const int *idata) {


		//	int numThreads = 128;
		//	int numBlocks = (n + numThreads - 1) / numThreads;

		//	int *dev_checkZeros, *dev_sumIndices, *dev_odata, *dev_idata;

		//	hipMalloc((void **)&dev_checkZeros, n * sizeof(int));
		//	checkCUDAErrorWithLine("hipMalloc dev_checkZeros failed!");
		//	hipMalloc((void **)&dev_sumIndices, n * sizeof(int));
		//	checkCUDAErrorWithLine("hipMalloc dev_sumIndices failed!");
		//	hipMalloc((void **)&dev_odata, n * sizeof(int));
		//	checkCUDAErrorWithLine("hipMalloc dev_odata failed!");
		//	hipMalloc((void **)&dev_idata, n * sizeof(int));
		//	checkCUDAErrorWithLine("hipMalloc dev_idata failed!");

		//	hipMemcpy(dev_idata, idata, n * sizeof(int), hipMemcpyHostToDevice);

		//	timer().startGpuTimer();

		//	StreamCompaction::Common::kernMapToBoolean << <numBlocks, numThreads >> > (n, dev_checkZeros, dev_idata);

		//	//int *checkZeros = new int[n];
		//	//hipMemcpy(checkZeros, dev_checkZeros, n * sizeof(int), hipMemcpyDeviceToHost);


		//	//int *sumIndices = new int[n];
		//	scanCompact(n, dev_sumIndices, dev_checkZeros);

		//	//hipMemcpy(dev_sumIndices, sumIndices, n * sizeof(int), hipMemcpyHostToDevice);

		//	StreamCompaction::Common::kernScatter << <numBlocks, numThreads >> > (n, dev_odata, dev_idata, dev_checkZeros, dev_sumIndices);


		//	timer().endGpuTimer();

		//	hipMemcpy(odata, dev_odata, n * sizeof(int), hipMemcpyDeviceToHost);

		//	int *sumIndices = new int[n];
		//	int *checkZeros = new int[n];
		//	hipMemcpy(checkZeros, dev_checkZeros, n * sizeof(int), hipMemcpyDeviceToHost);
		//	hipMemcpy(sumIndices, dev_sumIndices, n * sizeof(int), hipMemcpyDeviceToHost);
		//	int count = checkZeros[n - 1] == 0 ? sumIndices[n - 1] : sumIndices[n - 1] + 1;

		//	//delete[] checkZeros;
		//	//delete[] sumIndices;

		//	//printf("hey\n");

		//	hipFree(dev_idata);
		//	hipFree(dev_odata);
		//	hipFree(dev_checkZeros);
		//	hipFree(dev_sumIndices);

		//	delete[] sumIndices;
		//	delete[] checkZeros;

		//	return count;
		//}
    }
}
