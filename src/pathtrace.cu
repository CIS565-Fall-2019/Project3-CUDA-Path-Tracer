#include "hip/hip_runtime.h"
#include <cstdio>
#include <hip/hip_runtime.h>
#include <hip/hip_math_constants.h>
#include <cmath>
#include <thrust/execution_policy.h>
#include <thrust/random.h>
#include <thrust/remove.h>
#include <thrust/device_ptr.h>
#include <thrust/sort.h>
#include <thrust/partition.h>

#include "sceneStructs.h"
#include "scene.h"
#include "glm/glm.hpp"
#include "glm/gtx/norm.hpp"
#include "utilities.h"
#include "pathtrace.h"
#include "intersections.h"
#include "interactions.h"

static bool b_useFirstCache = true;
#define USE_COMPACTION 0
#define DEPTH_OF_FILED 0
#define MOTION_BLUR 0
#define DIRECT_LIGHT 0
#define SORT_MATERIAL 0

// static utilityCore::PerformanceTimer timer;

#define ERRORCHECK 1

#define FILENAME (strrchr(__FILE__, '/') ? strrchr(__FILE__, '/') + 1 : __FILE__)
#define checkCUDAError(msg) checkCUDAErrorFn(msg, FILENAME, __LINE__)
void checkCUDAErrorFn(const char* msg, const char* file, int line) {
#if ERRORCHECK
	hipDeviceSynchronize();
	hipError_t err = hipGetLastError();
	if (hipSuccess == err) {
		return;
	}

	fprintf(stderr, "CUDA error");
	if (file) {
		fprintf(stderr, " (%s:%d)", file, line);
	}
	fprintf(stderr, ": %s: %s\n", msg, hipGetErrorString(err));
#  ifdef _WIN32
	getchar();
#  endif
	exit(EXIT_FAILURE);
#endif
}

__host__ __device__
thrust::default_random_engine makeSeededRandomEngine(int iter, int index, int depth) {
	int h = utilhash((1 << 31) | (depth << 22) | iter) ^ utilhash(index);
	return thrust::default_random_engine(h);
}

//Kernel that writes the image to the OpenGL PBO directly.
__global__ void sendImageToPBO(uchar4* pbo, glm::ivec2 resolution,
	int iter, glm::vec3* image) {
	int x = (blockIdx.x * blockDim.x) + threadIdx.x;
	int y = (blockIdx.y * blockDim.y) + threadIdx.y;

	if (x < resolution.x && y < resolution.y) {
		int index = x + (y * resolution.x);
		glm::vec3 pix = image[index];

		glm::ivec3 color;
		color.x = glm::clamp((int)(pix.x / iter * 255.0), 0, 255);
		color.y = glm::clamp((int)(pix.y / iter * 255.0), 0, 255);
		color.z = glm::clamp((int)(pix.z / iter * 255.0), 0, 255);

		// Each thread writes one pixel location in the texture (textel)
		pbo[index].w = 0;
		pbo[index].x = color.x;
		pbo[index].y = color.y;
		pbo[index].z = color.z;
	}
}

static Scene* hst_scene = NULL;
static glm::vec3* dev_image = NULL;
static Geom* dev_geoms = NULL;
static Material* dev_materials = NULL;
static PathSegment* dev_paths = NULL;
static ShadeableIntersection* dev_intersections = NULL;

// TODO: static variables for device memory, any extra info you need, etc
// ...
static ShadeableIntersection* dev_first_intersections = NULL;
static PathSegment* dev_first_paths = NULL;
static Geom* dev_lights = NULL;


void pathtraceInit(Scene* scene) {
	hst_scene = scene;
	const Camera& cam = hst_scene->state.camera;
	const int pixelcount = cam.resolution.x * cam.resolution.y;

	hipMalloc(&dev_image, pixelcount * sizeof(glm::vec3));
	hipMemset(dev_image, 0, pixelcount * sizeof(glm::vec3));

	hipMalloc(&dev_paths, pixelcount * sizeof(PathSegment));

	hipMalloc(&dev_geoms, scene->geoms.size() * sizeof(Geom));
	hipMemcpy(dev_geoms, scene->geoms.data(), scene->geoms.size() * sizeof(Geom), hipMemcpyHostToDevice);

	hipMalloc(&dev_materials, scene->materials.size() * sizeof(Material));
	hipMemcpy(dev_materials, scene->materials.data(), scene->materials.size() * sizeof(Material), hipMemcpyHostToDevice);

	hipMalloc(&dev_intersections, pixelcount * sizeof(ShadeableIntersection));
	hipMemset(dev_intersections, 0, pixelcount * sizeof(ShadeableIntersection));

	// TODO: initialize any extra device memeory you need
	hipMalloc(&dev_first_intersections, pixelcount * sizeof(ShadeableIntersection));
	hipMemset(dev_first_intersections, 0, pixelcount * sizeof(ShadeableIntersection));

	hipMalloc(&dev_first_paths, pixelcount * sizeof(PathSegment));

	checkCUDAError("pathtraceInit");
}

void pathtraceFree() {
	hipFree(dev_image);  // no-op if dev_image is null
	hipFree(dev_paths);
	hipFree(dev_geoms);
	hipFree(dev_materials);
	hipFree(dev_intersections);
	// TODO: clean up any extra device memory you created
	hipFree(dev_first_intersections);
	hipFree(dev_first_paths);

	checkCUDAError("pathtraceFree");
}

// Took fromo PBRT 6.2.3

__host__ __device__ glm::vec2 ConcentricSampleDisk(glm::vec2 u) {
	//  <<Map uniform random numbers [0, 1] to [-1, 1]>>
	glm::vec2 uOffset = 2.f * u - glm::vec2(1, 1);

	// << Apply concentric mapping to point >>
	float theta, r;
	if (glm::abs(uOffset.x) > glm::abs(uOffset.y)) {
		r = uOffset.x;
		theta = HIP_PIO4_F * (uOffset.y / uOffset.x);
	}
	else {
		r = uOffset.y;
		theta = HIP_PIO2_F - HIP_PIO4_F * (uOffset.x / uOffset.y);
	}
	return r * glm::vec2(glm::cos(theta), glm::sin(theta));
}


/**
* Generate PathSegments with rays from the camera through the screen into the
* scene, which is the first bounce of rays.
*
* Antialiasing - add rays for sub-pixel sampling
* motion blur - jitter rays "in time"
* lens effect - jitter ray origin positions based on a lens
*/
__global__ void generateRayFromCamera(Camera cam, int iter, int traceDepth, PathSegment* pathSegments)
{
	int x = (blockIdx.x * blockDim.x) + threadIdx.x;
	int y = (blockIdx.y * blockDim.y) + threadIdx.y;

	if (x < cam.resolution.x && y < cam.resolution.y) {
		int index = x + (y * cam.resolution.x);
		PathSegment& segment = pathSegments[index];

		// TODO: implement antialiasing by jittering the ray
		thrust::default_random_engine rng = makeSeededRandomEngine(iter, index, segment.remainingBounces);
		glm::vec2 pLens(0, 0);
#if DEPTH_OF_FILED
		thrust::uniform_real_distribution<float> u01_len(0, 1);
		float lensRadius = 0.08f;
		pLens += lensRadius * ConcentricSampleDisk(glm::vec2(u01_len(rng), u01_len(rng)));
#endif // DEPTH_OF_FILED

#if MOTION_BLUR

		pLens += sinf(iter * 0.1f) *0.1f;

#endif // MOTION_BLUR
		segment.ray.origin = cam.position + glm::vec3(pLens, 0);
		segment.color = glm::vec3(1.0f, 1.0f, 1.0f);

		segment.ray.direction = glm::normalize(cam.view
			- cam.right * cam.pixelLength.x * ((float)x - (float)cam.resolution.x * 0.5f)
			- cam.up * cam.pixelLength.y * ((float)y - (float)cam.resolution.y * 0.5f)
		);

		segment.pixelIndex = index;
		segment.remainingBounces = traceDepth;
	}
}

// TODO:
// computeIntersections handles generating ray intersections ONLY.
// Generating new rays is handled in your shader(s).
// Feel free to modify the code below.
__global__ void computeIntersections(
	int depth
	, int num_paths
	, PathSegment* pathSegments
	, Geom* geoms
	, int geoms_size
	, ShadeableIntersection* intersections
)
{
	int path_index = blockIdx.x * blockDim.x + threadIdx.x;

	if (path_index < num_paths)
	{
		PathSegment pathSegment = pathSegments[path_index];

		float t;
		glm::vec3 intersect_point;
		glm::vec3 normal;
		float t_min = FLT_MAX;
		int hit_geom_index = -1;
		bool outside = true;

		glm::vec3 tmp_intersect;
		glm::vec3 tmp_normal;

		// naive parse through global geoms

		for (int i = 0; i < geoms_size; i++)
		{
			Geom& geom = geoms[i];

			if (geom.type == CUBE)
			{
				t = boxIntersectionTest(geom, pathSegment.ray, tmp_intersect, tmp_normal, outside);
			}
			else if (geom.type == SPHERE)
			{
				t = sphereIntersectionTest(geom, pathSegment.ray, tmp_intersect, tmp_normal, outside);
			}
			// TODO: add more intersection tests here... triangle? metaball? CSG?

			// Compute the minimum t from the intersection tests to determine what
			// scene geometry object was hit first.
			if (t > 0.0f && t_min > t)
			{
				t_min = t;
				hit_geom_index = i;
				intersect_point = tmp_intersect;
				normal = tmp_normal;
			}
		}

		if (hit_geom_index == -1)
		{
			intersections[path_index].t = -1.0f;
		}
		else
		{
			//The ray hits something
			intersections[path_index].t = t_min;
			intersections[path_index].materialId = geoms[hit_geom_index].materialid;
			intersections[path_index].surfaceNormal = normal;
		}
	}
}

__global__ void shadowTest(
	 int num_paths,
	 ShadeableIntersection* intersections,
	 Geom* geoms,
	 int geoms_size,
	 PathSegment* pathSegments) 
{
	int path_index = blockIdx.x * blockDim.x + threadIdx.x;

	if (path_index < num_paths)
	{
		PathSegment pathSegment = pathSegments[path_index];

		float t;
		glm::vec3 intersect_point;
		glm::vec3 normal;
		float t_min = FLT_MAX;
		int hit_geom_index = -1;
		bool outside = true;

		glm::vec3 tmp_intersect;
		glm::vec3 tmp_normal;

		// naive parse through global geoms other than lights

		for (int i = 1; i < geoms_size; i++)
		{
			Geom& geom = geoms[i];

			if (geom.type == CUBE)
			{
				t = boxIntersectionTest(geom, pathSegment.ray, tmp_intersect, tmp_normal, outside);
			}
			else if (geom.type == SPHERE)
			{
				t = sphereIntersectionTest(geom, pathSegment.ray, tmp_intersect, tmp_normal, outside);
			}
			// TODO: add more intersection tests here... triangle? metaball? CSG?

			// Compute the minimum t from the intersection tests to determine what
			// scene geometry object was hit first.
			if (t > 0.0f && t_min > t)
			{
				t_min = t;
				hit_geom_index = i;
				intersect_point = tmp_intersect;
				normal = tmp_normal;
			}
		}

		if (hit_geom_index == -1)
		{
			intersections[path_index].t = -1.0f;
		}
		else
		{
			//The ray hits something
			intersections[path_index].t = t_min;
			intersections[path_index].materialId = geoms[hit_geom_index].materialid;
			intersections[path_index].surfaceNormal = normal;

			// shade the shadow
			pathSegment.color = glm::vec3(1);

		}
	}

}

__global__ void computeRayToLight(
	 int num_paths
	, ShadeableIntersection* shadeableIntersections
	, PathSegment* pathSegments
	, Material* materials
	, Geom* light
)
{
	int idx = blockIdx.x * blockDim.x + threadIdx.x;
	if (idx >= num_paths || pathSegments[idx].remainingBounces <= 0) return;

	// Take a ray directly to a random point on an emissive object

	// 1. Scatter ray from intersection point to sampled point on light
	thrust::default_random_engine rng = makeSeededRandomEngine(0, num_paths, pathSegments[idx].remainingBounces);
	thrust::uniform_real_distribution<float> u01(0, 1);
	glm::vec4 pLocal(u01(rng) - .5f, u01(rng) - .5f, 0, 1);
	glm::vec4 sampled_L = light->transform * pLocal;
	glm::vec3 intersect = pathSegments[idx].ray.origin;

	pathSegments[idx].ray.direction = glm::normalize(glm::vec3(sampled_L) - intersect);
	pathSegments[idx].ray.origin += 0.0001f * pathSegments[idx].ray.direction;

	
	// 2. Intersection test/ see if it's in shadow

	// 3. Shade the path segment

}


__global__ void shadeMaterialNaive(
	int iter
	, int num_paths
	, ShadeableIntersection* shadeableIntersections
	, PathSegment* pathSegments
	, Material* materials
)
{
	int idx = blockIdx.x * blockDim.x + threadIdx.x;
	if (idx >= num_paths || pathSegments[idx].remainingBounces <=0 ) return;

	ShadeableIntersection intersection = shadeableIntersections[idx];
	if (intersection.t > 0.0f) { // if the intersection exists...
	  // Set up the RNG
	  // LOOK: this is how you use thrust's RNG! Please look at
	  // makeSeededRandomEngine as well.
		thrust::default_random_engine rng = makeSeededRandomEngine(iter, idx, pathSegments[idx].remainingBounces);
		thrust::uniform_real_distribution<float> u01(0, 1);

		Material material = materials[intersection.materialId];
		glm::vec3 materialColor = material.color;

		// If the material indicates that the object was a light, "light" the ray
		if (material.emittance > 0.0f) {
			pathSegments[idx].color *= (materialColor * material.emittance);
			pathSegments[idx].remainingBounces = 0;
		}
		// Otherwise, do some pseudo-lighting computation. This is actually more
		// like what you would expect from shading in a rasterizer like OpenGL.
		// TODO: replace this! you should be able to start with basically a one-liner
		else {

			glm::vec3 intersect = pathSegments[idx].ray.origin + intersection.t * pathSegments[idx].ray.direction;
			scatterRay(pathSegments[idx], intersect, intersection.surfaceNormal, material, rng);
		}
		// If there was no intersection, color the ray black.
		// Lots of renderers use 4 channel color, RGBA, where A = alpha, often
		// used for opacity, in which case they can indicate "no opacity".
		// This can be useful for post-processing and image compositing.
	}
	else { // No intersection
		pathSegments[idx].color = glm::vec3(0.0f);
		pathSegments[idx].remainingBounces = 0;
	}

}


// LOOK: "fake" shader demonstrating what you might do with the info in
// a ShadeableIntersection, as well as how to use thrust's random number
// generator. Observe that since the thrust random number generator basically
// adds "noise" to the iteration, the image should start off noisy and get
// cleaner as more iterations are computed.
//
// Note that this shader does NOT do a BSDF evaluation!
// Your shaders should handle that - this can allow techniques such as
// bump mapping.
__global__ void shadeFakeMaterial(
	int iter
	, int num_paths
	, ShadeableIntersection* shadeableIntersections
	, PathSegment* pathSegments
	, Material* materials
)
{
	int idx = blockIdx.x * blockDim.x + threadIdx.x;
	if (idx < num_paths)
	{
		ShadeableIntersection intersection = shadeableIntersections[idx];
		if (intersection.t > 0.0f) { // if the intersection exists...
		  // Set up the RNG
		  // LOOK: this is how you use thrust's RNG! Please look at
		  // makeSeededRandomEngine as well.
			thrust::default_random_engine rng = makeSeededRandomEngine(iter, idx, 0);
			thrust::uniform_real_distribution<float> u01(0, 1);

			Material material = materials[intersection.materialId];
			glm::vec3 materialColor = material.color;

			// If the material indicates that the object was a light, "light" the ray
			if (material.emittance > 0.0f) {
				pathSegments[idx].color *= (materialColor * material.emittance);
			}
			// Otherwise, do some pseudo-lighting computation. This is actually more
			// like what you would expect from shading in a rasterizer like OpenGL.
			// TODO: replace this! you should be able to start with basically a one-liner
			else {
				float lightTerm = glm::dot(intersection.surfaceNormal, glm::vec3(0.0f, 1.0f, 0.0f));



				pathSegments[idx].color *= (materialColor * lightTerm) * 0.3f + ((1.0f - intersection.t * 0.02f) * materialColor) * 0.7f;
				//  pathSegments[idx].color *= u01(rng); // apply some noise because why not
			}
			// If there was no intersection, color the ray black.
			// Lots of renderers use 4 channel color, RGBA, where A = alpha, often
			// used for opacity, in which case they can indicate "no opacity".
			// This can be useful for post-processing and image compositing.
		}
		else {
			pathSegments[idx].color = glm::vec3(0.0f);
		}
	}
}

// Add the current iteration's output to the overall image
__global__ void finalGather(int nPaths, glm::vec3* image, PathSegment* iterationPaths)
{
	int index = (blockIdx.x * blockDim.x) + threadIdx.x;

	if (index < nPaths)
	{
		PathSegment iterationPath = iterationPaths[index];
		image[iterationPath.pixelIndex] += iterationPath.color;
	}
}

/**
 * Wrapper for the __global__ call that sets up the kernel calls and does a ton
 * of memory management
 */

struct check_completion {
	__host__ __device__
	bool operator () (const PathSegment& seg) {
		return seg.remainingBounces > 0;
	}
};

struct material_sort {
	__host__ __device__
		bool operator() (const ShadeableIntersection& lhs, const ShadeableIntersection& rhs) {
		return lhs.materialId < rhs.materialId;
	}
};


void pathtrace(uchar4* pbo, int frame, int iter) {
	const int traceDepth = hst_scene->state.traceDepth;
	const Camera& cam = hst_scene->state.camera;
	const int pixelcount = cam.resolution.x * cam.resolution.y;

	// 2D block for generating ray from camera
	const dim3 blockSize2d(8, 8);
	const dim3 blocksPerGrid2d(
		(cam.resolution.x + blockSize2d.x - 1) / blockSize2d.x,
		(cam.resolution.y + blockSize2d.y - 1) / blockSize2d.y);

	// 1D block for path tracing
	const int blockSize1d = 128;

	///////////////////////////////////////////////////////////////////////////

	// Recap:
	// * Initialize array of path rays (using rays that come out of the camera)
	//   * You can pass the Camera object to that kernel.
	//   * Each path ray must carry at minimum a (ray, color) pair,
	//   * where color starts as the multiplicative identity, white = (1, 1, 1).
	//   * This has already been done for you.
	// * For each depth:
	//   * Compute an intersection in the scene for each path ray.
	//     A very naive version of this has been implemented for you, but feel
	//     free to add more primitives and/or a better algorithm.
	//     Currently, intersection distance is recorded as a parametric distance,
	//     t, or a "distance along the ray." t = -1.0 indicates no intersection.
	//     * Color is attenuated (multiplied) by reflections off of any object
	//   * TODO: Stream compact away all of the terminated paths.
	//     You may use either your implementation or `thrust::remove_if` or its
	//     cousins.
	//     * Note that you can't really use a 2D kernel launch any more - switch
	//       to 1D.
	//   * TODO: Shade the rays that intersected something or didn't bottom out.
	//     That is, color the ray by performing a color computation according
	//     to the shader, then generate a new ray to continue the ray path.
	//     We recommend just updating the ray's PathSegment in place.
	//     Note that this step may come before or after stream compaction,
	//     since some shaders you write may also cause a path to terminate.
	// * Finally, add this iteration's results to the image. This has been done
	//   for you.

	// TODO: perform one iteration of path tracing

	if (b_useFirstCache && iter > 0) {
		// Use cached dev_paths
		hipMemcpy(dev_paths, dev_first_paths, pixelcount * sizeof(PathSegment), hipMemcpyDeviceToDevice);
	}
	else { // first iteration
		generateRayFromCamera << <blocksPerGrid2d, blockSize2d >> > (cam, iter, traceDepth, dev_paths);
		// Cache camera rays
		if (b_useFirstCache)
			hipMemcpy(dev_first_paths, dev_paths, pixelcount * sizeof(PathSegment), hipMemcpyDeviceToDevice);
	}
	checkCUDAError("generate camera ray");

	int depth = 0;
	PathSegment* dev_path_end = dev_paths + pixelcount;

	int num_paths = dev_path_end - dev_paths;

	//timer.startGpuTimer();

	// --- PathSegment Tracing Stage ---
	// Shoot ray into scene, bounce between objects, push shading chunks
	while (true) {

#if USE_COMPACTION
			//	PathSegment* new_end = thrust::remove_if(thrust::device, dev_paths, dev_paths + num_paths, check_completion());
			PathSegment* new_end = thrust::partition(thrust::device, dev_paths, dev_paths + num_paths, check_completion());
			num_paths = new_end - dev_paths;
			if (num_paths < 1) break;
#endif

		// clean shading chunks
		hipMemset(dev_intersections, 0, pixelcount * sizeof(ShadeableIntersection));

		// tracing
		dim3 numblocksPathSegmentTracing = (num_paths + blockSize1d - 1) / blockSize1d;

		// Reuse first intersections for iter > 0
		if (b_useFirstCache && iter > 0 && depth == 0) {
			hipMemcpy(dev_intersections, dev_first_intersections, num_paths * sizeof(ShadeableIntersection), hipMemcpyDeviceToDevice);
		}
		else
		{
			computeIntersections << <numblocksPathSegmentTracing, blockSize1d >> > (
				depth
				, num_paths
				, dev_paths
				, dev_geoms
				, hst_scene->geoms.size()
				, dev_intersections
				);

		}
		checkCUDAError("trace one bounce");
		hipDeviceSynchronize();
		// Cache first dev_intersections
		if (b_useFirstCache && iter == 0 && depth == 0) {
			hipMemcpy(dev_first_intersections, dev_intersections, num_paths * sizeof(ShadeableIntersection), hipMemcpyDeviceToDevice);
		}

		depth++;

		// TODO:
		// --- Shading Stage ---
		// Shade path segments based on intersections and generate new rays by
	  // evaluating the BSDF.
	  // Start off with just a big kernel that handles all the different
	  // materials you have in the scenefile.
	  // TODO: compare between directly shading the path segments and shading
	  // path segments that have been reshuffled to be contiguous in memory.

#if SORT_MATERIAL
		thrust::device_ptr<ShadeableIntersection> intersection_for_mID(dev_intersections);
		thrust::device_ptr<PathSegment> thrust_paths(dev_paths);

		thrust::sort_by_key(intersection_for_mID, intersection_for_mID + num_paths, thrust_paths, material_sort());
#endif

		shadeMaterialNaive << <numblocksPathSegmentTracing, blockSize1d >> > (
			iter,
			num_paths,
			dev_intersections,
			dev_paths,
			dev_materials
			);

		if (depth > traceDepth)
			break;; // TODO: should be based off stream compaction results.
	}

	num_paths = dev_path_end - dev_paths;

#if DIRECT_LIGHT
	if (num_paths > 1) {
		dim3 numblocksPathSegmentTracing = (num_paths + blockSize1d - 1) / blockSize1d;

		computeIntersections << <numblocksPathSegmentTracing, blockSize1d >> > (
			depth
			, num_paths
			, dev_paths
			, dev_geoms
			, hst_scene->geoms.size()
			, dev_intersections
			);

		computeRayToLight << <numblocksPathSegmentTracing, blockSize1d >> > (
			num_paths,
			dev_intersections,
			dev_paths,
			dev_materials,
			dev_geoms
			);

		// Light must be the first one in dev_geoms
		shadowTest << <numblocksPathSegmentTracing, blockSize1d >> > (
			num_paths,
			dev_intersections,
			dev_geoms,
			hst_scene->geoms.size(),
			dev_paths
			);
	}
#endif // DIRECT_LIGHT

	// Assemble this iteration and apply it to the image
	dim3 numBlocksPixels = (pixelcount + blockSize1d - 1) / blockSize1d;
	finalGather << <numBlocksPixels, blockSize1d >> > (num_paths, dev_image, dev_paths);


//	timer.endGpuTimer();
//	std::cout << timer.getGpuElapsedTimeForPreviousOperation() << std::endl;
	///////////////////////////////////////////////////////////////////////////

	// Send results to OpenGL buffer for rendering
	sendImageToPBO << <blocksPerGrid2d, blockSize2d >> > (pbo, cam.resolution, iter, dev_image);

	// Retrieve image from GPU
	hipMemcpy(hst_scene->state.image.data(), dev_image,
		pixelcount * sizeof(glm::vec3), hipMemcpyDeviceToHost);

	checkCUDAError("pathtrace");
}
