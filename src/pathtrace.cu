#include "hip/hip_runtime.h"
#include <cstdio>
#include <hip/hip_runtime.h>
#include <cmath>
#include <thrust/execution_policy.h>
#include <thrust/random.h>
#include <thrust/remove.h>
#include <thrust/partition.h>

#include "sceneStructs.h"
#include "scene.h"
#include "glm/glm.hpp"
#include "glm/gtx/norm.hpp"
#include "utilities.h"
#include "pathtrace.h"
#include "intersections.h"
#include "interactions.h"

#define ERRORCHECK 1

// Part 1
#define TOGGLESTREAM true
#define TOGGLESORT true
#define TOGGLECACHE true

// Part 2
#define DEPTHOFFIELD false

// Bonus
//#define DIRECTLIGHTING true - relocated to scene.h

#define FILENAME (strrchr(__FILE__, '/') ? strrchr(__FILE__, '/') + 1 : __FILE__)
#define checkCUDAError(msg) checkCUDAErrorFn(msg, FILENAME, __LINE__)
void checkCUDAErrorFn(const char *msg, const char *file, int line) {
#if ERRORCHECK
	hipDeviceSynchronize();
	hipError_t err = hipGetLastError();
	if (hipSuccess == err) {
		return;
	}

	fprintf(stderr, "CUDA error");
	if (file) {
		fprintf(stderr, " (%s:%d)", file, line);
	}
	fprintf(stderr, ": %s: %s\n", msg, hipGetErrorString(err));
#  ifdef _WIN32
	getchar();
#  endif
	exit(EXIT_FAILURE);
#endif
}

__host__ __device__
thrust::default_random_engine makeSeededRandomEngine(int iter, int index, int depth) {
	int h = utilhash((1 << 31) | (depth << 22) | iter) ^ utilhash(index);
	return thrust::default_random_engine(h);
}

//Kernel that writes the image to the OpenGL PBO directly.
__global__ void sendImageToPBO(uchar4* pbo, glm::ivec2 resolution,
	int iter, glm::vec3* image) {
	int x = (blockIdx.x * blockDim.x) + threadIdx.x;
	int y = (blockIdx.y * blockDim.y) + threadIdx.y;

	if (x < resolution.x && y < resolution.y) {
		int index = x + (y * resolution.x);
		glm::vec3 pix = image[index];

		glm::ivec3 color;
		color.x = glm::clamp((int)(pix.x / iter * 255.0), 0, 255);
		color.y = glm::clamp((int)(pix.y / iter * 255.0), 0, 255);
		color.z = glm::clamp((int)(pix.z / iter * 255.0), 0, 255);

		// Each thread writes one pixel location in the texture (textel)
		pbo[index].w = 0;
		pbo[index].x = color.x;
		pbo[index].y = color.y;
		pbo[index].z = color.z;
	}
}

static Scene * hst_scene = NULL;
static glm::vec3 * dev_image = NULL;
static Geom * dev_geoms = NULL;
#if DIRECTLIGHTING
static Geom * dev_lights = NULL;
static int numLights = 0;
#endif
static Material * dev_materials = NULL;
static PathSegment * dev_paths = NULL;
static ShadeableIntersection * dev_intersections = NULL;
static ShadeableIntersection * dev_cache_first = NULL;
// TODO: static variables for device memory, any extra info you need, etc
// ...

void pathtraceInit(Scene *scene) {
	hst_scene = scene;
	const Camera &cam = hst_scene->state.camera;
	const int pixelcount = cam.resolution.x * cam.resolution.y;

	hipMalloc(&dev_image, pixelcount * sizeof(glm::vec3));
	hipMemset(dev_image, 0, pixelcount * sizeof(glm::vec3));

	hipMalloc(&dev_paths, pixelcount * sizeof(PathSegment));

	hipMalloc(&dev_geoms, scene->geoms.size() * sizeof(Geom));
	hipMemcpy(dev_geoms, scene->geoms.data(), scene->geoms.size() * sizeof(Geom), hipMemcpyHostToDevice);

#if DIRECTLIGHTING
	hipMalloc(&dev_lights, scene->lights.size() * sizeof(Geom));
	hipMemcpy(dev_lights, scene->lights.data(), scene->lights.size() * sizeof(Geom), hipMemcpyHostToDevice);
	numLights = scene->lights.size();
#endif

	hipMalloc(&dev_materials, scene->materials.size() * sizeof(Material));
	hipMemcpy(dev_materials, scene->materials.data(), scene->materials.size() * sizeof(Material), hipMemcpyHostToDevice);

	hipMalloc(&dev_intersections, pixelcount * sizeof(ShadeableIntersection));
	hipMemset(dev_intersections, 0, pixelcount * sizeof(ShadeableIntersection));
	
	hipMalloc(&dev_cache_first, pixelcount * sizeof(ShadeableIntersection));
	hipMemset(dev_cache_first, 0, pixelcount * sizeof(ShadeableIntersection));

	// TODO: initialize any extra device memeory you need

	checkCUDAError("pathtraceInit");
}

void pathtraceFree() {
	hipFree(dev_image);  // no-op if dev_image is null
	hipFree(dev_paths);
	hipFree(dev_geoms);
	hipFree(dev_materials);
	hipFree(dev_intersections);
	// TODO: clean up any extra device memory you created
	hipFree(dev_cache_first);

	checkCUDAError("pathtraceFree");
}

/**
* Generate PathSegments with rays from the camera through the screen into the
* scene, which is the first bounce of rays.
*
* Antialiasing - add rays for sub-pixel sampling
* motion blur - jitter rays "in time"
* lens effect - jitter ray origin positions based on a lens
*/
__global__ void generateRayFromCamera(Camera cam, int iter, int traceDepth, PathSegment* pathSegments)
{
	int x = (blockIdx.x * blockDim.x) + threadIdx.x;
	int y = (blockIdx.y * blockDim.y) + threadIdx.y;

	if (x < cam.resolution.x && y < cam.resolution.y) {
		int index = x + (y * cam.resolution.x);
		PathSegment & segment = pathSegments[index];

		segment.ray.origin = cam.position;
		segment.color = glm::vec3(1.0f, 1.0f, 1.0f);

		// TODO: implement antialiasing by jittering the ray
		segment.ray.direction = glm::normalize(cam.view
			- cam.right * cam.pixelLength.x * ((float)x - (float)cam.resolution.x * 0.5f)
			- cam.up * cam.pixelLength.y * ((float)y - (float)cam.resolution.y * 0.5f)
		);

		segment.pixelIndex = index;
		segment.remainingBounces = traceDepth;

#if DEPTHOFFIELD
		// ConcentricSampleDisk func: (with focalDistance = 6.5, lensRadius = 1.0)
		glm::vec2 xyCoord = glm::vec2(float(x) / float(cam.resolution.x), 
									  float(y) / float(cam.resolution.y));

		// Randomness for sample
		thrust::default_random_engine rng = makeSeededRandomEngine(iter, index, 0);
		thrust::uniform_real_distribution<float> u01(0, 1);
		
		// Sample point on lens
		glm::vec2 sample = glm::vec2(u01(rng), u01(rng));
		glm::vec3 pLens = 0.5f * squareToDiskConcentric(&sample);

		// Compute point on plane of focus
		glm::vec3 focus = cam.position + segment.ray.direction * 6.82f;
		glm::vec3 aperture = cam.position + cam.up * pLens.y + cam.right * pLens.x;

		segment.ray.direction = focus - aperture;
		segment.ray.origin = aperture;

#endif // DEPTHOFFIELD

	}
}

// TODO:
// computeIntersections handles generating ray intersections ONLY.
// Generating new rays is handled in your shader(s).
// Feel free to modify the code below.
__global__ void computeIntersections(
	int depth
	, int num_paths
	, PathSegment * pathSegments
	, Geom * geoms
	, int geoms_size
	, ShadeableIntersection * intersections
)
{
	int path_index = blockIdx.x * blockDim.x + threadIdx.x;

	if (path_index < num_paths)
	{
		PathSegment pathSegment = pathSegments[path_index];

		float t;
		glm::vec3 intersect_point;
		glm::vec3 normal;
		float t_min = FLT_MAX;
		int hit_geom_index = -1;
		bool outside = true;

		glm::vec3 tmp_intersect;
		glm::vec3 tmp_normal;

		// naive parse through global geoms

		for (int i = 0; i < geoms_size; i++)
		{
			Geom & geom = geoms[i];

			if (geom.type == CUBE)
			{
				t = boxIntersectionTest(geom, pathSegment.ray, tmp_intersect, tmp_normal, outside);
			}
			else if (geom.type == SPHERE)
			{
				t = sphereIntersectionTest(geom, pathSegment.ray, tmp_intersect, tmp_normal, outside);
			}
			// TODO: add more intersection tests here... triangle? metaball? CSG?
			else if (geom.type == CAPPY)
			{
				t = cappyIntersectionTest(geom, pathSegment.ray, tmp_intersect, tmp_normal, outside);
			}
			else if (geom.type == DUCK)
			{
				t = duckIntersectionTest(geom, pathSegment.ray, tmp_intersect, tmp_normal, outside);
			}
			
			// Compute the minimum t from the intersection tests to determine what
			// scene geometry object was hit first.
			if (t > 0.0f && t_min > t)
			{
				t_min = t;
				hit_geom_index = i;
				intersect_point = tmp_intersect;
				normal = tmp_normal;
			}
		}

		if (hit_geom_index == -1)
		{
			intersections[path_index].t = -1.0f;
		}
		else
		{
			//The ray hits something
			intersections[path_index].t = t_min;
			intersections[path_index].materialId = geoms[hit_geom_index].materialid;
			intersections[path_index].surfaceNormal = normal;
		}
	}
}


// LOOK: "fake" shader demonstrating what you might do with the info in
// a ShadeableIntersection, as well as how to use thrust's random number
// generator. Observe that since the thrust random number generator basically
// adds "noise" to the iteration, the image should start off noisy and get
// cleaner as more iterations are computed.
//
// Note that this shader does NOT do a BSDF evaluation!
// Your shaders should handle that - this can allow techniques such as
// bump mapping.
__global__ void shadeFakeMaterial(
	int iter
	, int num_paths
	, ShadeableIntersection * shadeableIntersections
	, PathSegment * pathSegments
	, Material * materials
)
{
	int idx = blockIdx.x * blockDim.x + threadIdx.x;
	if (idx < num_paths)
	{
		ShadeableIntersection intersection = shadeableIntersections[idx];
		if (intersection.t > 0.0f) { // if the intersection exists...
		  // Set up the RNG
		  // LOOK: this is how you use thrust's RNG! Please look at
		  // makeSeededRandomEngine as well.
			thrust::default_random_engine rng = makeSeededRandomEngine(iter, idx, 0);//pathSegments[idx].remainingBounces);
			thrust::uniform_real_distribution<float> u01(0, 1);

			Material material = materials[intersection.materialId];
			glm::vec3 materialColor = material.color;

			// If the material indicates that the object was a light, "light" the ray
			if (material.emittance > 0.0f) {
				pathSegments[idx].color *= (materialColor * material.emittance);
			}
			// Otherwise, do some pseudo-lighting computation. This is actually more
			// like what you would expect from shading in a rasterizer like OpenGL.
			// TODO: replace this! you should be able to start with basically a one-liner
			else {
				float lightTerm = glm::dot(intersection.surfaceNormal, glm::vec3(0.0f, 1.0f, 0.0f));
				pathSegments[idx].color *= (materialColor * lightTerm) * 0.3f + ((1.0f - intersection.t * 0.02f) * materialColor) * 0.7f;
				pathSegments[idx].color *= u01(rng); // apply some noise because why not
			}
			// If there was no intersection, color the ray black.
			// Lots of renderers use 4 channel color, RGBA, where A = alpha, often
			// used for opacity, in which case they can indicate "no opacity".
			// This can be useful for post-processing and image compositing.
		}
		else {
			pathSegments[idx].color = glm::vec3(0.0f);
		}
	}
}

// My material shader, based on a naive integrator
__global__ void shadeNaive(int iter,
	int num_paths,
	ShadeableIntersection * shadeableIntersections,
	PathSegment * pathSegments,
	Material * materials) {

	int idx = blockIdx.x * blockDim.x + threadIdx.x;
	if (idx < num_paths && pathSegments[idx].remainingBounces > 0)
	{
		ShadeableIntersection intersection = shadeableIntersections[idx];
		if (intersection.t > 0.0f) { // if the intersection exists...
		  // Set up the RNG
			thrust::default_random_engine rng = makeSeededRandomEngine(iter, idx, pathSegments[idx].remainingBounces);
			thrust::uniform_real_distribution<float> u01(0, 1);

			Material material = materials[intersection.materialId];
			glm::vec3 materialColor = material.color;

			// If the material indicates that the object was a light, "light" the ray
			if (material.emittance > 0.0f) {
				pathSegments[idx].color *= (materialColor * material.emittance); // plus?? light is additive?
				pathSegments[idx].remainingBounces = 0;
			}
			// Otherwise, do some pseudo-lighting computation. This is actually more
			// like what you would expect from shading in a rasterizer like OpenGL.
			else {
				scatterRay(pathSegments[idx],
					getPointOnRay(pathSegments[idx].ray, intersection.t),
					intersection.surfaceNormal, material, rng);
				if (!material.hasRefractive) {
					float lightTerm = glm::dot(glm::normalize(intersection.surfaceNormal), glm::normalize(pathSegments[idx].ray.direction));

					pathSegments[idx].color *= lightTerm * 1.37f;
				}//(materialColor * lightTerm) * 0.3f + ((1.0f - intersection.t * 0.02f) * materialColor) * 0.7f;
				//pathSegments[idx].color *= u01(rng); // apply some noise because why not
			}
			pathSegments[idx].remainingBounces--;
		}
		// If there was no intersection, color the ray black.
		else {
			pathSegments[idx].color = glm::vec3(0.0f);
			pathSegments[idx].remainingBounces = 0;
		}
	}
}

#if DIRECTLIGHTING
// BONUS: A direct lighting shader: takes a final ray directly to a random point 
// on an emissive object acting as a light source)

__global__ void shadeDirect(int iter,
	int num_paths,
	ShadeableIntersection * shadeableIntersections,
	PathSegment * pathSegments,
	Material * materials,
	Geom * lights, int numLights) {

	int idx = blockIdx.x * blockDim.x + threadIdx.x;
	if (idx < num_paths && pathSegments[idx].remainingBounces > 2 || 
							pathSegments[idx].remainingBounces == 1)
	{
		ShadeableIntersection intersection = shadeableIntersections[idx];
		if (intersection.t > 0.0f) { // if the intersection exists...
		  // Set up the RNG
			thrust::default_random_engine rng = makeSeededRandomEngine(iter, idx, pathSegments[idx].remainingBounces);
			thrust::uniform_real_distribution<float> u01(0, 1);

			Material material = materials[intersection.materialId];
			glm::vec3 materialColor = material.color;



			// If the material indicates that the object was a light, "light" the ray
			if (material.emittance > 0.0f) {
				pathSegments[idx].color *= (materialColor * material.emittance); // plus?? light is additive?
				pathSegments[idx].remainingBounces = 0;
			}
			// Otherwise, do some pseudo-lighting computation. This is actually more
			// like what you would expect from shading in a rasterizer like OpenGL.
			else {
				scatterRay(pathSegments[idx],
					getPointOnRay(pathSegments[idx].ray, intersection.t),
					intersection.surfaceNormal, material, rng);
				if (!material.hasRefractive) {
					float lightTerm = glm::dot(glm::normalize(intersection.surfaceNormal), glm::normalize(pathSegments[idx].ray.direction));

					pathSegments[idx].color *= lightTerm * 1.37f;
				}
			}
			pathSegments[idx].remainingBounces--;
		}
		// If there was no intersection, color the ray black.
		else {
			pathSegments[idx].color = glm::vec3(0.0f);
			pathSegments[idx].remainingBounces = 0;
		}

	// Direct Lighting component
	} else if (idx < num_paths && pathSegments[idx].remainingBounces == 2) {
		ShadeableIntersection intersection = shadeableIntersections[idx];
		if (intersection.t > 0.0f) { // if the intersection exists...
		  // Set up the RNG
			thrust::default_random_engine rng = makeSeededRandomEngine(iter, idx, pathSegments[idx].remainingBounces);
			thrust::uniform_real_distribution<float> u01(0, 1);

			Material material = materials[intersection.materialId];
			glm::vec3 materialColor = material.color;

			// If the material indicates that the object was a light, "light" the ray
			if (material.emittance > 0.0f) {
				pathSegments[idx].color *= (materialColor * material.emittance); // plus?? light is additive?
				pathSegments[idx].remainingBounces = 0;
			}
			// Otherwise, do some pseudo-lighting computation. This is actually more
			// like what you would expect from shading in a rasterizer like OpenGL.
			else {

				int randIdx = glm::min((int)std::floor(u01(rng) * numLights),
					numLights - 1);

				/*glm::vec3 colorLi = materials[lights[randIdx].materialid].color//;
				/*float emitt = * materials[lights[randIdx].materialid].emittance;*/

				
				// Sample a point on the light (squareplane form, will work for cube)
				glm::vec3 pd = glm::vec3((glm::vec2(u01(rng), u01(rng)) - 
										  glm::vec2(0.5)), 0.f);
				/*glm::vec3 lightNorm = glm::mat3(lights[randIdx].transform) * 
										glm::vec3(0.f, 0.f, 1.f);*/
				glm::vec3 lightPoint = glm::vec3(lights[randIdx].transform *
												glm::vec4(pd, 1.f));

				/*Ray rt;
				rt.origin = pathSegments->ray.origin;
				rt.direction = glm::normalize(lightPoint - rt.origin);*/

				// Still do this
				scatterRay(pathSegments[idx],
					getPointOnRay(pathSegments[idx].ray, intersection.t),
					intersection.surfaceNormal, material, rng);

				// But then change outgoing ray to point directly at the light, 
				// handle intersections in next pass
				pathSegments[idx].ray.direction = glm::normalize(lightPoint - 
													pathSegments[idx].ray.origin);

				if (!material.hasRefractive) {
					float lightTerm = glm::dot(glm::normalize(intersection.surfaceNormal), glm::normalize(pathSegments[idx].ray.direction));

					pathSegments[idx].color *= lightTerm;
				}
			}
			pathSegments[idx].remainingBounces--;
		}
		// If there was no intersection, color the ray black.
		else {
			pathSegments[idx].color = glm::vec3(0.0f);
			pathSegments[idx].remainingBounces = 0;
		}
	}
}
#endif

// Add the current iteration's output to the overall image
__global__ void finalGather(int nPaths, glm::vec3 * image, PathSegment * iterationPaths)
{
	int index = (blockIdx.x * blockDim.x) + threadIdx.x;

	if (index < nPaths)
	{
		PathSegment iterationPath = iterationPaths[index];
		image[iterationPath.pixelIndex] += iterationPath.color;
	}
}

// Used in thrust partition, check if these paths are complete
struct thrustHelper {
	__host__ __device__ bool operator()(const PathSegment& pathSegment) {
		return pathSegment.remainingBounces > 0;
	}
};

// Uses the material ids to compare adjacent materials
struct sortMaterial {
	__host__ __device__ bool operator()(
						const ShadeableIntersection& firstIntersection,
						const ShadeableIntersection& secondIntersection) {
		return firstIntersection.materialId < secondIntersection.materialId;
	}
};

/**
 * Wrapper for the __global__ call that sets up the kernel calls and does a ton
 * of memory management
 */
void pathtrace(uchar4 *pbo, int frame, int iter) {
	const int traceDepth = hst_scene->state.traceDepth;
	const Camera &cam = hst_scene->state.camera;
	const int pixelcount = cam.resolution.x * cam.resolution.y;

	// 2D block for generating ray from camera
	const dim3 blockSize2d(8, 8);
	const dim3 blocksPerGrid2d(
		(cam.resolution.x + blockSize2d.x - 1) / blockSize2d.x,
		(cam.resolution.y + blockSize2d.y - 1) / blockSize2d.y);

	// 1D block for path tracing
	const int blockSize1d = 128;

	///////////////////////////////////////////////////////////////////////////

	// Recap:
	// * Initialize array of path rays (using rays that come out of the camera)
	//   * You can pass the Camera object to that kernel.
	//   * Each path ray must carry at minimum a (ray, color) pair,
	//   * where color starts as the multiplicative identity, white = (1, 1, 1).
	//   * This has already been done for you.
	// * For each depth:
	//   * Compute an intersection in the scene for each path ray.
	//     A very naive version of this has been implemented for you, but feel
	//     free to add more primitives and/or a better algorithm.
	//     Currently, intersection distance is recorded as a parametric distance,
	//     t, or a "distance along the ray." t = -1.0 indicates no intersection.
	//     * Color is attenuated (multiplied) by reflections off of any object
	//   * TODO: Stream compact away all of the terminated paths.
	//     You may use either your implementation or `thrust::remove_if` or its
	//     cousins.
	//     * Note that you can't really use a 2D kernel launch any more - switch
	//       to 1D.
	//   * TODO: Shade the rays that intersected something or didn't bottom out.
	//     That is, color the ray by performing a color computation according
	//     to the shader, then generate a new ray to continue the ray path.
	//     We recommend just updating the ray's PathSegment in place.
	//     Note that this step may come before or after stream compaction,
	//     since some shaders you write may also cause a path to terminate.
	// * Finally, add this iteration's results to the image. This has been done
	//   for you.

	// TODO: perform one iteration of path tracing

	generateRayFromCamera << <blocksPerGrid2d, blockSize2d >> > (cam, iter, traceDepth, dev_paths);
	checkCUDAError("generate camera ray");

	int depth = 0;
	PathSegment* dev_path_end = dev_paths + pixelcount;
	int num_paths = dev_path_end - dev_paths;
	// Wanted to retain the currently remaining number of paths to pass into 
	// shadeNaive, but need the following for finalGather and 
	// numblocksPathSegmentTracing to draw correctly
	int num_paths_og = num_paths;

	// --- PathSegment Tracing Stage ---
	// Shoot ray into scene, bounce between objects, push shading chunks

	bool iterationComplete = false;
	while (!iterationComplete) {

		// clean shading chunks
		hipMemset(dev_intersections, 0, pixelcount * sizeof(ShadeableIntersection));

		// tracing
		dim3 numblocksPathSegmentTracing = (num_paths_og + blockSize1d - 1) / blockSize1d;
		
#if !TOGGLECACHE
		// Original code
		computeIntersections << <numblocksPathSegmentTracing, blockSize1d >> > (
			depth, num_paths, dev_paths, dev_geoms,
			hst_scene->geoms.size(), dev_intersections);
		checkCUDAError("trace one bounce");
#else
		// For the first iteration's initial intersection test
		if (iter == 1 && depth == 0) {
			// trace
			computeIntersections << <numblocksPathSegmentTracing, blockSize1d >> > (
				depth, num_paths, dev_paths, dev_geoms,
				hst_scene->geoms.size(), dev_cache_first);
			checkCUDAError("trace first bounce");
			
			// copy result to dev_intersections
			hipMemcpy(dev_intersections, dev_cache_first,
				pixelcount * sizeof(ShadeableIntersection), 
				hipMemcpyDeviceToDevice);

		// For all other iterations' initial intersections test
		} else if (depth == 0) {
			hipMemcpy(dev_intersections, dev_cache_first,
				pixelcount * sizeof(ShadeableIntersection),
				hipMemcpyDeviceToDevice);

		// Other bounces
		} else {
			computeIntersections << <numblocksPathSegmentTracing, blockSize1d >> > (
				depth, num_paths, dev_paths, dev_geoms,
				hst_scene->geoms.size(), dev_intersections);
			checkCUDAError("trace one bounce");
		}
#endif //TOGGLECACHE
		hipDeviceSynchronize();
		depth++;


		// Making intersections contiguous in memory by material type
#if TOGGLESORT
		thrust::sort_by_key(thrust::device, dev_intersections,
			dev_intersections + num_paths, dev_paths, sortMaterial());
#endif

		// TODO:
		// --- Shading Stage ---
		// Shade path segments based on intersections and generate new rays by
	  // evaluating the BSDF.
	  // Start off with just a big kernel that handles all the different
	  // materials you have in the scenefile.
	  // TODO: compare between directly shading the path segments and shading
	  // path segments that have been reshuffled to be contiguous in memory.

#if DIRECTLIGHTING
		// A direct lighting shader: Forces the last ray bounce to aim at a light
		shadeDirect << <numblocksPathSegmentTracing, blockSize1d >> > (
			iter,
			num_paths,
			dev_intersections,
			dev_paths,
			dev_materials,
			dev_lights, numLights);
#else
		// My shader based partially on my 461 naive-integrator and shadeFakeMaterial
		shadeNaive << <numblocksPathSegmentTracing, blockSize1d >> > (
			iter,
			num_paths,
			dev_intersections,
			dev_paths,
			dev_materials);
#endif

#if TOGGLESTREAM
		// Partitions paths, updates num, and checks if more iterations are needed
		dev_path_end = thrust::partition(thrust::device, dev_paths, dev_paths + num_paths, thrustHelper());
		num_paths = dev_path_end - dev_paths;
		iterationComplete = num_paths <= 0;
#endif // TOGGLESTREAM

		iterationComplete = iterationComplete || (depth == traceDepth); // Stop the iterations based off stream compaction results.
	}

	// Assemble this iteration and apply it to the image
	dim3 numBlocksPixels = (pixelcount + blockSize1d - 1) / blockSize1d;
	finalGather << <numBlocksPixels, blockSize1d >> > (num_paths_og, dev_image, dev_paths);

	///////////////////////////////////////////////////////////////////////////

	// Send results to OpenGL buffer for rendering
	sendImageToPBO << <blocksPerGrid2d, blockSize2d >> > (pbo, cam.resolution, iter, dev_image);

	// Retrieve image from GPU
	hipMemcpy(hst_scene->state.image.data(), dev_image,
		pixelcount * sizeof(glm::vec3), hipMemcpyDeviceToHost);

	checkCUDAError("pathtrace");

}
