#include "hip/hip_runtime.h"
#include <cstdio>
#include <hip/hip_runtime.h>
#include <cmath>

#include <thrust/execution_policy.h>
#include <thrust/random.h>
#include <thrust/remove.h>
#include <thrust/partition.h>
#include <thrust/device_ptr.h>

#include "glm/glm.hpp"
#include "glm/gtx/norm.hpp"
#include <glm/gtc/matrix_inverse.hpp>
#include <glm/gtc/matrix_transform.hpp>

#include "sceneStructs.h"
#include "scene.h"
#include "utilities.h"
#include "pathtrace.h"
#include "intersections.h"
#include "interactions.h"

#define ERRORCHECK 1

#define COMPACT 0 // print rays left
#define MATERIAL_SORT 1
#define CACHE_FIRST 1
#define ANTIALIAS 0
#define MOTION 0
#define DOF 0

#define FILENAME (strrchr(__FILE__, '/') ? strrchr(__FILE__, '/') + 1 : __FILE__)
#define checkCUDAError(msg) checkCUDAErrorFn(msg, FILENAME, __LINE__)
void checkCUDAErrorFn(const char *msg, const char *file, int line) {
#if ERRORCHECK
    hipDeviceSynchronize();
    hipError_t err = hipGetLastError();
    if (hipSuccess == err) {
        return;
    }

    fprintf(stderr, "CUDA error");
    if (file) {
        fprintf(stderr, " (%s:%d)", file, line);
    }
    fprintf(stderr, ": %s: %s\n", msg, hipGetErrorString(err));
#  ifdef _WIN32
    getchar();
#  endif
    exit(EXIT_FAILURE);
#endif
}

__host__ __device__
thrust::default_random_engine makeSeededRandomEngine(int iter, int index, int depth) {
    int h = utilhash((1 << 31) | (depth << 22) | iter) ^ utilhash(index);
    return thrust::default_random_engine(h);
}

//Kernel that writes the image to the OpenGL PBO directly.
__global__ void sendImageToPBO(uchar4* pbo, glm::ivec2 resolution,
        int iter, glm::vec3* image) {
    int x = (blockIdx.x * blockDim.x) + threadIdx.x;
    int y = (blockIdx.y * blockDim.y) + threadIdx.y;

    if (x < resolution.x && y < resolution.y) {
        int index = x + (y * resolution.x);
        glm::vec3 pix = image[index];

        glm::ivec3 color;
        color.x = glm::clamp((int) (pix.x / iter * 255.0), 0, 255);
        color.y = glm::clamp((int) (pix.y / iter * 255.0), 0, 255);
        color.z = glm::clamp((int) (pix.z / iter * 255.0), 0, 255);

        // Each thread writes one pixel location in the texture (textel)
        pbo[index].w = 0;
        pbo[index].x = color.x;
        pbo[index].y = color.y;
        pbo[index].z = color.z;
    }
}

static Scene * hst_scene = NULL;
static glm::vec3 * dev_image = NULL;
static Geom * dev_geoms = NULL;
static Material * dev_materials = NULL;
static PathSegment * dev_paths = NULL;
static ShadeableIntersection * dev_intersections = NULL;
// static variables for device memory, any extra info you need, etc
static ShadeableIntersection * dev_intersections_cache = NULL;

void pathtraceInit(Scene *scene) {
    hst_scene = scene;
    const Camera &cam = hst_scene->state.camera;
    const int pixelcount = cam.resolution.x * cam.resolution.y;

    hipMalloc(&dev_image, pixelcount * sizeof(glm::vec3));
    hipMemset(dev_image, 0, pixelcount * sizeof(glm::vec3));

  	hipMalloc(&dev_paths, pixelcount * sizeof(PathSegment));

  	hipMalloc(&dev_geoms, scene->geoms.size() * sizeof(Geom));
  	hipMemcpy(dev_geoms, scene->geoms.data(), scene->geoms.size() * sizeof(Geom), hipMemcpyHostToDevice);

  	hipMalloc(&dev_materials, scene->materials.size() * sizeof(Material));
  	hipMemcpy(dev_materials, scene->materials.data(), scene->materials.size() * sizeof(Material), hipMemcpyHostToDevice);

  	hipMalloc(&dev_intersections, pixelcount * sizeof(ShadeableIntersection));
  	hipMemset(dev_intersections, 0, pixelcount * sizeof(ShadeableIntersection));

    // initialize any extra device memeory you need
#if CACHE_FIRST
	hipMalloc(&dev_intersections_cache, pixelcount * sizeof(ShadeableIntersection));
	hipMemset(dev_intersections_cache, 0, pixelcount * sizeof(ShadeableIntersection));
#endif // CACHE_FIRST


    checkCUDAError("pathtraceInit");
}

void pathtraceFree() {
    hipFree(dev_image);  // no-op if dev_image is null
  	hipFree(dev_paths);
  	hipFree(dev_geoms);
  	hipFree(dev_materials);
  	hipFree(dev_intersections);
    // clean up any extra device memory you created
	hipFree(dev_intersections_cache);

    checkCUDAError("pathtraceFree");
}

__device__ glm::vec3 sampleAperture(thrust::default_random_engine &rng) {
	thrust::uniform_real_distribution<float> udof(0.0f, 1.0f);
	glm::vec2 offset = 2.0f * glm::vec2(udof(rng), udof(rng)) - glm::vec2(1, 1);
	float theta, r;
	if (offset.x == 0 && offset.y == 0) {
		return glm::vec3(0.0f);
	}
	else if (glm::abs(offset.x) > glm::abs(offset.y)) {
		r = offset.x;
		theta = (PI / 4.0f) * (offset.y / offset.x);
	}
	else {
		r = offset.y;
		theta = (PI / 2.0f) - (PI / 4.0f) * (offset.x / offset.y);
	}
	float u = r * glm::cos(theta);
	float v = r * glm::sin(theta);
	return glm::vec3(u, v, 0.0f);
}

/**
* Generate PathSegments with rays from the camera through the screen into the
* scene, which is the first bounce of rays.
*
* Antialiasing - add rays for sub-pixel sampling
* motion blur - jitter rays "in time"
* lens effect - jitter ray origin positions based on a lens
*/
__global__ void generateRayFromCamera(Camera cam, int iter, int traceDepth, PathSegment* pathSegments)
{
	int x = (blockIdx.x * blockDim.x) + threadIdx.x;
	int y = (blockIdx.y * blockDim.y) + threadIdx.y;

	if (x < cam.resolution.x && y < cam.resolution.y) {
		int index = x + (y * cam.resolution.x);
		PathSegment & segment = pathSegments[index];

		segment.ray.origin = cam.position;
		segment.color = glm::vec3(1.0f, 1.0f, 1.0f);

		thrust::default_random_engine rng = makeSeededRandomEngine(iter, index, segment.remainingBounces);
#if ANTIALIAS
		thrust::uniform_real_distribution<float> u01(-1.0f, 1.0f);
		x += u01(rng);
		y += u01(rng);
#endif // ANTIALIAS

#if DOF
		glm::vec3 lens = cam.lensRadius * sampleAperture(rng);
		glm::vec3 focus = segment.ray.direction * ( cam.focalDistance / glm::abs(segment.ray.direction.z));
		segment.ray.origin += lens;
		segment.ray.direction = glm::normalize(focus - lens);
#endif // DOF


		segment.ray.direction = glm::normalize(cam.view
			- cam.right * cam.pixelLength.x * (x - (float)cam.resolution.x * 0.5f)
			- cam.up * cam.pixelLength.y * (y - (float)cam.resolution.y * 0.5f)
			);

		segment.pixelIndex = index;
		segment.remainingBounces = traceDepth;
	}
}

// computeIntersections handles generating ray intersections ONLY.
// Generating new rays is handled in your shader(s).
// Feel free to modify the code below.
__global__ void computeIntersections(
	int depth
	, int num_paths
	, PathSegment * pathSegments
	, Geom * geoms
	, int geoms_size
	, ShadeableIntersection * intersections
	)
{
	int path_index = blockIdx.x * blockDim.x + threadIdx.x;

	if (path_index < num_paths)
	{
		PathSegment pathSegment = pathSegments[path_index];

		float t;
		glm::vec3 intersect_point;
		glm::vec3 normal;
		float t_min = FLT_MAX;
		int hit_geom_index = -1;
		bool outside = true;

		glm::vec3 tmp_intersect;
		glm::vec3 tmp_normal;

		// naive parse through global geoms
		for (int i = 0; i < geoms_size; i++)
		{
			Geom & geom = geoms[i];
			if (geom.type == CUBE)
			{
				t = boxIntersectionTest(geom, pathSegment.ray, tmp_intersect, tmp_normal, outside);
			}
			else if (geom.type == SPHERE)
			{
				t = sphereIntersectionTest(geom, pathSegment.ray, tmp_intersect, tmp_normal, outside);
			}
			// TODO: add more intersection tests here... triangle? metaball? CSG?

			// Compute the minimum t from the intersection tests to determine what
			// scene geometry object was hit first.
			if (t > 0.0f && t_min > t)
			{
				t_min = t;
				hit_geom_index = i;
				intersect_point = tmp_intersect;
				normal = tmp_normal;
			}
		}

		if (hit_geom_index == -1)
		{
			intersections[path_index].t = -1.0f;
		}
		else
		{
			//The ray hits something
			intersections[path_index].t = t_min;
			intersections[path_index].materialId = geoms[hit_geom_index].materialid;
			intersections[path_index].surfaceNormal = normal;
		}
	}
}

__global__ void shadeMaterial(
	int iter
	, int num_paths
	, ShadeableIntersection * shadeableIntersections
	, PathSegment * pathSegments
	, Material * materials
)
{
	int idx = blockIdx.x * blockDim.x + threadIdx.x;
	if (idx >= num_paths) return;
	
	ShadeableIntersection intersection = shadeableIntersections[idx];
	if (intersection.t > 0.0f) { // if the intersection exists...
		// Set up the RNG
		thrust::default_random_engine rng = makeSeededRandomEngine(iter, idx, pathSegments[idx].remainingBounces);
		thrust::uniform_real_distribution<float> u01(0, 1);

		Material material = materials[intersection.materialId];
		glm::vec3 materialColor = material.color;

		// If the material indicates that the object was a light, "light" the ray
		if (material.emittance > 0.0f) {
			pathSegments[idx].color *= (materialColor * material.emittance);
			pathSegments[idx].remainingBounces = 0;
		}
		else {
			scatterRay(pathSegments[idx], getPointOnRay(pathSegments[idx].ray, intersection.t), intersection.surfaceNormal, material, rng);
			pathSegments[idx].remainingBounces--;
		}
		// If there was no intersection, color the ray black.
		// Lots of renderers use 4 channel color, RGBA, where A = alpha, often
		// used for opacity, in which case they can indicate "no opacity".
		// This can be useful for post-processing and image compositing.
	}
	else {
		pathSegments[idx].color = glm::vec3(0.0f);
		pathSegments[idx].remainingBounces = 0;
	}
	
}

// Add the current iteration's output to the overall image
__global__ void finalGather(int nPaths, glm::vec3 * image, PathSegment * iterationPaths)
{
	int index = (blockIdx.x * blockDim.x) + threadIdx.x;

	if (index < nPaths)
	{
		PathSegment iterationPath = iterationPaths[index];
		image[iterationPath.pixelIndex] += iterationPath.color;
	}
}


struct sort_materialID {
	__host__ __device__
		bool operator() (const ShadeableIntersection &si1, const ShadeableIntersection &si2) {
		return si1.materialId < si2.materialId;
	}
};


struct not_terminated {
	__host__ __device__
		bool operator() (const PathSegment &pathSegment) {
		return pathSegment.remainingBounces > 0;
	}
};

__device__ glm::mat4 buildTransformationMatrix(glm::vec3 translation, glm::vec3 rotation, glm::vec3 scale) {
	glm::mat4 translationMat = glm::translate(glm::mat4(), translation);
	glm::mat4 rotationMat = glm::rotate(glm::mat4(), rotation.x * (float)PI / 180, glm::vec3(1, 0, 0));
	rotationMat = rotationMat * glm::rotate(glm::mat4(), rotation.y * (float)PI / 180, glm::vec3(0, 1, 0));
	rotationMat = rotationMat * glm::rotate(glm::mat4(), rotation.z * (float)PI / 180, glm::vec3(0, 0, 1));
	glm::mat4 scaleMat = glm::scale(glm::mat4(), scale);
	return translationMat * rotationMat * scaleMat;
}

__global__ void motion(int n, Geom *geom, float dt) {
	int idx = blockIdx.x * blockDim.x + threadIdx.x;
	if (idx >= n) {
		return;
	}
	geom[idx].translation += geom[idx].velocity*dt;
	geom[idx].transform = buildTransformationMatrix(
		geom[idx].translation, geom[idx].rotation, geom[idx].scale);
	geom[idx].inverseTransform = glm::inverse(geom[idx].transform);
	geom[idx].invTranspose = glm::inverseTranspose(geom[idx].transform);
}


/**
 * Wrapper for the __global__ call that sets up the kernel calls and does a ton
 * of memory management
 */
void pathtrace(uchar4 *pbo, int frame, int iter) {
    const int traceDepth = hst_scene->state.traceDepth;
    const Camera &cam = hst_scene->state.camera;
    const int pixelcount = cam.resolution.x * cam.resolution.y;

	// 2D block for generating ray from camera
    const dim3 blockSize2d(8, 8);
    const dim3 blocksPerGrid2d(
            (cam.resolution.x + blockSize2d.x - 1) / blockSize2d.x,
            (cam.resolution.y + blockSize2d.y - 1) / blockSize2d.y);

	// 1D block for path tracing
	const int blockSize1d = 128;

    ///////////////////////////////////////////////////////////////////////////

    // Recap:
    // * Initialize array of path rays (using rays that come out of the camera)
    //   * You can pass the Camera object to that kernel.
    //   * Each path ray must carry at minimum a (ray, color) pair,
    //   * where color starts as the multiplicative identity, white = (1, 1, 1).
    //   * This has already been done for you.
    // * For each depth:
    //   * Compute an intersection in the scene for each path ray.
    //     A very naive version of this has been implemented for you, but feel
    //     free to add more primitives and/or a better algorithm.
    //     Currently, intersection distance is recorded as a parametric distance,
    //     t, or a "distance along the ray." t = -1.0 indicates no intersection.
    //     * Color is attenuated (multiplied) by reflections off of any object
    //   * TODO: Stream compact away all of the terminated paths.
    //     You may use either your implementation or `thrust::remove_if` or its
    //     cousins.
    //     * Note that you can't really use a 2D kernel launch any more - switch
    //       to 1D.
    //   * TODO: Shade the rays that intersected something or didn't bottom out.
    //     That is, color the ray by performing a color computation according
    //     to the shader, then generate a new ray to continue the ray path.
    //     We recommend just updating the ray's PathSegment in place.
    //     Note that this step may come before or after stream compaction,
    //     since some shaders you write may also cause a path to terminate.
    // * Finally, add this iteration's results to the image. This has been done
    //   for you.

    // perform one iteration of path tracing

	generateRayFromCamera <<<blocksPerGrid2d, blockSize2d >>>(cam, iter, traceDepth, dev_paths);
	checkCUDAError("generate camera ray");

	int depth = 0;
	PathSegment* dev_path_end = dev_paths + pixelcount;
	int num_paths = dev_path_end - dev_paths;

	// --- PathSegment Tracing Stage ---
	// Shoot ray into scene, bounce between objects, push shading chunks

	bool iterationComplete = false;

	while (!iterationComplete) {
	// clean shading chunks
	hipMemset(dev_intersections, 0, pixelcount * sizeof(ShadeableIntersection));

	// tracing
	dim3 numblocksPathSegmentTracing = (num_paths + blockSize1d - 1) / blockSize1d;
#if CACHE_FIRST && !ANTIALIAS && !MOTION
	if (depth == 0) {
		if (iter == 1) {
			computeIntersections << <numblocksPathSegmentTracing, blockSize1d >> > (
				depth
				, num_paths
				, dev_paths
				, dev_geoms
				, hst_scene->geoms.size()
				, dev_intersections_cache
				);
			checkCUDAError("trace one bounce");
			hipDeviceSynchronize();
		}
		hipMemcpy(dev_intersections, dev_intersections_cache, num_paths * sizeof(ShadeableIntersection), hipMemcpyDeviceToDevice);
	}
	else {
		computeIntersections << <numblocksPathSegmentTracing, blockSize1d >> > (
			depth
			, num_paths
			, dev_paths
			, dev_geoms
			, hst_scene->geoms.size()
			, dev_intersections
			);
		checkCUDAError("trace one bounce");
		hipDeviceSynchronize();
	}
#else
	computeIntersections << <numblocksPathSegmentTracing, blockSize1d >> > (
		depth
		, num_paths
		, dev_paths
		, dev_geoms
		, hst_scene->geoms.size()
		, dev_intersections
		);
	checkCUDAError("trace one bounce");
	hipDeviceSynchronize();
#endif

	depth++;

	// --- Shading Stage ---
	// Shade path segments based on intersections and generate new rays by
	// evaluating the BSDF.
	// Start off with just a big kernel that handles all the different
	// materials you have in the scenefile.
	// TODO: compare between directly shading the path segments and shading
	// path segments that have been reshuffled to be contiguous in memory.
#if MATERIAL_SORT
	thrust::device_ptr<ShadeableIntersection> thrust_intersections(dev_intersections);
	thrust::device_ptr<PathSegment> thrust_paths(dev_paths);
	thrust::sort_by_key(thrust_intersections, thrust_intersections, thrust_paths + num_paths, sort_materialID());
#endif // MATERIAL_SORT


	shadeMaterial<<<numblocksPathSegmentTracing, blockSize1d>>> (
		iter,
		num_paths,
		dev_intersections,
		dev_paths,
		dev_materials
		);

	PathSegment* path_end = thrust::partition(thrust::device, dev_paths, dev_paths + num_paths, not_terminated());
	num_paths = path_end - dev_paths;

#if COMPACT 
	std::cout << "Depth: " << depth << " Rays: " << num_paths   << "\n";
#endif // COMPACT

	iterationComplete = (depth >= traceDepth || num_paths <= 0); // based off stream compaction results.
	}

#if MOTION
	dim3 numblocksMotionBlue = (hst_scene->geoms.size() + blockSize1d - 1) / blockSize1d;
	motion << <numblocksMotionBlue, blockSize1d >> > (hst_scene->geoms.size(), dev_geoms, 0.005f);
	checkCUDAError("motion blur");
#endif // MOTION

	// Assemble this iteration and apply it to the image
	num_paths = dev_path_end - dev_paths;
	dim3 numBlocksPixels = (pixelcount + blockSize1d - 1) / blockSize1d;
	finalGather<<<numBlocksPixels, blockSize1d>>>(num_paths, dev_image, dev_paths);

    ///////////////////////////////////////////////////////////////////////////

    // Send results to OpenGL buffer for rendering
    sendImageToPBO<<<blocksPerGrid2d, blockSize2d>>>(pbo, cam.resolution, iter, dev_image);

    // Retrieve image from GPU
    hipMemcpy(hst_scene->state.image.data(), dev_image,
            pixelcount * sizeof(glm::vec3), hipMemcpyDeviceToHost);

    checkCUDAError("pathtrace");

}
