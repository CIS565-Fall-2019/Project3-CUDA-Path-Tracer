#include "hip/hip_runtime.h"
#include <cstdio>
#include <hip/hip_runtime.h>
#include <cmath>
#include <thrust/execution_policy.h>
#include <thrust/random.h>
#include <thrust/remove.h>
#include <thrust/count.h>
#include <thrust/partition.h>
#include <thrust/sort.h>
#include <glm/gtc/matrix_inverse.hpp>
#include <chrono>
#include <ctime>
#include <ratio>

#include "sceneStructs.h"
#include "scene.h"
#include "glm/glm.hpp"
#include "glm/gtx/norm.hpp"
#include "utilities.h"
#include "pathtrace.h"
#include "intersections.h"
#include "interactions.h"
#include "efficient.h"

#define RECORD_TIMING 0
#define ERRORCHECK 1
#define CACHEFIRSTBOUNCE 0
#define RAYSORT 0
#define MOTION_BLUR 0
#define STREAMCOMPACT_BY_THRUST 0
#define STREAM_COMPACT 1
#define ANTI_ALIASING 1

#define FILENAME (strrchr(__FILE__, '/') ? strrchr(__FILE__, '/') + 1 : __FILE__)
#define checkCUDAError(msg) checkCUDAErrorFn(msg, FILENAME, __LINE__)
void checkCUDAErrorFn(const char *msg, const char *file, int line) {
#if ERRORCHECK
    hipDeviceSynchronize();
    hipError_t err = hipGetLastError();
    if (hipSuccess == err) {
        return;
    }

    fprintf(stderr, "CUDA error");
    if (file) {
        fprintf(stderr, " (%s:%d)", file, line);
    }
    fprintf(stderr, ": %s: %s\n", msg, hipGetErrorString(err));
#  ifdef _WIN32
    getchar();
#  endif
    exit(EXIT_FAILURE);
#endif
}

__host__ __device__
thrust::default_random_engine makeSeededRandomEngine(int iter, int index, int depth) {
    int h = utilhash((1 << 31) | (depth << 22) | iter) ^ utilhash(index);
    return thrust::default_random_engine(h);
}

//Kernel that writes the image to the OpenGL PBO directly.
__global__ void sendImageToPBO(uchar4* pbo, glm::ivec2 resolution,
        int iter, glm::vec3* image) {
    int x = (blockIdx.x * blockDim.x) + threadIdx.x;
    int y = (blockIdx.y * blockDim.y) + threadIdx.y;

    if (x < resolution.x && y < resolution.y) {
        int index = x + (y * resolution.x);
        glm::vec3 pix = image[index];

        glm::ivec3 color;
        color.x = glm::clamp((int) (pix.x / iter * 255.0), 0, 255);
        color.y = glm::clamp((int) (pix.y / iter * 255.0), 0, 255);
        color.z = glm::clamp((int) (pix.z / iter * 255.0), 0, 255);

        // Each thread writes one pixel location in the texture (textel)
        pbo[index].w = 0;
        pbo[index].x = color.x;
        pbo[index].y = color.y;
        pbo[index].z = color.z;
    }
}

static Scene * hst_scene = NULL;
static glm::vec3 * dev_image = NULL;
static Geom * dev_geoms = NULL;
static Material * dev_materials = NULL;
static PathSegment * dev_paths = NULL;
static ShadeableIntersection * dev_intersections = NULL;
static ShadeableIntersection * dev_first_intersections = NULL;
int *dev_remaining_paths = NULL;

hipEvent_t start, stop;
// TODO: static variables for device memory, any extra info you need, etc
// ...

void pathtraceInit(Scene *scene) {
    hst_scene = scene;
    const Camera &cam = hst_scene->state.camera;
    const int pixelcount = cam.resolution.x * cam.resolution.y;

    hipMalloc(&dev_image, pixelcount * sizeof(glm::vec3));
    hipMemset(dev_image, 0, pixelcount * sizeof(glm::vec3));

  	hipMalloc(&dev_paths, pixelcount * sizeof(PathSegment));

  	hipMalloc(&dev_geoms, scene->geoms.size() * sizeof(Geom));
  	hipMemcpy(dev_geoms, scene->geoms.data(), scene->geoms.size() * sizeof(Geom), hipMemcpyHostToDevice);

  	hipMalloc(&dev_materials, scene->materials.size() * sizeof(Material));
  	hipMemcpy(dev_materials, scene->materials.data(), scene->materials.size() * sizeof(Material), hipMemcpyHostToDevice);

  	hipMalloc(&dev_intersections, pixelcount * sizeof(ShadeableIntersection));
  	hipMemset(dev_intersections, 0, pixelcount * sizeof(ShadeableIntersection));

	hipMalloc(&dev_first_intersections, pixelcount * sizeof(ShadeableIntersection));
	hipMemset(dev_first_intersections, 0, pixelcount * sizeof(ShadeableIntersection));

	hipMalloc(&dev_remaining_paths, pixelcount * sizeof(int));
	hipMemset(dev_remaining_paths, 0, pixelcount * sizeof(int));

    // TODO: initialize any extra device memeory you need

    checkCUDAError("pathtraceInit");
}

void pathtraceFree() {
    hipFree(dev_image);  // no-op if dev_image is null
  	hipFree(dev_paths);
  	hipFree(dev_geoms);
  	hipFree(dev_materials);
  	hipFree(dev_intersections);
	hipFree(dev_remaining_paths);
    // TODO: clean up any extra device memory you created

    checkCUDAError("pathtraceFree");
}

/**
* Generate PathSegments with rays from the camera through the screen into the
* scene, which is the first bounce of rays.
*
* Antialiasing - add rays for sub-pixel sampling
* motion blur - jitter rays "in time"
* lens effect - jitter ray origin positions based on a lens
*/
__global__ void generateRayFromCamera(Camera cam, int iter, int traceDepth, PathSegment* pathSegments,int *dev_remaining_paths)
{
	int x = (blockIdx.x * blockDim.x) + threadIdx.x;
	int y = (blockIdx.y * blockDim.y) + threadIdx.y;

	if (x < cam.resolution.x && y < cam.resolution.y) {
		int index = x + (y * cam.resolution.x);
		PathSegment & segment = pathSegments[index];

		segment.ray.origin = cam.position;
		segment.ray.origin = cam.position;
		segment.color = glm::vec3(1.0f, 1.0f, 1.0f);
		
		#if ANTI_ALIASING && !CACHEFIRSTBOUNCE
			thrust::default_random_engine rng1 = makeSeededRandomEngine(iter , x , y);
			thrust::default_random_engine rng2 = makeSeededRandomEngine(iter, y, x);
			thrust::uniform_real_distribution<float> u01(-0.5, 0.5);

			x += u01(rng1);
			y += u01(rng2);
		#endif

		// TODO: implement antialiasing by jittering the ray
		segment.ray.direction = glm::normalize(cam.view
			- cam.right * cam.pixelLength.x * ((float)x - (float)cam.resolution.x * 0.5f)
			- cam.up * cam.pixelLength.y * ((float)y - (float)cam.resolution.y * 0.5f)
			);

		segment.pixelIndex = index;
		segment.remainingBounces = traceDepth;
		dev_remaining_paths[index] = index;
		//segment.notDead = 1;
	}
}

// TODO:
// computeIntersections handles generating ray intersections ONLY.
// Generating new rays is handled in your shader(s).
// Feel free to modify the code below.
__global__ void computeIntersections(
	int depth
	, int num_paths
	, PathSegment * pathSegments
	, Geom * geoms
	, int geoms_size
	, ShadeableIntersection * intersections
	,int *dev_remaining_paths
	)
{
	int idx = blockIdx.x * blockDim.x + threadIdx.x;

	if (idx < num_paths)
	{
		int path_index = dev_remaining_paths[idx];
		PathSegment pathSegment = pathSegments[path_index];

		float t;
		glm::vec3 intersect_point;
		glm::vec3 normal;
		float t_min = FLT_MAX;
		int hit_geom_index = -1;
		bool outside = true;

		glm::vec3 tmp_intersect;
		glm::vec3 tmp_normal;

		// naive parse through global geoms

		for (int i = 0; i < geoms_size; i++)
		{
			Geom & geom = geoms[i];

			if (geom.type == CUBE)
			{
				t = boxIntersectionTest(geom, pathSegment.ray, tmp_intersect, tmp_normal, outside);
			}
			else if (geom.type == SPHERE)
			{
				// Motion Blur Code <<Not working>>
			//for (int i = 0; i < hst_scene->geoms.size(); i++) {
			//	float t = 0.01f;
			//	Geom & motion_geom = hst_scene->geoms[i];
			//	if (motion_geom.hasMotion) {
			//		motion_geom.translation += mot * t;
			//		motion_geom.transform = utilityCore::buildTransformationMatrix(motion_geom.translation, motion_geom.rotation, motion_geom.scale);
			//		motion_geom.inverseTransform = glm::inverse(motion_geom.transform);
			//		motion_geom.invTranspose = glm::inverseTranspose(motion_geom.transform);
			//		printf("Hello motion value for object ID: %d is: %0.02f, %0.02f , %0.02f\n", motion_geom.materialid, motion_geom.translation.x, motion_geom.translation.y, motion_geom.translation.z);
					//printf("Hello motion value for object ID: %d is: %0.02f, %0.02f , %0.02f\n", motion_geom.translation.x, motion_geom.translation.y, motion_geom.translation.z);
					//printf("Hello motion value for object ID: %d is: %0.02f, %0.02f , %0.02f\n", motion_geom.translation.x, motion_geom.translation.y, motion_geom.translation.z);
				
				t = sphereIntersectionTest(geom, pathSegment.ray, tmp_intersect, tmp_normal, outside);
				
				//hipMemcpy(dev_geoms, &(hst_scene->geoms)[0], hst_scene->geoms.size() * sizeof(Geom), hipMemcpyHostToDevice);


			}
			// TODO: add more intersection tests here... triangle? metaball? CSG?

			// Compute the minimum t from the intersection tests to determine what
			// scene geometry object was hit first.
			if (t > 0.0f && t_min > t)
			{
				t_min = t;
				hit_geom_index = i;
				intersect_point = tmp_intersect;
				normal = tmp_normal;
			}
		}

		if (hit_geom_index == -1)
		{
			intersections[path_index].t = -1.0f;
		}
		else
		{
			//The ray hits something
			intersections[path_index].t = t_min;
			intersections[path_index].materialId = geoms[hit_geom_index].materialid;
			intersections[path_index].surfaceNormal = normal;
			intersections[path_index].intersectionPoint = tmp_intersect;
		}
		//pathSegment.remainingBounces--;


	}
}

__global__ void shaderKernel(int iter,int numPaths,int depth, ShadeableIntersection* shadeableIntersections, Material* materials, PathSegment* pathsegments,int *dev_remaining_paths) {
	int idxx = threadIdx.x + blockIdx.x * blockDim.x;
	if (idxx >= numPaths || pathsegments[dev_remaining_paths[idxx]].remainingBounces < 0)
		return;
	
	int idx = dev_remaining_paths[idxx];
	ShadeableIntersection &intersection = shadeableIntersections[idx];
	Material &material = materials[intersection.materialId];
	PathSegment &pathsegment = pathsegments[idx];

	if (intersection.t >= 0.0f) {
		if (material.emittance > 0.0f) {
			pathsegment.color *= material.color * material.emittance;
			//printf("Hello \n");
			pathsegment.remainingBounces =0;

		}
		else {
				thrust::default_random_engine rng = makeSeededRandomEngine(iter, idxx, depth);
				//if (intersection.materialId == 4)
					scatterRay(pathsegment, intersection.intersectionPoint, intersection.surfaceNormal, material, rng);
				//else
				//	scatterRay(pathsegment, intersection.intersectionPoint, intersection.surfaceNormal, material, rng);
				pathsegment.remainingBounces--;
		}	
	}
	else {
		pathsegment.color = glm::vec3(0.0f);
		pathsegment.remainingBounces = 0;
		
	}

	#if STREAM_COMPACT
		if (pathsegment.remainingBounces<=0)
			dev_remaining_paths[idxx] = -1;
	#endif	
}


// LOOK: "fake" shader demonstrating what you might do with the info in
// a ShadeableIntersection, as well as how to use thrust's random number
// generator. Observe that since the thrust random number generator basically
// adds "noise" to the iteration, the image should start off noisy and get
// cleaner as more iterations are computed.
//
// Note that this shader does NOT do a BSDF evaluation!
// Your shaders should handle that - this can allow techniques such as
// bump mapping.
__global__ void shadeFakeMaterial (
  int iter
  , int num_paths
	, ShadeableIntersection * shadeableIntersections
	, PathSegment * pathSegments
	, Material * materials
	)
{
  int idx = blockIdx.x * blockDim.x + threadIdx.x;
  if (idx < num_paths)
  {
    ShadeableIntersection intersection = shadeableIntersections[idx];
    if (intersection.t > 0.0f) { // if the intersection exists...
      // Set up the RNG
      // LOOK: this is how you use thrust's RNG! Please look at
      // makeSeededRandomEngine as well.
      thrust::default_random_engine rng = makeSeededRandomEngine(iter, idx, 0);
      thrust::uniform_real_distribution<float> u01(0, 1);

      Material material = materials[intersection.materialId];
      glm::vec3 materialColor = material.color;

      // If the material indicates that the object was a light, "light" the ray
      if (material.emittance > 0.0f) {
        pathSegments[idx].color *= (materialColor * material.emittance);
      }
      // Otherwise, do some pseudo-lighting computation. This is actually more
      // like what you would expect from shading in a rasterizer like OpenGL.
      // TODO: replace this! you should be able to start with basically a one-liner
      else {
        float lightTerm = glm::dot(intersection.surfaceNormal, glm::vec3(0.0f, 1.0f, 0.0f));
        pathSegments[idx].color *= (materialColor * lightTerm) * 0.3f + ((1.0f - intersection.t * 0.02f) * materialColor) * 0.7f;
        pathSegments[idx].color *= u01(rng); // apply some noise because why not
      }
    // If there was no intersection, color the ray black.
    // Lots of renderers use 4 channel color, RGBA, where A = alpha, often
    // used for opacity, in which case they can indicate "no opacity".
    // This can be useful for post-processing and image compositing.
    } else {
      pathSegments[idx].color = glm::vec3(0.0f);
    }
  }
}

// Add the current iteration's output to the overall image
__global__ void finalGather(int nPaths, glm::vec3 * image, PathSegment * iterationPaths)
{
	int index = (blockIdx.x * blockDim.x) + threadIdx.x;

	if (index < nPaths)
	{
		PathSegment iterationPath = iterationPaths[index];
		image[iterationPath.pixelIndex] += iterationPath.color;
	}
}


struct pathsDead
{
	__host__ __device__
		bool operator()(const PathSegment &segment)
	{
		return (segment.remainingBounces > 0);
	}
};

struct cmp
{
	__host__ __device__  bool operator()(const ShadeableIntersection& intersect1, const ShadeableIntersection& intersect2) const
	{
		return (intersect1.materialId < intersect2.materialId);
	}
};


/**
 * Wrapper for the __global__ call that sets up the kernel calls and does a ton
 * of memory management
 */
void pathtrace(uchar4 *pbo, int frame, int iter) {
    const int traceDepth = hst_scene->state.traceDepth;
    const Camera &cam = hst_scene->state.camera;
    const int pixelcount = cam.resolution.x * cam.resolution.y;

	// 2D block for generating ray from camera
    const dim3 blockSize2d(8, 8);
    const dim3 blocksPerGrid2d(
            (cam.resolution.x + blockSize2d.x - 1) / blockSize2d.x,
            (cam.resolution.y + blockSize2d.y - 1) / blockSize2d.y);

	// 1D block for path tracing
	const int blockSize1d = 128;
	
	
    ///////////////////////////////////////////////////////////////////////////

    // Recap:
    // * Initialize array of path rays (using rays that come out of the camera)
    //   * You can pass the Camera object to that kernel.
    //   * Each path ray must carry at minimum a (ray, color) pair,
    //   * where color starts as the multiplicative identity, white = (1, 1, 1).
    //   * This has already been done for you.
    // * For each depth:
    //   * Compute an intersection in the scene for each path ray.
    //     A very naive version of this has been implemented for you, but feel
    //     free to add more primitives and/or a better algorithm.
    //     Currently, intersection distance is recorded as a parametric distance,
    //     t, or a "distance along the ray." t = -1.0 indicates no intersection.
    //     * Color is attenuated (multiplied) by reflections off of any object
    //   * TODO: Stream compact away all of the terminated paths.
    //     You may use either your implementation or `thrust::remove_if` or its
    //     cousins.
    //     * Note that you can't really use a 2D kernel launch any more - switch
    //       to 1D.
    //   * TODO: Shade the rays that intersected something or didn't bottom out.
    //     That is, color the ray by performing a color computation according
    //     to the shader, then generate a new ray to continue the ray path.
    //     We recommend just updating the ray's PathSegment in place.
    //     Note that this step may come before or after stream compaction,
    //     since some shaders you write may also cause a path to terminate.
    // * Finally, add this iteration's results to the image. This has been done
    //   for you.

    // TODO: perform one iteration of path tracing
	//int traceDepth = 4;
	generateRayFromCamera <<<blocksPerGrid2d, blockSize2d >>>(cam, iter, traceDepth, dev_paths, dev_remaining_paths);
	checkCUDAError("generate camera ray");

	//printf("Hello World\n");

	PathSegment* dev_path_end = dev_paths + pixelcount;
	int numPaths = dev_path_end - dev_paths;
	bool cacheFirstBounce = false;

	// --- PathSegment Tracing Stage ---
	// Shoot ray into scene, bounce between objects, push shading chunks
	bool iterationComplete = false;
	int depth = 0;

	# if CACHEFIRSTBOUNCE
		if (iter == 1) {
			cacheFirstBounce = true;
			printf("Hello");
		}	
	#endif


	while (!iterationComplete) {

		// clean shading chunks
		hipMemset(dev_intersections, 0, pixelcount * sizeof(ShadeableIntersection));

		// tracing
		dim3 numblocksPathSegmentTracing = (numPaths + blockSize1d - 1) / blockSize1d;

		#if RECORD_TIMING
			using namespace std::chrono;
			high_resolution_clock::time_point t1 = high_resolution_clock::now();
		#endif

		if (cacheFirstBounce) {
			computeIntersections << <numblocksPathSegmentTracing, blockSize1d >> > (
				depth
				, numPaths
				, dev_paths
				, dev_geoms
				, hst_scene->geoms.size()
				, dev_first_intersections
				, dev_remaining_paths
				);
			checkCUDAError("First trace bounce failed");
			hipDeviceSynchronize();
			hipMemcpy(dev_intersections, dev_first_intersections, numPaths * sizeof(ShadeableIntersection), hipMemcpyDeviceToDevice);
			cacheFirstBounce = false;
		}
		else if (depth == 0 && CACHEFIRSTBOUNCE) {
			hipMemcpy(dev_intersections, dev_first_intersections, numPaths * sizeof(ShadeableIntersection), hipMemcpyDeviceToDevice);
		}
		else {
			computeIntersections << <numblocksPathSegmentTracing, blockSize1d >> > (
				depth
				, numPaths
				, dev_paths
				, dev_geoms
				, hst_scene->geoms.size()
				, dev_intersections,
				dev_remaining_paths
				);
			checkCUDAError("Computer Intersections failed");
			hipDeviceSynchronize();
		}

		depth++;
		
		shaderKernel << <numblocksPathSegmentTracing, blockSize1d >> > (iter, numPaths, depth, dev_intersections, dev_materials, dev_paths, dev_remaining_paths);
		checkCUDAError("Gathering the final Image failed");
		hipDeviceSynchronize();

		#if STREAMCOMPACT_BY_THRUST
			dev_path_end = thrust::partition(thrust::device,dev_paths, dev_paths + numPaths, pathsDead());
			numPaths = dev_path_end - dev_paths;
		#endif		

		#if STREAM_COMPACT
			numPaths = StreamCompaction::Efficient::compact(numPaths,dev_remaining_paths);
		#endif		

		
		#if RAYSORT
			thrust::sort_by_key(thrust::device, dev_intersections, dev_intersections + numPaths, dev_paths, cmp());
		#endif

		if (depth >= traceDepth || numPaths<=0)
			iterationComplete = true; // TODO: should be based off stream compaction results.

		#if RECORD_TIMING
			high_resolution_clock::time_point t2 = high_resolution_clock::now();
			duration<double> time_span = duration_cast<duration<double>>(t2 - t1);
			if (iter < 5)
				std::cout << "For iter "<<iter<<", and depth "<<depth<<", it took" << time_span.count() << " seconds. The number of live paths are "<<numPaths<<endl;
		#endif
		
		// TODO:
		// --- Shading Stage ---
		// Shade path segments based on intersections and generate new rays by
		// evaluating the BSDF.
		// Start off with just a big kernel that handles all the different
		// materials you have in the scenefile.
		// TODO: compare between directly shading the path segments and shading
		// path segments that have been reshuffled to be contiguous in memory.

		/*
		shadeFakeMaterial << <numblocksPathSegmentTracing, blockSize1d >> > (
			iter,
			num_paths,
			dev_intersections,
			dev_paths,
			dev_materials
			);
		*/

		// Assemble this iteration and apply it to the image
	}
    ///////////////////////////////////////////////////////////////////////////
	numPaths = pixelcount;
	dim3 numBlocksPixels = (pixelcount + blockSize1d - 1) / blockSize1d;
	finalGather << <numBlocksPixels, blockSize1d >> > (numPaths, dev_image, dev_paths);

    // Send results to OpenGL buffer for rendering
    sendImageToPBO<<<blocksPerGrid2d, blockSize2d>>>(pbo, cam.resolution, iter, dev_image);

    // Retrieve image from GPU
    hipMemcpy(hst_scene->state.image.data(), dev_image,
            pixelcount * sizeof(glm::vec3), hipMemcpyDeviceToHost);

    checkCUDAError("pathtrace");
}
