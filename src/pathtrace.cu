#include "hip/hip_runtime.h"
#include <cstdio>
#include <hip/hip_runtime.h>
#include <cmath>
#include <thrust/execution_policy.h>
#include <thrust/random.h>
#include <thrust/partition.h>
#include <thrust/sort.h>

#include "sceneStructs.h"
#include "scene.h"
#include "glm/glm.hpp"
#include "glm/gtx/norm.hpp"
#include "utilities.h"
#include "pathtrace.h"
#include "intersections.h"
#include "interactions.h"

#define ERRORCHECK 1
#define CACHE_FIRST 1
#define STREAM_COMPACTION 1
#define SORT_BY_MATERIALS 1
#define DEPTH_OF_FIELD 0
#define MOTION_BLUR 0


#define FILENAME (strrchr(__FILE__, '/') ? strrchr(__FILE__, '/') + 1 : __FILE__)
#define checkCUDAError(msg) checkCUDAErrorFn(msg, FILENAME, __LINE__)
void checkCUDAErrorFn(const char *msg, const char *file, int line) {
#if ERRORCHECK
    hipDeviceSynchronize();
    hipError_t err = hipGetLastError();
    if (hipSuccess == err) {
        return;
    }

    fprintf(stderr, "CUDA error");
    if (file) {
        fprintf(stderr, " (%s:%d)", file, line);
    }
    fprintf(stderr, ": %s: %s\n", msg, hipGetErrorString(err));
#  ifdef _WIN32
    getchar();
#  endif
    exit(EXIT_FAILURE);
#endif
}

__host__ __device__
thrust::default_random_engine makeSeededRandomEngine(int iter, int index, int depth) {
    int h = utilhash((1 << 31) | (depth << 22) | iter) ^ utilhash(index);
    return thrust::default_random_engine(h);
}

//Kernel that writes the image to the OpenGL PBO directly.
__global__ void sendImageToPBO(uchar4* pbo, glm::ivec2 resolution,
        int iter, glm::vec3* image) {
    int x = (blockIdx.x * blockDim.x) + threadIdx.x;
    int y = (blockIdx.y * blockDim.y) + threadIdx.y;

    if (x < resolution.x && y < resolution.y) {
        int index = x + (y * resolution.x);
        glm::vec3 pix = image[index];

        glm::ivec3 color;
        color.x = glm::clamp((int) (pix.x / iter * 255.0), 0, 255);
        color.y = glm::clamp((int) (pix.y / iter * 255.0), 0, 255);
        color.z = glm::clamp((int) (pix.z / iter * 255.0), 0, 255);

        // Each thread writes one pixel location in the texture (textel)
        pbo[index].w = 0;
        pbo[index].x = color.x;
        pbo[index].y = color.y;
        pbo[index].z = color.z;
    }
}


static Scene * hst_scene = NULL;
static glm::vec3 * dev_image = NULL;
static Geom * dev_geoms = NULL;
static Material * dev_materials = NULL;
static PathSegment * dev_paths = NULL;
static ShadeableIntersection * dev_intersections = NULL;
// TODO: static variables for device memory, any extra info you need, etc
static ShadeableIntersection * dev_firstIntersections = NULL;
static int * dev_materialId;
static int * dev_sortedId;



struct rayTermination
{
	__host__ __device__
	bool operator()(const int &indices)
	{
		return indices == -1;
	}
};


void pathtraceInit(Scene *scene) {
    hst_scene = scene;
    const Camera &cam = hst_scene->state.camera;
    const int pixelcount = cam.resolution.x * cam.resolution.y;

    hipMalloc(&dev_image, pixelcount * sizeof(glm::vec3));
    hipMemset(dev_image, 0, pixelcount * sizeof(glm::vec3));

  	hipMalloc(&dev_paths, pixelcount * sizeof(PathSegment));

  	hipMalloc(&dev_geoms, scene->geoms.size() * sizeof(Geom));
  	hipMemcpy(dev_geoms, scene->geoms.data(), scene->geoms.size() * sizeof(Geom), hipMemcpyHostToDevice);

  	hipMalloc(&dev_materials, scene->materials.size() * sizeof(Material));
  	hipMemcpy(dev_materials, scene->materials.data(), scene->materials.size() * sizeof(Material), hipMemcpyHostToDevice);

  	hipMalloc(&dev_intersections, pixelcount * sizeof(ShadeableIntersection));
  	hipMemset(dev_intersections, 0, pixelcount * sizeof(ShadeableIntersection));

    // TODO: initialize any extra device memeory you need
	hipMalloc(&dev_firstIntersections, pixelcount * sizeof(ShadeableIntersection));
	hipMemset(dev_firstIntersections, 0, pixelcount * sizeof(ShadeableIntersection));
	hipMalloc(&dev_materialId, pixelcount * sizeof(int));
	hipMalloc(&dev_sortedId, pixelcount * sizeof(int));

    checkCUDAError("pathtraceInit");
}

void pathtraceFree() {
    hipFree(dev_image);  // no-op if dev_image is null
  	hipFree(dev_paths);
  	hipFree(dev_geoms);
  	hipFree(dev_materials);
  	hipFree(dev_intersections);
    // TODO: clean up any extra device memory you created
	hipFree(dev_firstIntersections);
	hipFree(dev_materialId);
	hipFree(dev_sortedId);
    checkCUDAError("pathtraceFree");
}

/**
* Generate PathSegments with rays from the camera through the screen into the
* scene, which is the first bounce of rays.
*
* Antialiasing - add rays for sub-pixel sampling
* motion blur - jitter rays "in time"
* lens effect - jitter ray origin positions based on a lens
*/
__global__ void generateRayFromCamera(Camera cam, int iter, int traceDepth, PathSegment* pathSegments, int* sortedId)
{
	int x = (blockIdx.x * blockDim.x) + threadIdx.x;
	int y = (blockIdx.y * blockDim.y) + threadIdx.y;

	if (x < cam.resolution.x && y < cam.resolution.y) {
		int index = x + (y * cam.resolution.x);
		PathSegment & segment = pathSegments[index];

		thrust::default_random_engine rng = makeSeededRandomEngine(iter, index, traceDepth);
		thrust::uniform_real_distribution<float> u01(0, 1);

		segment.ray.origin = cam.position;
        segment.color = glm::vec3(1.0f, 1.0f, 1.0f);

#if MOTION_BLUR
		thrust::normal_distribution<float> n01(0, 1);
		float a = abs(n01(rng));
		cam.view = cam.view + glm::vec3(0.1f, 0.f, 0.2f) * a;
#endif


		segment.ray.direction = glm::normalize(cam.view
			- cam.right * cam.pixelLength.x * ((float)x - (float)cam.resolution.x * 0.5f)
			- cam.up * cam.pixelLength.y * ((float)y - (float)cam.resolution.y * 0.5f)
			);
#if DEPTH_OF_FIELD
		float radius = cam.lensRadius;
		float focalDistance = cam.focalDist;
		float l = radius * u01(rng);
		float angle = 2 * PI * u01(rng);
		glm::vec3 pLens(l * cos(angle), l*sin(angle), 0.f);
		float ft = focalDistance / glm::abs(segment.ray.direction.z);
		glm::vec3 pFocus = ft * segment.ray.direction + segment.ray.origin;

		segment.ray.origin += pLens;
		segment.ray.direction = glm::normalize(pFocus - segment.ray.origin);
#endif
 
		segment.pixelIndex = index;
		segment.remainingBounces = traceDepth;
		sortedId[index] = index;
	}
}

// TODO:
// computeIntersections handles generating ray intersections ONLY.
// Generating new rays is handled in your shader(s).
// Feel free to modify the code below.
__global__ void computeIntersections(
	int depth
	, int num_paths
	, PathSegment * pathSegments
	, Geom * geoms
	, int geoms_size
	, ShadeableIntersection * intersections
	, int * materialId
	, int * sortedId
	)
{
	int path_index = blockIdx.x * blockDim.x + threadIdx.x;
	
	if (path_index < num_paths)
	{
		int sorted = sortedId[path_index];
		PathSegment pathSegment = pathSegments[sorted];

		float t;
		glm::vec3 intersect_point;
		glm::vec3 normal;
		float t_min = FLT_MAX;
		int hit_geom_index = -1;
		bool outside = true;

		glm::vec3 tmp_intersect;
		glm::vec3 tmp_normal;

		// naive parse through global geoms

		for (int i = 0; i < geoms_size; i++)
		{
			Geom & geom = geoms[i];

			if (geom.type == CUBE)
			{
				t = boxIntersectionTest(geom, pathSegment.ray, tmp_intersect, tmp_normal, outside);
			}
			else if (geom.type == SPHERE)
			{
				t = sphereIntersectionTest(geom, pathSegment.ray, tmp_intersect, tmp_normal, outside);
			}
			// TODO: add more intersection tests here... triangle? metaball? CSG?


			// Compute the minimum t from the intersection tests to determine what
			// scene geometry object was hit first.
			if (t > 0.0f && t_min > t)
			{
				t_min = t;
				hit_geom_index = i;
				normal = tmp_normal;
			}
		}

		if (hit_geom_index == -1)
		{
			intersections[sorted].t = -1.0f;
		}
		else
		{
			//The ray hits something
			intersections[sorted].t = t_min;
			intersections[sorted].materialId = geoms[hit_geom_index].materialid;
			intersections[sorted].surfaceNormal = normal;
			materialId[path_index] = intersections[sorted].materialId;
		}
	}
}



__global__ void shadeFakeMaterial(
	int iter
	, int depth
	, int num_paths
	, ShadeableIntersection * shadeableIntersections
	, PathSegment * pathSegments
	, Material * materials
	, int * sortedId
	)
{
  int idx = blockIdx.x * blockDim.x + threadIdx.x;
  if (idx < num_paths)
  {
	  int sorted = sortedId[idx];

      ShadeableIntersection intersection = shadeableIntersections[sorted];
	  PathSegment & segment = pathSegments[sorted];


    if (intersection.t > 0.0f) { // if the intersection exists...
      // Set up the RNG
      thrust::default_random_engine rng = makeSeededRandomEngine(iter, sorted, depth);
      thrust::uniform_real_distribution<float> u01(0, 1);

      Material material = materials[intersection.materialId];
      glm::vec3 materialColor = material.color;

      // If the material indicates that the object was a light, "light" the ray
      if (material.emittance > 0.0f) {
        segment.color *= (materialColor * material.emittance);
		segment.remainingBounces = 0;
      }
      // Otherwise, do some pseudo-lighting computation. This is actually more
      // like what you would expect from shading in a rasterizer like OpenGL.
      // TODO: replace this! you should be able to start with basically a one-liner
      else {
		  scatterRay(segment, getPointOnRay(segment.ray, intersection.t), intersection.surfaceNormal, material, rng);
		  segment.remainingBounces -= 1;
      }
    // If there was no intersection, color the ray black.
    // Lots of renderers use 4 channel color, RGBA, where A = alpha, often
    // used for opacity, in which case they can indicate "no opacity".
    // This can be useful for post-processing and image compositing.
    } else {
      segment.color = glm::vec3(0.0f);
	  segment.remainingBounces = 0;
    }

#if	STREAM_COMPACTION
	sortedId[idx] = segment.remainingBounces <= 0 ? -1: sorted;
#endif
  }
}

// Add the current iteration's output to the overall image
__global__ void finalGather(int nPaths, glm::vec3 * image, PathSegment * iterationPaths)
{
	int index = (blockIdx.x * blockDim.x) + threadIdx.x;

	if (index < nPaths)
	{
		PathSegment iterationPath = iterationPaths[index];
		image[iterationPath.pixelIndex] += iterationPath.color;
	}
}

/**
 * Wrapper for the __global__ call that sets up the kernel calls and does a ton
 * of memory management
 */
void pathtrace(uchar4 *pbo, int frame, int iter) {
    const int traceDepth = hst_scene->state.traceDepth;
    const Camera &cam = hst_scene->state.camera;
    const int pixelcount = cam.resolution.x * cam.resolution.y;

	// 2D block for generating ray from camera
    const dim3 blockSize2d(8, 8);
    const dim3 blocksPerGrid2d(
            (cam.resolution.x + blockSize2d.x - 1) / blockSize2d.x,
            (cam.resolution.y + blockSize2d.y - 1) / blockSize2d.y);

	// 1D block for path tracing
	const int blockSize1d = 128;

    ///////////////////////////////////////////////////////////////////////////

    // Recap:
    // * Initialize array of path rays (using rays that come out of the camera)
    //   * You can pass the Camera object to that kernel.
    //   * Each path ray must carry at minimum a (ray, color) pair,
    //   * where color starts as the multiplicative identity, white = (1, 1, 1).
    //   * This has already been done for you.
    // * For each depth:
    //   * Compute an intersection in the scene for each path ray.
    //     A very naive version of this has been implemented for you, but feel
    //     free to add more primitives and/or a better algorithm.
    //     Currently, intersection distance is recorded as a parametric distance,
    //     t, or a "distance along the ray." t = -1.0 indicates no intersection.
    //     * Color is attenuated (multiplied) by reflections off of any object
    //   * TODO: Stream compact away all of the terminated paths.
    //     You may use either your implementation or `thrust::remove_if` or its
    //     cousins.
    //     * Note that you can't really use a 2D kernel launch any more - switch
    //       to 1D.
    //   * TODO: Shade the rays that intersected something or didn't bottom out.
    //     That is, color the ray by performing a color computation according
    //     to the shader, then generate a new ray to continue the ray path.
    //     We recommend just updating the ray's PathSegment in place.
    //     Note that this step may come before or after stream compaction,
    //     since some shaders you write may also cause a path to terminate.
    // * Finally, add this iteration's results to the image. This has been done
    //   for you.

    // TODO: perform one iteration of path tracing

	generateRayFromCamera <<<blocksPerGrid2d, blockSize2d >>>(cam, iter, traceDepth, dev_paths, dev_sortedId);
	checkCUDAError("generate camera ray");

	int depth = 0;
	PathSegment* dev_path_end = dev_paths + pixelcount;
	int num_paths = dev_path_end - dev_paths;

	// --- PathSegment Tracing Stage ---
	// Shoot ray into scene, bounce between objects, push shading chunks

  bool iterationComplete = false;

	while (!iterationComplete) {
		// clean shading chunks
		hipMemset(dev_intersections, 0, pixelcount * sizeof(ShadeableIntersection));
		// tracing
		dim3 numblocksPathSegmentTracing = (num_paths + blockSize1d - 1) / blockSize1d;

#if CACHE_FIRST
		if (depth == 0) {
			if (iter == 1) {
				computeIntersections << <numblocksPathSegmentTracing, blockSize1d >> > (
					depth
					, num_paths
					, dev_paths
					, dev_geoms
					, hst_scene->geoms.size()
					, dev_intersections
					, dev_materialId
					, dev_sortedId
					);
				checkCUDAError("trace one bounce");
				hipMemcpy(dev_firstIntersections, dev_intersections, pixelcount * sizeof(ShadeableIntersection), hipMemcpyDeviceToDevice);
				checkCUDAError("copy first bounces");
			}
			else {
				// use cached first bounces
				hipMemcpy(dev_intersections, dev_firstIntersections, pixelcount * sizeof(ShadeableIntersection), hipMemcpyDeviceToDevice);
				checkCUDAError("copy back first bounces");
			}
		}
		else {
			computeIntersections << <numblocksPathSegmentTracing, blockSize1d >> > (
				depth
				, num_paths
				, dev_paths
				, dev_geoms
				, hst_scene->geoms.size()
				, dev_intersections
				, dev_materialId
				, dev_sortedId
				);
			checkCUDAError("trace one bounce");
		}

#else
		computeIntersections << <numblocksPathSegmentTracing, blockSize1d >> > (
			depth
			, num_paths
			, dev_paths
			, dev_geoms
			, hst_scene->geoms.size()
			, dev_intersections
			, dev_materialId
			, dev_sortedId
			);
		checkCUDAError("trace one bounce");
#endif
		hipDeviceSynchronize();

		
	  // --- Shading Stage ---

	  // Start off with just a big kernel that handles all the different
	  // materials you have in the scenefile.
	  // TODO: compare between directly shading the path segments and shading
	  // path segments that have been reshuffled to be contiguous in memory.
#if STREAM_COMPACTION
	  thrust::sort_by_key(thrust::device, dev_materialId, dev_materialId + num_paths, dev_sortedId);
#endif

	  shadeFakeMaterial<<<numblocksPathSegmentTracing, blockSize1d>>> (
		iter,
		depth,
		num_paths,
		dev_intersections,
		dev_paths,
		dev_materials,
		dev_sortedId
	  );
	  depth++;
	  if (depth >= traceDepth) {
		  iterationComplete = true;
	  }
	  #if SORT_BY_MATERIALS
	  else {
		  num_paths = thrust::remove_if(thrust::device, dev_sortedId, dev_sortedId + num_paths, rayTermination()) - dev_sortedId;
		  iterationComplete = num_paths <= 0;
		  std::cout << " " << num_paths;
	  }
	  #endif
	}

  // Assemble this iteration and apply it to the image
  dim3 numBlocksPixels = (pixelcount + blockSize1d - 1) / blockSize1d;
	finalGather<<<numBlocksPixels, blockSize1d>>>(pixelcount, dev_image, dev_paths);

    ///////////////////////////////////////////////////////////////////////////

    // Send results to OpenGL buffer for rendering
    sendImageToPBO<<<blocksPerGrid2d, blockSize2d>>>(pbo, cam.resolution, iter, dev_image);

    // Retrieve image from GPU
    hipMemcpy(hst_scene->state.image.data(), dev_image,
            pixelcount * sizeof(glm::vec3), hipMemcpyDeviceToHost);

    checkCUDAError("pathtrace");
}
