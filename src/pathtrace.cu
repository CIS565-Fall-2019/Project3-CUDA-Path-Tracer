#include "hip/hip_runtime.h"
#include <cstdio>
#include <hip/hip_runtime.h>
#include <cmath>
#include <thrust/execution_policy.h>
#include <thrust/random.h>
#include <thrust/remove.h>
#include <thrust/device_vector.h>
#include <thrust/partition.h>

#include "sceneStructs.h"
#include "scene.h"
#include "glm/glm.hpp"
#include "glm/gtx/norm.hpp"
#include "utilities.h"
#include "pathtrace.h"
#include "intersections.h"
#include "interactions.h"

#define ERRORCHECK 1

#define CACHE_FIRST_BOUNCE 1
#define SORT_BY_MATERIAL 1

#define FILENAME (strrchr(__FILE__, '/') ? strrchr(__FILE__, '/') + 1 : __FILE__)
#define checkCUDAError(msg) checkCUDAErrorFn(msg, FILENAME, __LINE__)
void checkCUDAErrorFn(const char *msg, const char *file, int line) {
#if ERRORCHECK
    hipDeviceSynchronize();
    hipError_t err = hipGetLastError();
    if (hipSuccess == err) {
        return;
    }

    fprintf(stderr, "CUDA error");
    if (file) {
        fprintf(stderr, " (%s:%d)", file, line);
    }
    fprintf(stderr, ": %s: %s\n", msg, hipGetErrorString(err));
#  ifdef _WIN32
    getchar();
#  endif
    exit(EXIT_FAILURE);
#endif
}

__host__ __device__
thrust::default_random_engine makeSeededRandomEngine(int iter, int index, int depth) {
    int h = utilhash((1 << 31) | (depth << 22) | iter) ^ utilhash(index);
    return thrust::default_random_engine(h);
}

//Kernel that writes the image to the OpenGL PBO directly.
__global__ void sendImageToPBO(uchar4* pbo, glm::ivec2 resolution,
        int iter, glm::vec3* image) {
    int x = (blockIdx.x * blockDim.x) + threadIdx.x;
    int y = (blockIdx.y * blockDim.y) + threadIdx.y;

    if (x < resolution.x && y < resolution.y) {
        int index = x + (y * resolution.x);
        glm::vec3 pix = image[index];

        glm::ivec3 color;
        color.x = glm::clamp((int) (pix.x / iter * 255.0), 0, 255);
        color.y = glm::clamp((int) (pix.y / iter * 255.0), 0, 255);
        color.z = glm::clamp((int) (pix.z / iter * 255.0), 0, 255);

        // Each thread writes one pixel location in the texture (textel)
        pbo[index].w = 0;
        pbo[index].x = color.x;
        pbo[index].y = color.y;
        pbo[index].z = color.z;
    }
}

static Scene * hst_scene = NULL;
static glm::vec3 * dev_image = NULL;
static Geom * dev_geoms = NULL;
static Material * dev_materials = NULL;
static PathSegment * dev_paths = NULL;
static ShadeableIntersection * dev_intersections = NULL;
static ShadeableIntersection * dev_cached_intersections = NULL;
// TODO: static variables for device memory, any extra info you need, etc
// ...

void pathtraceInit(Scene *scene) {
    hst_scene = scene;
    const Camera &cam = hst_scene->state.camera;
    const int pixelcount = cam.resolution.x * cam.resolution.y;

    hipMalloc(&dev_image, pixelcount * sizeof(glm::vec3));
    hipMemset(dev_image, 0, pixelcount * sizeof(glm::vec3));

  	hipMalloc(&dev_paths, pixelcount * sizeof(PathSegment));

  	hipMalloc(&dev_geoms, scene->geoms.size() * sizeof(Geom));
  	hipMemcpy(dev_geoms, scene->geoms.data(), scene->geoms.size() * sizeof(Geom), hipMemcpyHostToDevice);

  	hipMalloc(&dev_materials, scene->materials.size() * sizeof(Material));
  	hipMemcpy(dev_materials, scene->materials.data(), scene->materials.size() * sizeof(Material), hipMemcpyHostToDevice);

  	hipMalloc(&dev_intersections, pixelcount * sizeof(ShadeableIntersection));
  	hipMemset(dev_intersections, 0, pixelcount * sizeof(ShadeableIntersection));

	hipMalloc(&dev_cached_intersections, pixelcount * sizeof(ShadeableIntersection));
	hipMemset(dev_cached_intersections, 0, pixelcount * sizeof(ShadeableIntersection));

    // TODO: initialize any extra device memeory you need

    checkCUDAError("pathtraceInit");
}

void pathtraceFree() {
    hipFree(dev_image);  // no-op if dev_image is null
  	hipFree(dev_paths);
  	hipFree(dev_geoms);
  	hipFree(dev_materials);
  	hipFree(dev_intersections);
  	hipFree(dev_cached_intersections);
    // TODO: clean up any extra device memory you created

    checkCUDAError("pathtraceFree");
}

__host__ __device__ bool fequals(float a, float b) {
	float diff = a - b;
	return (diff < EPSILON) && (-diff < EPSILON);
}

__host__ __device__ glm::vec3 squareToDiskConcentric(glm::vec2 sample) {
	float a = 2.f * sample.x - 1.f;
	float b = 2.f * sample.y - 1.f;
	float r, phi;

	if (a > -b) {
		if (a > b) {
			r = a;
			phi = (PI / 4.f) * (b / a);
		}
		else {
			r = b;
			phi = (PI / 4.f) * (2 - (a / b));
		}
	}
	else {
		if (a < b) {
			r = -a;
			phi = (PI / 4.f) * (4.f + (b / a));
		}
		else {
			r = -b;
			if (!fequals(b, 0.f)) {
				phi = (PI / 4.f) * (6.f - (a / b));
			}
			else {
				phi = 0.f;
			}
		}
	}

	float x = r * cos(phi);
	float y = r * sin(phi);
	return glm::vec3(x, y, 0.f);
}

/**
* Generate PathSegments with rays from the camera through the screen into the
* scene, which is the first bounce of rays.
*
* Antialiasing - add rays for sub-pixel sampling
* motion blur - jitter rays "in time"
* lens effect - jitter ray origin positions based on a lens
*/
__global__ void generateRayFromCamera(Camera cam, int iter, int traceDepth, PathSegment* pathSegments)
{
	int x = (blockIdx.x * blockDim.x) + threadIdx.x;
	int y = (blockIdx.y * blockDim.y) + threadIdx.y;

	if (x < cam.resolution.x && y < cam.resolution.y) {
		int index = x + (y * cam.resolution.x);
		PathSegment & segment = pathSegments[index];

		segment.ray.origin = cam.position;
		segment.color = glm::vec3(1.0f, 1.0f, 1.0f);

		// TODO: implement antialiasing by jittering the ray
		segment.ray.direction = glm::normalize(cam.view
			- cam.right * cam.pixelLength.x * ((float)x - (float)cam.resolution.x * 0.5f)
			- cam.up * cam.pixelLength.y * ((float)y - (float)cam.resolution.y * 0.5f)
			);

		// depth of field via thin lens approximation
		
		if (cam.lensRadius > 0) {
			// sample point on lens
			thrust::default_random_engine rng = makeSeededRandomEngine(iter, index, 0);
			thrust::uniform_real_distribution<float> u01(0, 1);
			glm::vec2 xi = glm::vec2(u01(rng), u01(rng));
			glm::vec3 lensPoint = cam.lensRadius * squareToDiskConcentric(xi);
			
			// compute point on plane of focus
			glm::vec3 focusPoint = segment.ray.origin + segment.ray.direction * cam.focalDistance;

			// update ray
			segment.ray.origin += lensPoint.x * cam.right + lensPoint.y * cam.up;
			segment.ray.direction = glm::normalize(focusPoint - segment.ray.origin);
		}

		segment.pixelIndex = index;
		segment.remainingBounces = traceDepth;
	}
}

// TODO:
// computeIntersections handles generating ray intersections ONLY.
// Generating new rays is handled in your shader(s).
// Feel free to modify the code below.
__global__ void computeIntersections(int depth, int num_paths, PathSegment *pathSegments, 
	Geom *geoms, int geoms_size, ShadeableIntersection *intersections) {
	int path_index = blockIdx.x * blockDim.x + threadIdx.x;

	if (path_index < num_paths)
	{
		PathSegment pathSegment = pathSegments[path_index];

		float t;
		glm::vec3 intersect_point;
		glm::vec3 normal;
		float t_min = FLT_MAX;
		int hit_geom_index = -1;
		bool outside = true;

		glm::vec3 tmp_intersect;
		glm::vec3 tmp_normal;

		// naive parse through global geoms

		for (int i = 0; i < geoms_size; i++)
		{
			Geom & geom = geoms[i];

			if (geom.type == CUBE)
			{
				t = boxIntersectionTest(geom, pathSegment.ray, tmp_intersect, tmp_normal, outside);
			}
			else if (geom.type == SPHERE)
			{
				t = sphereIntersectionTest(geom, pathSegment.ray, tmp_intersect, tmp_normal, outside);
			}
			// TODO: add more intersection tests here... triangle? metaball? CSG?

			// Compute the minimum t from the intersection tests to determine what
			// scene geometry object was hit first.
			if (t > 0.0f && t_min > t)
			{
				t_min = t;
				hit_geom_index = i;
				intersect_point = tmp_intersect;
				normal = tmp_normal;
			}
		}

		if (hit_geom_index == -1)
		{
			intersections[path_index].t = -1.0f;
		}
		else
		{
			//The ray hits something
			intersections[path_index].t = t_min;
			intersections[path_index].materialId = geoms[hit_geom_index].materialid;
			intersections[path_index].surfaceNormal = normal;
			intersections[path_index].pos = intersect_point;
		}
	}
}

// LOOK: "fake" shader demonstrating what you might do with the info in
// a ShadeableIntersection, as well as how to use thrust's random number
// generator. Observe that since the thrust random number generator basically
// adds "noise" to the iteration, the image should start off noisy and get
// cleaner as more iterations are computed.
//
// Note that this shader does NOT do a BSDF evaluation!
// Your shaders should handle that - this can allow techniques such as
// bump mapping.
__global__ void shadeFakeMaterial(int iter, int num_paths, ShadeableIntersection * shadeableIntersections, 
	PathSegment * pathSegments, Material * materials) {
	int idx = blockIdx.x * blockDim.x + threadIdx.x;
	if (idx < num_paths)
	{
		ShadeableIntersection intersection = shadeableIntersections[idx];
		if (intersection.t > 0.0f) { // if the intersection exists...
			// Set up the RNG
			// LOOK: this is how you use thrust's RNG! Please look at
			// makeSeededRandomEngine as well.
			thrust::default_random_engine rng = makeSeededRandomEngine(iter, idx, 0);
			thrust::uniform_real_distribution<float> u01(0, 1);

			Material material = materials[intersection.materialId];
			glm::vec3 materialColor = material.color;

			// If the material indicates that the object was a light, "light" the ray
			if (material.emittance > 0.0f) {
				pathSegments[idx].color *= (materialColor * material.emittance);
			}
			// Otherwise, do some pseudo-lighting computation. This is actually more
			// like what you would expect from shading in a rasterizer like OpenGL.
			// TODO: replace this! you should be able to start with basically a one-liner
			else {
				float lightTerm = glm::dot(intersection.surfaceNormal, glm::vec3(0.0f, 1.0f, 0.0f));
				pathSegments[idx].color *= (materialColor * lightTerm) * 0.3f + ((1.0f - intersection.t * 0.02f) * materialColor) * 0.7f;
				pathSegments[idx].color *= u01(rng); // apply some noise because why not

			}
		} else {
			// If there was no intersection, color the ray black.
			// Lots of renderers use 4 channel color, RGBA, where A = alpha, often
			// used for opacity, in which case they can indicate "no opacity".
			// This can be useful for post-processing and image compositing.
			pathSegments[idx].color = glm::vec3(0.0f);
		}
	}
}


__global__ void shadeMaterial(int iter, int num_paths, ShadeableIntersection *shadeableIntersections,
	PathSegment *pathSegments, Material *materials) {
	int idx = blockIdx.x * blockDim.x + threadIdx.x;
	if (idx < num_paths)
	{
		ShadeableIntersection intersection = shadeableIntersections[idx];
		PathSegment& pathSegment = pathSegments[idx];

		if (intersection.t > 0.0f) { // if the intersection exists...
			// set up the RNG
			thrust::default_random_engine rng = makeSeededRandomEngine(iter, idx, 0);

			Material material = materials[intersection.materialId];

			// If the material indicates that the object was a light, "light" the ray
			if (material.emittance > 0.0f) {
				pathSegment.color *= (material.color * material.emittance);
				pathSegment.remainingBounces = 0; // np bounces after hitting light
			}
			else {
				// calculate accumulated color and new bounced ray
				scatterRay(pathSegment, intersection.pos, intersection.surfaceNormal, material, rng);
				pathSegment.remainingBounces--;
			}
		}
		else {
			// If there was no intersection, color the ray black.
			// Lots of renderers use 4 channel color, RGBA, where A = alpha, often
			// used for opacity, in which case they can indicate "no opacity".
			// This can be useful for post-processing and image compositing.
			pathSegment.color = glm::vec3(0.0f);
			pathSegment.remainingBounces = 0;
		}
	}
}


// Add the current iteration's output to the overall image
__global__ void finalGather(int nPaths, glm::vec3 *image, PathSegment *iterationPaths)
{
	int index = (blockIdx.x * blockDim.x) + threadIdx.x;

	if (index < nPaths)
	{
		PathSegment iterationPath = iterationPaths[index];
		image[iterationPath.pixelIndex] += iterationPath.color;
	}
}

struct isNotTerminated {
	__host__ __device__ bool operator()(const PathSegment &p) {
		return (p.remainingBounces > 0);
	}
};

struct sortMaterialId {
	__host__ __device__ bool operator()(const ShadeableIntersection &i1, const ShadeableIntersection &i2) {
		return (i1.materialId > i2.materialId);
	}
};


/**
 * Wrapper for the __global__ call that sets up the kernel calls and does a ton
 * of memory management
 */
void pathtrace(uchar4 *pbo, int frame, int iter) {
    const int traceDepth = hst_scene->state.traceDepth;
    const Camera &cam = hst_scene->state.camera;
    const int pixelcount = cam.resolution.x * cam.resolution.y;

	// 2D block for generating ray from camera
    const dim3 blockSize2d(8, 8);
    const dim3 blocksPerGrid2d(
            (cam.resolution.x + blockSize2d.x - 1) / blockSize2d.x,
            (cam.resolution.y + blockSize2d.y - 1) / blockSize2d.y);

	// 1D block for path tracing
	const int blockSize1d = 128;

	///////////////////////////////////////////////////////////////////////////

    // Recap:
    // * Initialize array of path rays (using rays that come out of the camera)
    //   * You can pass the Camera object to that kernel.
    //   * Each path ray must carry at minimum a (ray, color) pair,
    //   * where color starts as the multiplicative identity, white = (1, 1, 1).
    //   * This has already been done for you.
    // * For each depth:
    //   * Compute an intersection in the scene for each path ray.
    //     A very naive version of this has been implemented for you, but feel
    //     free to add more primitives and/or a better algorithm.
    //     Currently, intersection distance is recorded as a parametric distance,
    //     t, or a "distance along the ray." t = -1.0 indicates no intersection.
    //     * Color is attenuated (multiplied) by reflections off of any object
    //   * TODO: Stream compact away all of the terminated paths.
    //     You may use either your implementation or `thrust::remove_if` or its
    //     cousins.
    //     * Note that you can't really use a 2D kernel launch any more - switch
    //       to 1D.
    //   * TODO: Shade the rays that intersected something or didn't bottom out.
    //     That is, color the ray by performing a color computation according
    //     to the shader, then generate a new ray to continue the ray path.
    //     We recommend just updating the ray's PathSegment in place.
    //     Note that this step may come before or after stream compaction,
    //     since some shaders you write may also cause a path to terminate.
    // * Finally, add this iteration's results to the image. This has been done
    //   for you.

    // TODO: perform one iteration of path tracing

	generateRayFromCamera<<<blocksPerGrid2d, blockSize2d>>>(cam, iter, traceDepth, dev_paths);
	checkCUDAError("generate camera ray");

	int depth = 0;
	PathSegment* dev_path_end = dev_paths + pixelcount;
	int num_paths = dev_path_end - dev_paths;
	int remaining_paths = num_paths;

	// --- PathSegment Tracing Stage ---
	// Shoot ray into scene, bounce between objects, push shading chunks

	// for debugging
	ShadeableIntersection* test = new ShadeableIntersection[num_paths];

	bool iterationComplete = false;
	while (!iterationComplete) {

		// clean shading chunks
		hipMemset(dev_intersections, 0, pixelcount * sizeof(ShadeableIntersection));

		// tracing
		dim3 numblocksPathSegmentTracing = (remaining_paths + blockSize1d - 1) / blockSize1d;
		
#if CACHE_FIRST_BOUNCE
		if (depth == 0) {
			if (iter == 1) {
				// if first bounce of first iteration, cache intersections
				computeIntersections<<<numblocksPathSegmentTracing, blockSize1d>>>(depth, remaining_paths, dev_paths, dev_geoms, hst_scene->geoms.size(), dev_intersections);
				hipMemcpy(dev_cached_intersections, dev_intersections, pixelcount * sizeof(ShadeableIntersection), hipMemcpyDeviceToDevice);
			}
			else {
				// if first bounce of later iterations, pull from cache
				hipMemcpy(dev_intersections, dev_cached_intersections, pixelcount * sizeof(ShadeableIntersection), hipMemcpyDeviceToDevice);
			}
		}
		else {
#endif // #if CACHE_FIRST_BOUNCE 
			computeIntersections<<<numblocksPathSegmentTracing, blockSize1d>>>(depth, remaining_paths, dev_paths, dev_geoms, hst_scene->geoms.size(), dev_intersections);
#if CACHE_FIRST_BOUNCE
		}
#endif // #if CACHE_FIRST_BOUNCE 

		checkCUDAError("trace one bounce");
		hipDeviceSynchronize();
		depth++;

		// debug code
		//hipMemcpy(test, dev_intersections, pixelcount * sizeof(ShadeableIntersection), hipMemcpyDeviceToHost);


#if SORT_BY_MATERIAL
		// before we shade, sort path segments and intersections by material id
		thrust::sort_by_key(thrust::device, dev_intersections, dev_intersections + remaining_paths, dev_paths, sortMaterialId());
#endif // #if SORT_BY_MATERIAL

		//hipMemcpy(test, dev_intersections, pixelcount * sizeof(ShadeableIntersection), hipMemcpyDeviceToHost);


		// TODO:
		// --- Shading Stage ---
		// Shade path segments based on intersections and generate new rays by
		// evaluating the BSDF.
		// Start off with just a big kernel that handles all the different
		// materials you have in the scenefile.
		// TODO: compare between directly shading the path segments and shading
		// path segments that have been reshuffled to be contiguous in memory.

		shadeMaterial<<<numblocksPathSegmentTracing, blockSize1d>>>(iter, remaining_paths, dev_intersections, dev_paths, dev_materials);

		//hipMemcpy(test, dev_paths, num_paths * sizeof(PathSegment), hipMemcpyDeviceToHost);

		// run stream compaction to remove terminated rays
		dev_path_end = thrust::partition(thrust::device, dev_paths, dev_paths + remaining_paths, isNotTerminated());
		//dev_path_end = thrust::partition(thrust::device, dev_paths, dev_paths + num_paths, isNotTerminated());
		remaining_paths = dev_path_end - dev_paths;

		//hipMemcpy(test, dev_paths, num_paths * sizeof(PathSegment), hipMemcpyDeviceToHost);

		// if all rays are terminated, iteration is complete
		iterationComplete = (remaining_paths == 0) || (depth > traceDepth); // TODO: should be based off stream compaction results.
	}

	// Assemble this iteration and apply it to the image
	dim3 numBlocksPixels = (pixelcount + blockSize1d - 1) / blockSize1d;
	finalGather<<<numBlocksPixels, blockSize1d>>>(num_paths, dev_image, dev_paths);

    ///////////////////////////////////////////////////////////////////////////

    // Send results to OpenGL buffer for rendering
    sendImageToPBO<<<blocksPerGrid2d, blockSize2d>>>(pbo, cam.resolution, iter, dev_image);

    // Retrieve image from GPU
    hipMemcpy(hst_scene->state.image.data(), dev_image, pixelcount * sizeof(glm::vec3), hipMemcpyDeviceToHost);

    checkCUDAError("pathtrace");
}
