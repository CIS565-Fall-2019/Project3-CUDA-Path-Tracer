#include "hip/hip_runtime.h"
#include <cstdio>
#include <hip/hip_runtime.h>
#include <cmath>
#include <thrust/execution_policy.h>
#include <thrust/random.h>
#include <thrust/remove.h>
#include <thrust/partition.h>
#include "sceneStructs.h"
#include "scene.h"
#include "glm/glm.hpp"
#include "glm/gtx/norm.hpp"
#include "utilities.h"
#include "pathtrace.h"
#include "intersections.h"
#include "interactions.h"
#include <glm/gtc/matrix_inverse.hpp>
#include<glm/gtc/matrix_transform.hpp>
#include<chrono>


#define ERRORCHECK 1
#define MATERIAL_SORT 0
#define STREAM_COMPACT 1
#define CACHE_BOUNCE 1
#define MOTION_BLUR 0
#define ANTI_ALIAS 0
#define DENOISE 0

#define FILENAME (strrchr(__FILE__, '/') ? strrchr(__FILE__, '/') + 1 : __FILE__)
#define checkCUDAError(msg) checkCUDAErrorFn(msg, FILENAME, __LINE__)
void checkCUDAErrorFn(const char *msg, const char *file, int line) {
#if ERRORCHECK
    hipDeviceSynchronize();
    hipError_t err = hipGetLastError();
    if (hipSuccess == err) {
        return;
    }

    fprintf(stderr, "CUDA error");
    if (file) {
        fprintf(stderr, " (%s:%d)", file, line);
    }
    fprintf(stderr, ": %s: %s\n", msg, hipGetErrorString(err));
#  ifdef _WIN32
    getchar();
#  endif
    exit(EXIT_FAILURE);
#endif
}

__host__ __device__
thrust::default_random_engine makeSeededRandomEngine(int iter, int index, int depth) {
    int h = utilhash((1 << 31) | (depth << 22) | iter) ^ utilhash(index);
    return thrust::default_random_engine(h);
}

//Kernel that writes the image to the OpenGL PBO directly.
__global__ void sendImageToPBO(uchar4* pbo, glm::ivec2 resolution,
        int iter, glm::vec3* image) {
    int x = (blockIdx.x * blockDim.x) + threadIdx.x;
    int y = (blockIdx.y * blockDim.y) + threadIdx.y;

    if (x < resolution.x && y < resolution.y) {
        int index = x + (y * resolution.x);
        glm::vec3 pix = image[index];

        glm::ivec3 color;
        color.x = glm::clamp((int) (pix.x / iter * 255.0), 0, 255);
        color.y = glm::clamp((int) (pix.y / iter * 255.0), 0, 255);
        color.z = glm::clamp((int) (pix.z / iter * 255.0), 0, 255);

        // Each thread writes one pixel location in the texture (textel)
        pbo[index].w = 0;
        pbo[index].x = color.x;
        pbo[index].y = color.y;
        pbo[index].z = color.z;
    }
}

static Scene * hst_scene = NULL;
static glm::vec3 * dev_image = NULL;
static Geom * dev_geoms = NULL;
static Material * dev_materials = NULL;
static PathSegment * dev_paths = NULL;
static ShadeableIntersection * dev_intersections = NULL;
static ShadeableIntersection * dev_intersections_first = NULL;
static glm::vec3 * dev_normal = NULL;
static glm::vec3 * dev_albedo = NULL;

// TODO: static variables for device memory, any extra info you need, etc
// ...

void pathtraceInit(Scene *scene) {
    hst_scene = scene;
    const Camera &cam = hst_scene->state.camera;
    const int pixelcount = cam.resolution.x * cam.resolution.y;

    hipMalloc(&dev_image, pixelcount * sizeof(glm::vec3));
    hipMemset(dev_image, 0, pixelcount * sizeof(glm::vec3));

	hipMalloc(&dev_normal, pixelcount * sizeof(glm::vec3));
	hipMemset(dev_normal, 0, pixelcount * sizeof(glm::vec3));

	hipMalloc(&dev_albedo, pixelcount * sizeof(glm::vec3));
	hipMemset(dev_albedo, 0, pixelcount * sizeof(glm::vec3));

  	hipMalloc(&dev_paths, pixelcount * sizeof(PathSegment));

  	hipMalloc(&dev_geoms, scene->geoms.size() * sizeof(Geom));
  	hipMemcpy(dev_geoms, scene->geoms.data(), scene->geoms.size() * sizeof(Geom), hipMemcpyHostToDevice);

  	hipMalloc(&dev_materials, scene->materials.size() * sizeof(Material));
  	hipMemcpy(dev_materials, scene->materials.data(), scene->materials.size() * sizeof(Material), hipMemcpyHostToDevice);

  	hipMalloc(&dev_intersections, pixelcount * sizeof(ShadeableIntersection));
  	hipMemset(dev_intersections, 0, pixelcount * sizeof(ShadeableIntersection));

	hipMalloc(&dev_intersections_first, pixelcount * sizeof(ShadeableIntersection));
	hipMemset(dev_intersections_first, 0, pixelcount * sizeof(ShadeableIntersection));


    // TODO: initialize any extra device memeory you need

    checkCUDAError("pathtraceInit");
}

void pathtraceFree() {
    hipFree(dev_image);  // no-op if dev_image is null
  	hipFree(dev_paths);
  	hipFree(dev_geoms);
  	hipFree(dev_materials);
  	hipFree(dev_intersections);
    // TODO: clean up any extra device memory you created
	hipFree(dev_intersections_first);
	hipFree(dev_normal);
	hipFree(dev_albedo);
    checkCUDAError("pathtraceFree");
}

/**
* Generate PathSegments with rays from the camera through the screen into the
* scene, which is the first bounce of rays.
*
* Antialiasing - add rays for sub-pixel sampling
* motion blur - jitter rays "in time"
* lens effect - jitter ray origin positions based on a lens
*/
__global__ void generateRayFromCamera(Camera cam, int iter, int traceDepth, PathSegment* pathSegments)
{
	int x = (blockIdx.x * blockDim.x) + threadIdx.x;
	int y = (blockIdx.y * blockDim.y) + threadIdx.y;

	if (x < cam.resolution.x && y < cam.resolution.y) {
		int index = x + (y * cam.resolution.x);
		PathSegment & segment = pathSegments[index];

		segment.ray.origin = cam.position;
		segment.color = glm::vec3(1.0f, 1.0f, 1.0f);
	
		// TODO: implement antialiasing by jittering the ray
		thrust::default_random_engine rng = makeSeededRandomEngine(iter, index, traceDepth);
		thrust::uniform_real_distribution<float> dist(0, 1);
		if (ANTI_ALIAS) {
			segment.ray.direction = glm::normalize(cam.view
				- cam.right * cam.pixelLength.x * ((float)x - (float)cam.resolution.x * 0.5f + dist(rng))
				- cam.up * cam.pixelLength.y * ((float)y - (float)cam.resolution.y * 0.5f + dist(rng))
			);
		}
		else {
			segment.ray.direction = glm::normalize(cam.view
				- cam.right * cam.pixelLength.x * ((float)x - (float)cam.resolution.x * 0.5f)
				- cam.up * cam.pixelLength.y * ((float)y - (float)cam.resolution.y * 0.5f)
			);
		}
		
		segment.pixelIndex = index;
		segment.remainingBounces = traceDepth;
	}
}

// TODO:
// computeIntersections handles generating ray intersections ONLY.
// Generating new rays is handled in your shader(s).
// Feel free to modify the code below.
__global__ void computeIntersections(
	int depth
	, int num_paths
	, PathSegment * pathSegments
	, Geom * geoms
	, int geoms_size
	, ShadeableIntersection * intersections
	, glm::vec3 * dev_albedo
	, glm::vec3 * dev_normal
	, int iter
	)
{
	int path_index = blockIdx.x * blockDim.x + threadIdx.x;

	if (path_index < num_paths)
	{
		PathSegment pathSegment = pathSegments[path_index];

		float t;
		glm::vec3 intersect_point;
		glm::vec3 normal;
		float t_min = FLT_MAX;
		int hit_geom_index = -1;
		bool outside = true;

		glm::vec3 tmp_intersect;
		glm::vec3 tmp_normal;

		// naive parse through global geoms

		for (int i = 0; i < geoms_size; i++)
		{
			Geom & geom = geoms[i];

			if (geom.type == CUBE)
			{
				t = boxIntersectionTest(geom, pathSegment.ray, tmp_intersect, tmp_normal, outside);
			}
			else if (geom.type == SPHERE)
			{
				t = sphereIntersectionTest(geom, pathSegment.ray, tmp_intersect, tmp_normal, outside);
			}
			// TODO: add more intersection tests here... triangle? metaball? CSG?

			// Compute the minimum t from the intersection tests to determine what
			// scene geometry object was hit first.
			if (t > 0.0f && t_min > t)
			{
				t_min = t;
				hit_geom_index = i;
				intersect_point = tmp_intersect;
				normal = tmp_normal;
			}
		}

		if (hit_geom_index == -1)
		{
			intersections[path_index].t = -1.0f;
		}
		else
		{
			//The ray hits something
			intersections[path_index].t = t_min;
			intersections[path_index].materialId = geoms[hit_geom_index].materialid;
			intersections[path_index].surfaceNormal = normal;
		}
		if (DENOISE && iter == 0) {
			dev_normal[path_index] = intersections[path_index].surfaceNormal;
			dev_albedo[path_index] = pathSegment.color;
		}
	}
}

// LOOK: "fake" shader demonstrating what you might do with the info in
// a ShadeableIntersection, as well as how to use thrust's random number
// generator. Observe that since the thrust random number generator basically
// adds "noise" to the iteration, the image should start off noisy and get
// cleaner as more iterations are computed.
//
// Note that this shader does NOT do a BSDF evaluation!
// Your shaders should handle that - this can allow techniques such as
// bump mapping.

__global__ void shadeRealMaterial(
	int iter
	, int num_paths
	, ShadeableIntersection * shadeableIntersections
	, PathSegment * pathSegments
	, Material * materials
)
{
	int idx = blockIdx.x * blockDim.x + threadIdx.x;
	if (idx < num_paths && pathSegments[idx].remainingBounces >0)
	{
		

		ShadeableIntersection intersection = shadeableIntersections[idx];
		if (intersection.t > 0.0f) { // if the intersection exists...
		  // Set up the RNG
		  // LOOK: this is how you use thrust's RNG! Please look at
		  // makeSeededRandomEngine as well.
			glm::vec3 intersect = getPointOnRay(pathSegments[idx].ray, intersection.t);
			thrust::default_random_engine rng = makeSeededRandomEngine(iter, idx, pathSegments[idx].remainingBounces);
			Material material = materials[intersection.materialId];
			glm::vec3 materialColor = material.color;

			// If the material indicates that the object was a light, "light" the ray
			if (material.emittance > 0.0f) {
				pathSegments[idx].color = pathSegments[idx].color *(materialColor * material.emittance);

				pathSegments[idx].remainingBounces = 0;
			}
			// Otherwise, do some pseudo-lighting computation. This is actually more
			// like what you would expect from shading in a rasterizer like OpenGL.
			// TODO: replace this! you should be able to start with basically a one-liner
			else if(pathSegments[idx].remainingBounces > 0) {
				scatterRay(pathSegments[idx], intersection.t, intersect, intersection.surfaceNormal, material, rng);
				pathSegments[idx].remainingBounces--;
				
			}
			// If there was no intersection, color the ray black.
			// Lots of renderers use 4 channel color, RGBA, where A = alpha, often
			// used for opacity, in which case they can indicate "no opacity".
			// This can be useful for post-processing and image compositing.
		}
		else {
			pathSegments[idx].color = glm::vec3(0.0f);
			pathSegments[idx].remainingBounces = 0;
		}
	}
}

// Add the current iteration's output to the overall image
__global__ void finalGather(int nPaths, glm::vec3 * image, PathSegment * iterationPaths)
{
	int index = (blockIdx.x * blockDim.x) + threadIdx.x;

	if (index < nPaths)
	{
		PathSegment iterationPath = iterationPaths[index];
		image[iterationPath.pixelIndex] += iterationPath.color;
	}
}

__device__ glm::mat4 buildTransformationMatrix(glm::vec3 translation, glm::vec3 rotation, glm::vec3 scale) {
	glm::mat4 translationMat = glm::translate(glm::mat4(), translation);
	glm::mat4 rotationMat = glm::rotate(glm::mat4(), rotation.x * (float)PI / 180, glm::vec3(1, 0, 0));
	rotationMat = rotationMat * glm::rotate(glm::mat4(), rotation.y * (float)PI / 180, glm::vec3(0, 1, 0));
	rotationMat = rotationMat * glm::rotate(glm::mat4(), rotation.z * (float)PI / 180, glm::vec3(0, 0, 1));
	glm::mat4 scaleMat = glm::scale(glm::mat4(), scale);
	return translationMat * rotationMat * scaleMat;
}

__global__ void motionBlur(Geom geom[], int numGeoms, float dt) {
	int index = (blockIdx.x * blockDim.x) + threadIdx.x;
	if (index >= numGeoms)
		return;
	geom[index].translation += geom[index].velocity*dt;
	geom[index].transform = buildTransformationMatrix(
		geom[index].translation, geom[index].rotation, geom[index].scale);
	geom[index].inverseTransform = glm::inverse(geom[index].transform);
	geom[index].invTranspose = glm::inverseTranspose(geom[index].transform);
}
struct my_partition_functor
{
	__host__ __device__
		bool operator()(const PathSegment &pathSegment) {
		if (pathSegment.remainingBounces > 0) {
			return true;
		}
		return false;
	}
};

struct my_comp_functor
{

	__host__ __device__
		bool operator()(const ShadeableIntersection &s1, const ShadeableIntersection &s2) {
		if (s1.materialId < s2.materialId) {
			return true;
		}
		return false;
	}
};
/**
 * Wrapper for the __global__ call that sets up the kernel calls and does a ton
 * of memory management
 */
void pathtrace(uchar4 *pbo, int frame, int iter) {
    const int traceDepth = hst_scene->state.traceDepth;
    const Camera &cam = hst_scene->state.camera;
    const int pixelcount = cam.resolution.x * cam.resolution.y;

	// 2D block for generating ray from camera
    const dim3 blockSize2d(8, 8);
    const dim3 blocksPerGrid2d(
            (cam.resolution.x + blockSize2d.x - 1) / blockSize2d.x,
            (cam.resolution.y + blockSize2d.y - 1) / blockSize2d.y);

	// 1D block for path tracing
	const int blockSize1d = 128;

    ///////////////////////////////////////////////////////////////////////////

    // Recap:
    // * Initialize array of path rays (using rays that come out of the camera)
    //   * You can pass the Camera object to that kernel.
    //   * Each path ray must carry at minimum a (ray, color) pair,
    //   * where color starts as the multiplicative identity, white = (1, 1, 1).
    //   * This has already been done for you.
    // * For each depth:
    //   * Compute an intersection in the scene for each path ray.
    //     A very naive version of this has been implemented for you, but feel
    //     free to add more primitives and/or a better algorithm.
    //     Currently, intersection distance is recorded as a parametric distance,
    //     t, or a "distance along the ray." t = -1.0 indicates no intersection.
    //     * Color is attenuated (multiplied) by reflections off of any object
    //   * TODO: Stream compact away all of the terminated paths.
    //     You may use either your implementation or `thrust::remove_if` or its
    //     cousins.
    //     * Note that you can't really use a 2D kernel launch any more - switch
    //       to 1D.
    //   * TODO: Shade the rays that intersected something or didn't bottom out.
    //     That is, color the ray by performing a color computation according
    //     to the shader, then generate a new ray to continue the ray path.
    //     We recommend just updating the ray's PathSegment in place.
    //     Note that this step may come before or after stream compaction,
    //     since some shaders you write may also cause a path to terminate.
    // * Finally, add this iteration's results to the image. This has been done
    //   for you.

    // TODO: perform one iteration of path tracing

	generateRayFromCamera <<<blocksPerGrid2d, blockSize2d >>>(cam, iter, traceDepth, dev_paths);
	checkCUDAError("generate camera ray");

	int depth = 0;
	PathSegment* dev_path_end = dev_paths + pixelcount;
	int num_paths = dev_path_end - dev_paths;

	// --- PathSegment Tracing Stage ---
	// Shoot ray into scene, bounce between objects, push shading chunks

  bool iterationComplete = false;
  bool firstIteration = true;
  auto start = std::chrono::high_resolution_clock::now();

  dim3 numblocksPathSegmentTracing = (hst_scene->geoms.size() + blockSize1d - 1) / blockSize1d;
  if(MOTION_BLUR)
	motionBlur << <numblocksPathSegmentTracing, blockSize1d >> > (dev_geoms, hst_scene->geoms.size(), 0.1);
	while (!iterationComplete) {

	// clean shading chunks
	hipMemset(dev_intersections, 0, pixelcount * sizeof(ShadeableIntersection));

	// tracing
	dim3 numblocksPathSegmentTracing = (num_paths + blockSize1d - 1) / blockSize1d;
	ShadeableIntersection * inter = dev_intersections;
	if (CACHE_BOUNCE) {
		if (depth == 0) {
			if (firstIteration) {
				computeIntersections << <numblocksPathSegmentTracing, blockSize1d >> > (
					depth
					, num_paths
					, dev_paths
					, dev_geoms
					, hst_scene->geoms.size()
					, dev_intersections
					, dev_albedo
					, dev_normal
					, iter
					);
				checkCUDAError("trace one bounce");
				hipMemcpy(dev_intersections_first, dev_intersections, pixelcount * sizeof(ShadeableIntersection), hipMemcpyDeviceToDevice);
				firstIteration = false;
			}
			else {
				inter = dev_intersections_first;
			}
		}
		else {
			computeIntersections << <numblocksPathSegmentTracing, blockSize1d >> > (
				depth
				, num_paths
				, dev_paths
				, dev_geoms
				, hst_scene->geoms.size()
				, dev_intersections
				, dev_albedo
				, dev_normal
				, iter
				);
			checkCUDAError("trace one bounce");
		}
	}
	else {
		computeIntersections << <numblocksPathSegmentTracing, blockSize1d >> > (
			depth
			, num_paths
			, dev_paths
			, dev_geoms
			, hst_scene->geoms.size()
			, dev_intersections
			, dev_albedo
			, dev_normal
			, iter
			);
		checkCUDAError("trace one bounce");
	}
	
	hipDeviceSynchronize();
	depth++;


	// TODO:
	// --- Shading Stage ---
	// Shade path segments based on intersections and generate new rays by
  // evaluating the BSDF.
  // Start off with just a big kernel that handles all the different
  // materials you have in the scenefile.
  // TODO: compare between directly shading the path segments and shading
  // path segments that have been reshuffled to be contiguous in memory.

  shadeRealMaterial<<<numblocksPathSegmentTracing, blockSize1d>>> (
    iter,
    num_paths,
    inter,
    dev_paths,
    dev_materials
  );

  if (STREAM_COMPACT) {
	  dev_path_end = thrust::partition(thrust::device, dev_paths, dev_paths + num_paths, my_partition_functor());
	  num_paths = dev_path_end - dev_paths;
  }
  if (MATERIAL_SORT) {
	  thrust::sort_by_key(thrust::device, dev_intersections, dev_intersections + num_paths, dev_paths, my_comp_functor());
  }


  // TODO: should be based off stream compaction results.
  if (num_paths <= 0) {
	  iterationComplete = true;
	  num_paths = pixelcount;
  }
  iterationComplete = (depth == traceDepth);
	
}
	

  // Assemble this iteration and apply it to the image
  dim3 numBlocksPixels = (pixelcount + blockSize1d - 1) / blockSize1d;
	finalGather<<<numBlocksPixels, blockSize1d>>>(num_paths, dev_image, dev_paths);

    ///////////////////////////////////////////////////////////////////////////

    // Send results to OpenGL buffer for rendering
    sendImageToPBO<<<blocksPerGrid2d, blockSize2d>>>(pbo, cam.resolution, iter, dev_image);
    // Retrieve image from GPU
    hipMemcpy(hst_scene->state.image.data(), dev_image,
            pixelcount * sizeof(glm::vec3), hipMemcpyDeviceToHost);
	hipMemcpy(hst_scene->state.normals.data(), dev_normal,
		pixelcount * sizeof(glm::vec3), hipMemcpyDeviceToHost);
	hipMemcpy(hst_scene->state.albedos.data(), dev_albedo,
		pixelcount * sizeof(glm::vec3), hipMemcpyDeviceToHost);

    checkCUDAError("pathtrace");
}
