#include "hip/hip_runtime.h"
#include <cstdio>
#include <hip/hip_runtime.h>
#include <cmath>
#include <thrust/execution_policy.h>
#include <thrust/random.h>
#include <thrust/remove.h>
#include <thrust/partition.h>
#include <chrono>

#include "sceneStructs.h"
#include "scene.h"
#include "glm/glm.hpp"
#include "glm/gtx/norm.hpp"
#include "utilities.h"
#include "pathtrace.h"
#include "intersections.h"
#include "interactions.h"

using time_point_t = std::chrono::high_resolution_clock::time_point;
time_point_t time_start;
time_point_t time_end;
float avg_time = 0;

#define ERRORCHECK 1

#define FILENAME (strrchr(__FILE__, '/') ? strrchr(__FILE__, '/') + 1 : __FILE__)
#define checkCUDAError(msg) checkCUDAErrorFn(msg, FILENAME, __LINE__)
void checkCUDAErrorFn(const char *msg, const char *file, int line) {
#if ERRORCHECK
    hipDeviceSynchronize();
    hipError_t err = hipGetLastError();
    if (hipSuccess == err) {
        return;
    }

    fprintf(stderr, "CUDA error");
    if (file) {
        fprintf(stderr, " (%s:%d)", file, line);
    }
    fprintf(stderr, ": %s: %s\n", msg, hipGetErrorString(err));
#  ifdef _WIN32
    getchar();
#  endif
    exit(EXIT_FAILURE);
#endif
}

struct ray_continuation_condition {
  __host__ __device__ bool operator()(const PathSegment& s) {
    return s.remainingBounces > 0;
  }
};

struct material_id_comparator {
  __host__ __device__ bool operator()(const ShadeableIntersection& s1, const ShadeableIntersection& s2) {
    return s1.materialId < s2.materialId;
  }
};

__host__ __device__
thrust::default_random_engine makeSeededRandomEngine(int iter, int index, int depth) {
    int h = utilhash((1 << 31) | (depth << 22) | iter) ^ utilhash(index);
    return thrust::default_random_engine(h);
}

//Kernel that writes the image to the OpenGL PBO directly.
__global__ void sendImageToPBO(uchar4* pbo, glm::ivec2 resolution,
        int iter, glm::vec3* image) {
    int x = (blockIdx.x * blockDim.x) + threadIdx.x;
    int y = (blockIdx.y * blockDim.y) + threadIdx.y;

    if (x < resolution.x && y < resolution.y) {
        int index = x + (y * resolution.x);
        glm::vec3 pix = image[index];

        glm::ivec3 color;
        color.x = glm::clamp((int) (pix.x / iter * 255.0), 0, 255);
        color.y = glm::clamp((int) (pix.y / iter * 255.0), 0, 255);
        color.z = glm::clamp((int) (pix.z / iter * 255.0), 0, 255);

        // Each thread writes one pixel location in the texture (textel)
        pbo[index].w = 0;
        pbo[index].x = color.x;
        pbo[index].y = color.y;
        pbo[index].z = color.z;
    }
}

static Scene * hst_scene = NULL;
static glm::vec3 * dev_image = NULL;
static Geom * dev_geoms = NULL;
static Material * dev_materials = NULL;
static PathSegment * dev_paths = NULL;
static ShadeableIntersection * dev_intersections = NULL;
// TODO: static variables for device memory, any extra info you need, etc
// ...
static ShadeableIntersection * dev_intersections_cache = NULL;    // cache first iteration.
static Triangle * dev_triangles = NULL;                           // triangles

void pathtraceInit(Scene *scene) {
    hst_scene = scene;
    const Camera &cam = hst_scene->state.camera;
    const int pixelcount = cam.resolution.x * cam.resolution.y;

    hipMalloc(&dev_image, pixelcount * sizeof(glm::vec3));
    hipMemset(dev_image, 0, pixelcount * sizeof(glm::vec3));

  	hipMalloc(&dev_paths, pixelcount * sizeof(PathSegment));

  	hipMalloc(&dev_geoms, scene->geoms.size() * sizeof(Geom));
  	hipMemcpy(dev_geoms, scene->geoms.data(), scene->geoms.size() * sizeof(Geom), hipMemcpyHostToDevice);

  	hipMalloc(&dev_materials, scene->materials.size() * sizeof(Material));
  	hipMemcpy(dev_materials, scene->materials.data(), scene->materials.size() * sizeof(Material), hipMemcpyHostToDevice);

  	hipMalloc(&dev_intersections, pixelcount * sizeof(ShadeableIntersection));
  	hipMemset(dev_intersections, 0, pixelcount * sizeof(ShadeableIntersection));

    // TODO: initialize any extra device memeory you need
    hipMalloc(&dev_triangles, scene->triangles.size() * sizeof(Triangle));
    hipMemcpy(dev_triangles, scene->triangles.data(), scene->triangles.size() * sizeof(Triangle), hipMemcpyHostToDevice);

    checkCUDAError("pathtraceInit");
}

void pathtraceFree() {
    hipFree(dev_image);  // no-op if dev_image is null
  	hipFree(dev_paths);
  	hipFree(dev_geoms);
  	hipFree(dev_materials);
  	hipFree(dev_intersections);

    // TODO: clean up any extra device memory you created
    hipFree(dev_intersections_cache);
    hipFree(dev_triangles);

    checkCUDAError("pathtraceFree");
}

/**
* Generate PathSegments with rays from the camera through the screen into the
* scene, which is the first bounce of rays.
*
* Antialiasing - add rays for sub-pixel sampling
* motion blur - jitter rays "in time"
* lens effect - jitter ray origin positions based on a lens
*/
__global__ void generateRayFromCamera(Camera cam, int iter, int traceDepth, PathSegment* pathSegments)
{
	int x = (blockIdx.x * blockDim.x) + threadIdx.x;
	int y = (blockIdx.y * blockDim.y) + threadIdx.y;

	if (x < cam.resolution.x && y < cam.resolution.y) {
		int index = x + (y * cam.resolution.x);
		PathSegment & segment = pathSegments[index];
    
    segment.ray.origin = cam.position;
    segment.color = glm::vec3(1.0f, 1.0f, 1.0f);

    thrust::default_random_engine rng = makeSeededRandomEngine(iter, index, 0);
    thrust::uniform_real_distribution<float> u01(0, 1);

    // motion blur
    thrust::normal_distribution<float> n01(0, 1);
    float t = abs(n01(rng));
    glm::vec3 view = cam.view * (1 - t) + (cam.view + cam.motion) * t;

    if (cam.antialiasing) {
      segment.ray.direction = glm::normalize(view
        - cam.right * cam.pixelLength.x * ((float)x + u01(rng) - (float)cam.resolution.x * 0.5f)
        - cam.up * cam.pixelLength.y * ((float)y + u01(rng) - (float)cam.resolution.y * 0.5f)
      );
    } else {
      segment.ray.direction = glm::normalize(view
        - cam.right * cam.pixelLength.x * ((float)x - (float)cam.resolution.x * 0.5f)
        - cam.up * cam.pixelLength.y * ((float)y - (float)cam.resolution.y * 0.5f)
      );
    }

    if (cam.depth_of_field) {
      // sample point on lens
      float r = u01(rng) * cam.lens_radius;
      float theta = u01(rng) * 2 * PI;
      glm::vec3 p_lens(r * cos(theta), r * sin(theta), 0.0f);

      // compute point on plane of focus
      float ft = cam.focal_distance / glm::abs(segment.ray.direction.z);
      glm::vec3 p_focus = segment.ray.origin + ft * segment.ray.direction;

      // update ray for effect of lens
      segment.ray.origin += p_lens;
      segment.ray.direction = glm::normalize(p_focus - segment.ray.origin);
    }

		segment.pixelIndex = index;
		segment.remainingBounces = traceDepth;
	}
}

// TODO:
// computeIntersections handles generating ray intersections ONLY.
// Generating new rays is handled in your shader(s).
// Feel free to modify the code below.
__global__ void computeIntersections(
	int depth
	, int num_paths
	, PathSegment * pathSegments
	, Geom * geoms
  , Triangle* triangles
	, int geoms_size
	, ShadeableIntersection * intersections
	)
{
	int path_index = blockIdx.x * blockDim.x + threadIdx.x;

	if (path_index < num_paths)
	{
		PathSegment pathSegment = pathSegments[path_index];

#ifndef STREAM_COMPACTION
    if (pathSegment.remainingBounces <= 0) return;
#endif // !STREAM_COMPACTION

		float t;
		glm::vec3 intersect_point;
		glm::vec3 normal;
		float t_min = FLT_MAX;
		int hit_geom_index = -1;
		bool outside = true;

		glm::vec3 tmp_intersect;
		glm::vec3 tmp_normal;

		// naive parse through global geoms

		for (int i = 0; i < geoms_size; i++)
		{
			Geom & geom = geoms[i];

			if (geom.type == CUBE)
			{
				t = boxIntersectionTest(geom, pathSegment.ray, tmp_intersect, tmp_normal, outside);
			}
			else if (geom.type == SPHERE)
			{
				t = sphereIntersectionTest(geom, pathSegment.ray, tmp_intersect, tmp_normal, outside);
      }
      else if (geom.type == MESH)
      {
        t = meshIntersectionTest(geom, triangles, pathSegment.ray, tmp_intersect, tmp_normal, outside);
      }

			// Compute the minimum t from the intersection tests to determine what
			// scene geometry object was hit first.
			if (t > 0.0f && t_min > t)
			{
				t_min = t;
				hit_geom_index = i;
				intersect_point = tmp_intersect;
				normal = tmp_normal;
			}
		}

		if (hit_geom_index == -1)
		{
			intersections[path_index].t = -1.0f;
		}
		else
		{
			//The ray hits something
			intersections[path_index].t = t_min;
			intersections[path_index].materialId = geoms[hit_geom_index].materialid;
			intersections[path_index].surfaceNormal = normal;
		}
	}
}

// LOOK: "fake" shader demonstrating what you might do with the info in
// a ShadeableIntersection, as well as how to use thrust's random number
// generator. Observe that since the thrust random number generator basically
// adds "noise" to the iteration, the image should start off noisy and get
// cleaner as more iterations are computed.
//
// Note that this shader does NOT do a BSDF evaluation!
// Your shaders should handle that - this can allow techniques such as
// bump mapping.
__global__ void shadeFakeMaterial (
  int iter
  , int num_paths
	, ShadeableIntersection * shadeableIntersections
	, PathSegment * pathSegments
	, Material * materials
	)
{
  int idx = blockIdx.x * blockDim.x + threadIdx.x;
  if (idx < num_paths)
  {
    ShadeableIntersection intersection = shadeableIntersections[idx];
    if (intersection.t > 0.0f) { // if the intersection exists...
      // Set up the RNG
      // LOOK: this is how you use thrust's RNG! Please look at
      // makeSeededRandomEngine as well.
      thrust::default_random_engine rng = makeSeededRandomEngine(iter, idx, 0);
      thrust::uniform_real_distribution<float> u01(0, 1);

      Material material = materials[intersection.materialId];
      glm::vec3 materialColor = material.color;

      // If the material indicates that the object was a light, "light" the ray
      if (material.emittance > 0.0f) {
        pathSegments[idx].color *= (materialColor * material.emittance);
      }
      // Otherwise, do some pseudo-lighting computation. This is actually more
      // like what you would expect from shading in a rasterizer like OpenGL.
      // TODO: replace this! you should be able to start with basically a one-liner
      else {
        float lightTerm = glm::dot(intersection.surfaceNormal, glm::vec3(0.0f, 1.0f, 0.0f));
        pathSegments[idx].color *= (materialColor * lightTerm) * 0.3f + ((1.0f - intersection.t * 0.02f) * materialColor) * 0.7f;
        pathSegments[idx].color *= u01(rng); // apply some noise because why not
      }
    // If there was no intersection, color the ray black.
    // Lots of renderers use 4 channel color, RGBA, where A = alpha, often
    // used for opacity, in which case they can indicate "no opacity".
    // This can be useful for post-processing and image compositing.
    } else {
      pathSegments[idx].color = glm::vec3(0.0f);
    }
  }
}

// The implementation of the real shader
__global__ void shadeRealMaterial(
  int iter
  , int depth
  , int num_paths
  , ShadeableIntersection * shadeableIntersections
  , PathSegment * pathSegments
  , Material * materials
)
{
  int idx = blockIdx.x * blockDim.x + threadIdx.x;
  if (idx < num_paths)
  {
    ShadeableIntersection intersection = shadeableIntersections[idx];
    PathSegment &pathSegment = pathSegments[idx];

#ifndef STREAM_COMPACTION
    if (pathSegment.remainingBounces <= 0) return;
#endif // !STREAM_COMPACTION

    if (intersection.t > 0.0f) { // if the intersection exists...

      // Set up the RNG
      thrust::default_random_engine rng = makeSeededRandomEngine(iter, idx, depth);

      Material material = materials[intersection.materialId];
      glm::vec3 materialColor = material.color;

      if (material.emittance > 0.0f) {  // Hit light (Terminate)
        pathSegment.color *= (materialColor * material.emittance);
        pathSegment.remainingBounces = -1;
      }
      else {  // Hit Material (Bounce)
        glm::vec3 intersect = pathSegment.ray.origin + intersection.t * pathSegment.ray.direction;
        scatterRay(pathSegment, intersect, intersection.surfaceNormal, material, rng);
      }
    }
    else {    // No Intersection (Terminate)
      pathSegment.color = glm::vec3(0.0f);
      pathSegment.remainingBounces = -1;
    }
  }
}

// Add the current iteration's output to the overall image
__global__ void finalGather(int nPaths, glm::vec3 * image, PathSegment * iterationPaths)
{
	int index = (blockIdx.x * blockDim.x) + threadIdx.x;

	if (index < nPaths)
	{
		PathSegment iterationPath = iterationPaths[index];
		image[iterationPath.pixelIndex] += iterationPath.color;
	}
}

/**
 * Wrapper for the __global__ call that sets up the kernel calls and does a ton
 * of memory management
 */
void pathtrace(uchar4 *pbo, int frame, int iter) {
    const int traceDepth = hst_scene->state.traceDepth;
    const Camera &cam = hst_scene->state.camera;
    const int pixelcount = cam.resolution.x * cam.resolution.y;

	  // 2D block for generating ray from camera
    const dim3 blockSize2d(8, 8);
    const dim3 blocksPerGrid2d(
            (cam.resolution.x + blockSize2d.x - 1) / blockSize2d.x,
            (cam.resolution.y + blockSize2d.y - 1) / blockSize2d.y);

	  // 1D block for path tracing
	  const int blockSize1d = 128;

    ///////////////////////////////////////////////////////////////////////////

    // Recap:
    // * Initialize array of path rays (using rays that come out of the camera)
    //   * You can pass the Camera object to that kernel.
    //   * Each path ray must carry at minimum a (ray, color) pair,
    //   * where color starts as the multiplicative identity, white = (1, 1, 1).
    //   * This has already been done for you.
    // * For each depth:
    //   * Compute an intersection in the scene for each path ray.
    //     A very naive version of this has been implemented for you, but feel
    //     free to add more primitives and/or a better algorithm.
    //     Currently, intersection distance is recorded as a parametric distance,
    //     t, or a "distance along the ray." t = -1.0 indicates no intersection.
    //     * Color is attenuated (multiplied) by reflections off of any object
    //   * TODO: Stream compact away all of the terminated paths.
    //     You may use either your implementation or `thrust::remove_if` or its
    //     cousins.
    //     * Note that you can't really use a 2D kernel launch any more - switch
    //       to 1D.
    //   * TODO: Shade the rays that intersected something or didn't bottom out.
    //     That is, color the ray by performing a color computation according
    //     to the shader, then generate a new ray to continue the ray path.
    //     We recommend just updating the ray's PathSegment in place.
    //     Note that this step may come before or after stream compaction,
    //     since some shaders you write may also cause a path to terminate.
    // * Finally, add this iteration's results to the image. This has been done
    //   for you.

    // TODO: perform one iteration of path tracing

	  generateRayFromCamera <<<blocksPerGrid2d, blockSize2d >>>(cam, iter, traceDepth, dev_paths);
	  checkCUDAError("generate camera ray");

	  int depth = 0;
	  PathSegment* dev_path_end = dev_paths + pixelcount;
	  int num_paths = dev_path_end - dev_paths;

	  // --- PathSegment Tracing Stage ---
	  // Shoot ray into scene, bounce between objects, push shading chunks
    time_start = std::chrono::high_resolution_clock::now();

    bool iterationComplete = false;
	  while (!iterationComplete) {

	      // clean shading chunks
	      hipMemset(dev_intersections, 0, pixelcount * sizeof(ShadeableIntersection));

	      // path trace
	      dim3 numblocksPathSegmentTracing = (num_paths + blockSize1d - 1) / blockSize1d;
#if defined(CACHE_FIRST_ITERATION)
        if (iter != 0 || (iter == 0 && depth == 0) || cam.antialiasing) {
          computeIntersections<<<numblocksPathSegmentTracing, blockSize1d>>> (
            depth
            , num_paths
            , dev_paths
            , dev_geoms
            , dev_triangles
            , hst_scene->geoms.size()
            , dev_intersections
            );
          checkCUDAError("trace one bounce");
          if (iter == 0 && !cam.antialiasing) {      // bulid cache
            hipMemcpy(dev_intersections_cache, dev_intersections, pixelcount * sizeof(ShadeableIntersection), hipMemcpyDeviceToDevice);
          }
        } else {                // use cache
          hipMemcpy(dev_intersections, dev_intersections_cache, pixelcount * sizeof(ShadeableIntersection), hipMemcpyDeviceToDevice);
        }
#else
        computeIntersections<<<numblocksPathSegmentTracing, blockSize1d>>> (
          depth
          , num_paths
          , dev_paths
          , dev_geoms
          , dev_triangles
          , hst_scene->geoms.size()
          , dev_intersections
          );
        checkCUDAError("trace one bounce");
#endif // CACHE_FIRST_ITERATION
        hipDeviceSynchronize();
	      depth++;

	      // TODO:
	      // --- Shading Stage ---
	      // Shade path segments based on intersections and generate new rays by
        // evaluating the BSDF.
        // Start off with just a big kernel that handles all the different
        // materials you have in the scenefile.
        // TODO: compare between directly shading the path segments and shading
        // path segments that have been reshuffled to be contiguous in memory.

#ifdef SORT_BY_MATERIALS
        thrust::sort_by_key(thrust::device, dev_intersections, dev_intersections + num_paths, dev_paths, material_id_comparator());
#endif // SORT_BY_MATERIALS

        // do shading
        shadeRealMaterial<<<numblocksPathSegmentTracing, blockSize1d>>> (
          iter,
          depth,
          num_paths,
          dev_intersections,
          dev_paths,
          dev_materials
        );
        hipDeviceSynchronize();

#ifdef STREAM_COMPACTION
        dev_path_end = thrust::partition(thrust::device, dev_paths, dev_paths + num_paths, ray_continuation_condition());
        num_paths = dev_path_end - dev_paths;
        iterationComplete = (num_paths <= 0);
#else
        iterationComplete = (depth >= traceDepth);
#endif  // STREAM_COMPACTION
	  }

    time_end = std::chrono::high_resolution_clock::now();
    std::chrono::duration<double, std::milli> duro = time_end - time_start;
    float prev_elapsed_time_cpu_milliseconds =
      static_cast<decltype(prev_elapsed_time_cpu_milliseconds)>(duro.count());
    avg_time = (avg_time * (iter - 1) + prev_elapsed_time_cpu_milliseconds) / (iter);
    cout << "Iter:" << iter << ", Time:" << prev_elapsed_time_cpu_milliseconds << ", Avg Time:" << avg_time << endl;

    // Assemble this iteration and apply it to the image
    dim3 numBlocksPixels = (pixelcount + blockSize1d - 1) / blockSize1d;
	  finalGather<<<numBlocksPixels, blockSize1d>>>(pixelcount, dev_image, dev_paths);

    ///////////////////////////////////////////////////////////////////////////

    // Send results to OpenGL buffer for rendering
    sendImageToPBO<<<blocksPerGrid2d, blockSize2d>>>(pbo, cam.resolution, iter, dev_image);

    // Retrieve image from GPU
    hipMemcpy(hst_scene->state.image.data(), dev_image,
            pixelcount * sizeof(glm::vec3), hipMemcpyDeviceToHost);

    checkCUDAError("pathtrace");

}
