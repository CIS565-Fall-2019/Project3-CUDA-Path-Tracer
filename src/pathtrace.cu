#include "hip/hip_runtime.h"
#include <cstdio>
#include <hip/hip_runtime.h>
#include <cmath>
#include <thrust/execution_policy.h>
#include <thrust/random.h>
#include <thrust/remove.h>
#include <thrust/partition.h>
#include <>
#include <hip/device_functions.h>
#include <texture_indirect_functions.h>

#include "sceneStructs.h"
#include "scene.h"
#include "glm/glm.hpp"
#include "glm/gtx/norm.hpp"
#include "utilities.h"
#include "pathtrace.h"
#include "intersections.h"
#include "interactions.h"

#define ERRORCHECK 1



#define FILENAME (strrchr(__FILE__, '/') ? strrchr(__FILE__, '/') + 1 : __FILE__)
#define checkCUDAError(msg) checkCUDAErrorFn(msg, FILENAME, __LINE__)
void checkCUDAErrorFn(const char *msg, const char *file, int line) {
#if ERRORCHECK
    hipDeviceSynchronize();
    hipError_t err = hipGetLastError();
    if (hipSuccess == err) {
        return;
    }

    fprintf(stderr, "CUDA error");
    if (file) {
        fprintf(stderr, " (%s:%d)", file, line);
    }
    fprintf(stderr, ": %d: %s: %s\n", line, msg, hipGetErrorString(err));
#  ifdef _WIN32
    getchar();
#  endif
    exit(EXIT_FAILURE);
#endif
}



__host__ __device__
thrust::default_random_engine makeSeededRandomEngine(int iter, int index, int depth) {
    int h = utilhash((1 << 31) | (depth << 22) | iter) ^ utilhash(index);
    return thrust::default_random_engine(h);
}

//Kernel that writes the image to the OpenGL PBO directly.
__global__ void sendImageToPBO(uchar4* pbo, glm::ivec2 resolution,
        int iter, glm::vec3* image) {
    int x = (blockIdx.x * blockDim.x) + threadIdx.x;
    int y = (blockIdx.y * blockDim.y) + threadIdx.y;

    if (x < resolution.x && y < resolution.y) {
        int index = x + (y * resolution.x);
        glm::vec3 pix = image[index];

        glm::ivec3 color;
        color.x = glm::clamp((int) (pix.x / iter * 255.0), 0, 255);
        color.y = glm::clamp((int) (pix.y / iter * 255.0), 0, 255);
        color.z = glm::clamp((int) (pix.z / iter * 255.0), 0, 255);

        // Each thread writes one pixel location in the texture (textel)
        pbo[index].w = 0;
        pbo[index].x = color.x;
        pbo[index].y = color.y;
        pbo[index].z = color.z;
    }
}

static hipTextureObject_t texobj0, texobj1, texobj2, texobj3, texobj4, texobj5, texobj6, texobj7;

typedef struct textureWrapper {
	hipTextureObject_t contents[8];
} TextureWrapper;

static TextureWrapper texObjects = { texobj0, texobj1, texobj2, texobj3,
									 texobj4, texobj5, texobj6, texobj7 };


static Scene * hst_scene = NULL;
static glm::vec3 * dev_image = NULL;
static Geom * dev_geoms = NULL;
static Triangle* dev_tris = NULL;
static Material * dev_materials = NULL;
static PathSegment* dev_paths = NULL;
static ShadeableIntersection* dev_intersections = NULL;
static ShadeableIntersection* dev_intersections_first = NULL;
// TODO: static variables for device memory, any extra info you need, etc
// ...

void pathtraceInit(Scene *scene) {
    hst_scene = scene;
    const Camera &cam = hst_scene->state.camera;
    const int pixelcount = cam.resolution.x * cam.resolution.y;

    hipMalloc(&dev_image, pixelcount * sizeof(glm::vec3));
    hipMemset(dev_image, 0, pixelcount * sizeof(glm::vec3));

  	hipMalloc(&dev_paths, pixelcount * sizeof(PathSegment));

  	hipMalloc(&dev_geoms, scene->geoms.size() * sizeof(Geom));
  	hipMemcpy(dev_geoms, scene->geoms.data(), scene->geoms.size() * sizeof(Geom), hipMemcpyHostToDevice);

	hipMalloc(&dev_tris, scene->triangles.size() * sizeof(Triangle));
	hipMemcpy(dev_tris, scene->triangles.data(), scene->triangles.size() * sizeof(Triangle), hipMemcpyHostToDevice);

  	hipMalloc(&dev_materials, scene->materials.size() * sizeof(Material));
  	hipMemcpy(dev_materials, scene->materials.data(), scene->materials.size() * sizeof(Material), hipMemcpyHostToDevice);

  	hipMalloc(&dev_intersections, pixelcount * sizeof(ShadeableIntersection));
  	hipMemset(dev_intersections, 0, pixelcount * sizeof(ShadeableIntersection));

	hipMalloc(&dev_intersections_first, pixelcount * sizeof(ShadeableIntersection));
	hipMemset(dev_intersections_first, 0, pixelcount * sizeof(ShadeableIntersection));

	for (int i = 0; i < scene->textures.size(); i++) {
		scene->textures[i].putOntoDevice(i);
	}//for
    // TODO: initialize any extra device memeory you need

    checkCUDAError("pathtraceInit");

}

void pathtraceFree(Scene* scene) {
    hipFree(dev_image);  // no-op if dev_image is null
  	hipFree(dev_paths);
  	hipFree(dev_geoms);
	hipFree(dev_tris);
  	hipFree(dev_materials);
  	hipFree(dev_intersections);
	hipFree(dev_intersections_first);

	for (int i = 0; i < scene->textures.size(); i++) {
		scene->textures[i].freeFromDevice(i);
	}//for

	//TODO: free textures off device?

    checkCUDAError("pathtraceFree");
}

/**
* Generate PathSegments with rays from the camera through the screen into the
* scene, which is the first bounce of rays.
*
* Antialiasing - add rays for sub-pixel sampling
* motion blur - jitter rays "in time"
* lens effect - jitter ray origin positions based on a lens
*/
__global__ void generateRayFromCamera(Camera cam, int iter, int traceDepth, PathSegment* pathSegments)
{
	int x = (blockIdx.x * blockDim.x) + threadIdx.x;
	int y = (blockIdx.y * blockDim.y) + threadIdx.y;


	if (x < cam.resolution.x && y < cam.resolution.y) {

		int index = x + (y * cam.resolution.x);
		PathSegment & segment = pathSegments[index];

		segment.ray.origin = cam.position;
		segment.color = glm::vec3(1.0f, 1.0f, 1.0f);
		segment.curIOR = 1.0;
		

		float xfloat = (float)x;
		float yfloat = (float)y;

#if ANTIALIASING
		thrust::default_random_engine rng = makeSeededRandomEngine(iter, index, 0);
		thrust::uniform_real_distribution<float> u01(-0.5, 0.5);
		float xFac = u01(rng);
		float yFac = u01(rng);
		xfloat += xFac;
		yfloat += yFac;
#endif

		segment.ray.direction = glm::normalize(cam.view
			- cam.right * cam.pixelLength.x * (xfloat - (float)cam.resolution.x * 0.5f)
			- cam.up * cam.pixelLength.y * (yfloat - (float)cam.resolution.y * 0.5f)
			);


		segment.pixelIndex = index;
		segment.remainingBounces = traceDepth;
	}
}

// TODO:
// computeIntersections handles generating ray intersections ONLY.
// Generating new rays is handled in your shader(s).
// Feel free to modify the code below.
__global__ void computeIntersections(
		int depth,
		int num_paths,
		PathSegment * pathSegments,
		Geom * geoms,
		Triangle* tris,
		int geoms_size,
		ShadeableIntersection * intersections){

	int path_index = blockIdx.x * blockDim.x + threadIdx.x;
	if (path_index >= num_paths) return;

	PathSegment& pathSegment = pathSegments[path_index];

	float t;
	gvec3 intersect_point;
	gvec3 normal;
	float t_min = FLT_MAX;
	int hit_geom_index = -1;
	int hit_tri_index = -1;
	bool outside = true;

	bool tmp_outside = true;
	gvec3 tmp_intersect;
	gvec3 tmp_normal;
	int tmp_tri_index;
	float2 tmp_uv = { -1.0, -1.0 };
	float2 min_uv = { -1.0, -1.0 };

	// naive parse through global geoms

	for (int i = 0; i < geoms_size; i++) {
		Geom& geom = geoms[i];
		tmp_uv = { -1.0, -1.0 };

		if (geom.type == CUBE) {
			t = boxIntersectionTest(geom, pathSegment.ray, tmp_intersect, tmp_normal, tmp_outside, &tmp_uv);
		}
		else if (geom.type == SPHERE) {
			t = sphereIntersectionTest(geom, pathSegment.ray, tmp_intersect, tmp_normal, tmp_outside, &tmp_uv);
		}
		else if (geom.type == MESH) {
			t = meshIntersectionTest(geom, pathSegment.ray, tmp_intersect, tmp_normal, tmp_outside, tris, &tmp_tri_index, &tmp_uv);
		}
		/*
		else if (geom.type == TRIANGLE) {
			t = triangleIntersectionTest(geom, pathSegment.ray, tmp_intersect, tmp_normal);
		}
		*/

		// Compute the minimum t from the intersection tests to determine what
		// scene geometry object was hit first.
		if (t > 0.0f && t_min > t) {
			t_min = t;
			hit_geom_index = i;
			intersect_point = tmp_intersect;
			normal = tmp_normal;
			outside = tmp_outside;
			min_uv = tmp_uv;
			if (geom.type == MESH) {
				hit_tri_index = tmp_tri_index;
			}
			else {
				hit_tri_index = -1;
			}
		}
	}//for each geom

	if (hit_geom_index == -1) {
		intersections[path_index].t = -1.0f;
	}
	else {
		//The ray hits something
		intersections[path_index].t = t_min;
		intersections[path_index].surfaceNormal = normal;
		intersections[path_index].leaving = !outside;
		intersections[path_index].uv = min_uv;
		if (hit_tri_index > -1) {
			int myMaterial = tris[hit_tri_index].materialid;
			intersections[path_index].materialId = myMaterial;
		}//if
		else {
			intersections[path_index].materialId = geoms[hit_geom_index].materialid;
		}//else
	}
}

__global__ void shadeRealMaterial(
	int iter,
	int num_paths,
	ShadeableIntersection* shadeableIntersections,
	PathSegment* pathSegments,
	Material* materials, TextureWrapper twrap) {
	int idx = blockIdx.x * blockDim.x + threadIdx.x;

	hipTextureObject_t textureReference;

	if (idx >= num_paths) return;

	ShadeableIntersection intersection = shadeableIntersections[idx];
	PathSegment* incoming = &pathSegments[idx];
	if (intersection.t > 0.0f) { // if the intersection exists...
	  // Set up the RNG

		thrust::default_random_engine rng = makeSeededRandomEngine(iter, idx, 0);
		thrust::uniform_real_distribution<float> u01(0, 1);

		Material material = materials[intersection.materialId];
		gvec3 materialColor = material.color;

		if (material.textureId >= 0) {
			textureReference = twrap.contents[material.textureId];
		}

		// If the material indicates that the object was a light, "light" the ray

		if (material.emittance > 0.0f) {

#if TEX_EMISSIVE
			if (material.textureMask & TEXTURE_EMISSIVE) {
				float4 emissiveText = tex2DLayered<float4>(textureReference, intersection.uv.x, intersection.uv.y, TEXTURE_LAYER_EMISSIVE);
				gvec3 emissiveColor = gvec3(emissiveText.x, emissiveText.y, emissiveText.z);
				if (glm::length(emissiveColor) > 0.04) {
					incoming->color *= emissiveColor * material.emittance;
					incoming->remainingBounces = 0;//stop bouncing here!
					return;
				}//if we're emitting light
			}//checking for emissive
			else {
#endif
				incoming->color *= (materialColor * material.emittance);
				incoming->remainingBounces = 0;//stop bouncing here!
				return;
#if TEX_EMISSIVE
			}
#endif
		}

		incoming->remainingBounces--;

		scatterRay(*incoming, 
					getPointOnRayEp(incoming->ray, intersection.t), 
					intersection.surfaceNormal, 
					material, 
					intersection.leaving, 
					intersection.uv, 
					textureReference, rng);


	}//if we have an intersection
	else {
		incoming->color = gvec3(0.0f);
		incoming->remainingBounces = 0;
	}//no hit
}//shadeRealMaterial


// Add the current iteration's output to the overall image
__global__ void finalGather(int nPaths, glm::vec3 * image, PathSegment * iterationPaths)
{
	int index = (blockIdx.x * blockDim.x) + threadIdx.x;

	if (index < nPaths){
		PathSegment iterationPath = iterationPaths[index];
		image[iterationPath.pixelIndex] += iterationPath.color;
	}
}

/*thrust construct for telling whether we're out of bounces*/
struct hasRemainingBounces {
	__host__ __device__ bool operator()(const PathSegment x) {
		return x.remainingBounces != 0;
	}
};

/*thrust construct for sorting by material id*/
struct materialIdLess {
	__host__ __device__ bool operator()(const ShadeableIntersection& a, const ShadeableIntersection& b) {
		if (a.t < 0 && b.t > 0) return true;//might be extraneous
		if (b.t < 0 && a.t > 0) return false;//might be extraneous
		return(a.materialId < b.materialId);
	}
};

/**
 Wrapper for the __global__ call that sets up the kernel calls and does a ton
 of memory management
 */
void pathtrace(uchar4* pbo, int frame, int iter) {
	const int traceDepth = hst_scene->state.traceDepth;
	const Camera& cam = hst_scene->state.camera;
	const int pixelcount = cam.resolution.x * cam.resolution.y;

	// 2D block for generating ray from camera
	const dim3 blockSize2d(8, 8);
	const dim3 blocksPerGrid2d(
		(cam.resolution.x + blockSize2d.x - 1) / blockSize2d.x,
		(cam.resolution.y + blockSize2d.y - 1) / blockSize2d.y);

	// 1D block for path tracing
	const int blockSize1d = 128;


	//makes our initial path segments in dev_paths; contains the ray, a color, a pixelIndex, and bounce count for each
	generateRayFromCamera <<<blocksPerGrid2d, blockSize2d >>> (cam, iter, traceDepth, dev_paths);
	checkCUDAError("generate camera ray");

	int depth = 0;
	PathSegment* dev_path_end = dev_paths + pixelcount;
	int num_paths = dev_path_end - dev_paths;//ok, clever use of pointer math...

	int total_paths = num_paths;
	// --- PathSegment Tracing Stage ---
	// Shoot ray into scene, bounce between objects, push shading chunks

	bool iterationComplete = false;
	while (!iterationComplete) {

		// clean shading chunks
		hipMemset(dev_intersections, 0, pixelcount * sizeof(ShadeableIntersection));

#if CACHING_FIRST && !ANTIALIASING
		// tracing
		dim3 numblocksPathSegmentTracing = (num_paths + blockSize1d - 1) / blockSize1d;
		if (depth == 0 && iter == 1) {
			computeIntersections <<<numblocksPathSegmentTracing, blockSize1d >>> (
				depth,
				num_paths,
				dev_paths,
				dev_geoms,
				dev_tris,
				hst_scene->geoms.size(),
				dev_intersections_first);
			checkCUDAError("trace one bounce");
			hipDeviceSynchronize();
			hipMemcpy(dev_intersections, dev_intersections_first, num_paths * sizeof(ShadeableIntersection), hipMemcpyDeviceToDevice);
		}//if the first set of intersections
		else if (depth == 0) {
			hipMemcpy(dev_intersections, dev_intersections_first, num_paths * sizeof(ShadeableIntersection), hipMemcpyDeviceToDevice);
		}//use the cached intersections
		else {
			computeIntersections << <numblocksPathSegmentTracing, blockSize1d >> > (
				depth,
				num_paths,
				dev_paths,
				dev_geoms,
				dev_tris,
				hst_scene->geoms.size(),
				dev_intersections);
			checkCUDAError("trace one bounce");
			hipDeviceSynchronize();
		}
#else
		// tracing
		dim3 numblocksPathSegmentTracing = (num_paths + blockSize1d - 1) / blockSize1d;
		computeIntersections << <numblocksPathSegmentTracing, blockSize1d >> > (
			depth,
			num_paths,
			dev_paths,
			dev_geoms,
			dev_tris,
			hst_scene->geoms.size(),
			dev_intersections);
		checkCUDAError("trace one bounce");
		hipDeviceSynchronize();
#endif

		depth++;

#if SORTING_MATERIAL
		thrust::sort_by_key(thrust::device, dev_intersections, dev_intersections + num_paths, dev_paths, materialIdLess());
#endif

		shadeRealMaterial <<<numblocksPathSegmentTracing, blockSize1d >>> (
			iter,
			num_paths,
			dev_intersections,
			dev_paths,
			dev_materials,
			texObjects);
		checkCUDAError("shadeRealMaterial");
		hipDeviceSynchronize();

		PathSegment* newEnd = thrust::partition(thrust::device, dev_paths, dev_paths + num_paths, hasRemainingBounces());
		num_paths = newEnd - dev_paths;


		if (num_paths == 0) iterationComplete = true;
		//iterationComplete = true; // TODO: should be based off stream compaction results.
	}//while !iterationComplete

	// Assemble this iteration and apply it to the image
	dim3 numBlocksPixels = (pixelcount + blockSize1d - 1) / blockSize1d;
	finalGather <<<numBlocksPixels, blockSize1d >>> (total_paths, dev_image, dev_paths);
	checkCUDAError("finalGather");

	///////////////////////////////////////////////////////////////////////////

	// Send results to OpenGL buffer for rendering
	sendImageToPBO <<<blocksPerGrid2d, blockSize2d >>> (pbo, cam.resolution, iter, dev_image);

	// Retrieve image from GPU
	hipMemcpy(hst_scene->state.image.data(), dev_image,
		pixelcount * sizeof(glm::vec3), hipMemcpyDeviceToHost);

	checkCUDAError("pathtrace");
}
hipArray* Texture::putOntoDevice(int textureIndex) {
	hipChannelFormatDesc f4 = hipCreateChannelDesc<float4>();
	hipExtent extents = make_hipExtent(width, height, 4);
	hipMalloc3DArray(&cu_3darray, &f4, extents, hipArrayLayered);
	hipError_t err = hipGetLastError();


	float4* h_data = (float4*)malloc(width * height * 4 * sizeof(float4));
	fillIntoF4Array(h_data);

	hipMemcpy3DParms myparms = { 0 };
	myparms.srcPos = make_hipPos(0, 0, 0);
	myparms.dstPos = make_hipPos(0, 0, 0);
	myparms.srcPtr = make_hipPitchedPtr(h_data, width * sizeof(float4), width, height);
	myparms.dstArray = cu_3darray;
	myparms.extent = extents;
	myparms.kind = hipMemcpyHostToDevice;
	hipMemcpy3D(&myparms);
	err = hipGetLastError();
	if (err != hipSuccess) {
		printf("Error on the 3d memcpy! Err %d\n", err);
		exit(-1);
	}
	//check cuda error

	hipResourceDesc    texRes;
	memset(&texRes, 0, sizeof(hipResourceDesc));
	texRes.resType = hipResourceTypeArray;
	texRes.res.array.array = cu_3darray;
	hipTextureDesc texDescr;
	memset(&texDescr, 0, sizeof(hipTextureDesc));
	texDescr.normalizedCoords = true;
	texDescr.filterMode = hipFilterModeLinear;
	texDescr.addressMode[0] = hipAddressModeClamp;
	texDescr.addressMode[1] = hipAddressModeClamp;
	texDescr.addressMode[2] = hipAddressModeClamp;
	texDescr.readMode = hipReadModeElementType;

	hipCreateTextureObject(&(texObjects.contents[textureIndex]), &texRes, &texDescr, NULL);
	//hipCreateTextureObject(&texobj0, &texRes, &texDescr, NULL);
	err = hipGetLastError();
	if (err != hipSuccess) {
		printf("Error on the creating texture objects! Err %d\n", err);
		exit(-1);
	}

	free(h_data);//no need to keep it locally anymore

	return cu_3darray;
}//putOntoDevice

void Texture::freeFromDevice(int textureIndex) {
	hipDestroyTextureObject(texObjects.contents[textureIndex]);
	hipFreeArray(cu_3darray);
}//freeFromDevice