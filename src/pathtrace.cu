#include "hip/hip_runtime.h"
#include <cstdio>
#include <hip/hip_runtime.h>
#include <cmath>
#include <thrust/execution_policy.h>
#include <thrust/random.h>
#include <thrust/remove.h>
#include <thrust/partition.h>

#include "sceneStructs.h"
#include "scene.h"
#include "glm/glm.hpp"
#include "glm/gtx/norm.hpp"
#include <glm/gtc/matrix_inverse.hpp>
#include <glm/gtc/matrix_transform.hpp>
#include "utilities.h"
#include "pathtrace.h"
#include "intersections.h"
#include "interactions.h"
#include "efficient.h"

#define ERRORCHECK 1


#define MOTION_BLUR 0
#define ANTI_ALIAS 1
#define STREAM_COMPACT 0
#define STREAM_COMPACT_THRUST 1 // toggle either stream compact
#define SORT_BY_MATERIAL 0

#define FILENAME (strrchr(__FILE__, '/') ? strrchr(__FILE__, '/') + 1 : __FILE__)
#define checkCUDAError(msg) checkCUDAErrorFn(msg, FILENAME, __LINE__)
void checkCUDAErrorFn(const char *msg, const char *file, int line) {
#if ERRORCHECK
	hipDeviceSynchronize();
	hipError_t err = hipGetLastError();
	if (hipSuccess == err) {
		return;
	}

	fprintf(stderr, "CUDA error");
	if (file) {
		fprintf(stderr, " (%s:%d)", file, line);
	}
	fprintf(stderr, ": %s: %s\n", msg, hipGetErrorString(err));
#  ifdef _WIN32
	getchar();
#  endif
	exit(EXIT_FAILURE);
#endif
}

__device__ glm::mat4 buildTransformationMatrix(glm::vec3 translation, glm::vec3 rotation, glm::vec3 scale) {
	glm::mat4 translationMat = glm::translate(glm::mat4(), translation);
	glm::mat4 rotationMat = glm::rotate(glm::mat4(), rotation.x * (float)PI / 180, glm::vec3(1, 0, 0));
	rotationMat = rotationMat * glm::rotate(glm::mat4(), rotation.y * (float)PI / 180, glm::vec3(0, 1, 0));
	rotationMat = rotationMat * glm::rotate(glm::mat4(), rotation.z * (float)PI / 180, glm::vec3(0, 0, 1));
	glm::mat4 scaleMat = glm::scale(glm::mat4(), scale);
	return translationMat * rotationMat * scaleMat;
}

__host__ __device__
thrust::default_random_engine makeSeededRandomEngine(int iter, int index, int depth) {
	int h = utilhash((1 << 31) | (depth << 22) | iter) ^ utilhash(index);
	return thrust::default_random_engine(h);
}

__global__ void kernMotionBlur(int n, Geom* dev_geoms, int iter) {

	int idx = blockIdx.x * blockDim.x + threadIdx.x;

	if (idx > n) return;

	
	if (dev_geoms[idx].type == SPHERE) {
		//printf("%f %f %f\n", dev_geoms[idx].translation.x, dev_geoms[idx].translation.y, dev_geoms[idx].translation.z);
		float vel = 0.001;
		dev_geoms[idx].translation -= glm::vec3(vel);
		dev_geoms[idx].transform = buildTransformationMatrix(dev_geoms[idx].translation, dev_geoms[idx].rotation, dev_geoms[idx].scale);
		
		//float vel = 0.01;
		//dev_geoms[idx].transform = dev_geoms[idx].initialTransform + glm::mat4(
		//	1.0, 0.0, 0.0, iter*vel,
		//	0.0, 1.0, 0.0, iter*vel,
		//	0.0, 0.0, 1.0, 0.0,
		//	0.0, 0.0, 0.0, 1.0) * dev_geoms[idx].transform;

		dev_geoms[idx].inverseTransform = glm::inverse(dev_geoms[idx].transform);
		dev_geoms[idx].invTranspose = glm::inverseTranspose(dev_geoms[idx].transform);
	}
}


//Kernel that writes the image to the OpenGL PBO directly.
__global__ void sendImageToPBO(uchar4* pbo, glm::ivec2 resolution,
		int iter, glm::vec3* image) {
	int x = (blockIdx.x * blockDim.x) + threadIdx.x;
	int y = (blockIdx.y * blockDim.y) + threadIdx.y;

	if (x < resolution.x && y < resolution.y) {
		int index = x + (y * resolution.x);
		glm::vec3 pix = image[index];

		glm::ivec3 color;
		color.x = glm::clamp((int) (pix.x / iter * 255.0), 0, 255);
		color.y = glm::clamp((int) (pix.y / iter * 255.0), 0, 255);
		color.z = glm::clamp((int) (pix.z / iter * 255.0), 0, 255);

		// Each thread writes one pixel location in the texture (textel)
		pbo[index].w = 0;
		pbo[index].x = color.x;
		pbo[index].y = color.y;
		pbo[index].z = color.z;
	}
}

static Scene * hst_scene = NULL;
static glm::vec3 * dev_image = NULL;
static Geom * dev_geoms = NULL;
static Material * dev_materials = NULL;
static PathSegment * dev_paths = NULL;
static int* dev_alive_paths = NULL;
static ShadeableIntersection * dev_intersections = NULL;
static ShadeableIntersection * dev_first_intersections = NULL;
static ShadeableIntersection * dev_intersections_orig = NULL;
// TODO: static variables for device memory, any extra info you need, etc
// ...

void pathtraceInit(Scene *scene) {
	hst_scene = scene;
	const Camera &cam = hst_scene->state.camera;
	const int pixelcount = cam.resolution.x * cam.resolution.y;

	hipMalloc(&dev_image, pixelcount * sizeof(glm::vec3));
	hipMemset(dev_image, 0, pixelcount * sizeof(glm::vec3));

	hipMalloc(&dev_paths, pixelcount * sizeof(PathSegment));
	hipMalloc(&dev_alive_paths, pixelcount * sizeof(int));

	hipMalloc(&dev_geoms, scene->geoms.size() * sizeof(Geom));
	hipMemcpy(dev_geoms, scene->geoms.data(), scene->geoms.size() * sizeof(Geom), hipMemcpyHostToDevice);

	hipMalloc(&dev_materials, scene->materials.size() * sizeof(Material));
	hipMemcpy(dev_materials, scene->materials.data(), scene->materials.size() * sizeof(Material), hipMemcpyHostToDevice);

	hipMalloc(&dev_intersections, pixelcount * sizeof(ShadeableIntersection));
	hipMemset(dev_intersections, 0, pixelcount * sizeof(ShadeableIntersection));


	hipMalloc(&dev_first_intersections, pixelcount * sizeof(ShadeableIntersection));
	hipMemset(dev_first_intersections, 0, pixelcount * sizeof(ShadeableIntersection));

	checkCUDAError("pathtraceInit");
}

void pathtraceFree() {
	hipFree(dev_image);  // no-op if dev_image is null
	hipFree(dev_paths);
	hipFree(dev_geoms);
	hipFree(dev_materials);
	hipFree(dev_intersections);
	hipFree(dev_first_intersections);

	checkCUDAError("pathtraceFree");
}

/**
* Generate PathSegments with rays from the camera through the screen into the
* scene, which is the first bounce of rays.
*
* Antialiasing - add rays for sub-pixel sampling
* motion blur - jitter rays "in time"
* lens effect - jitter ray origin positions based on a lens
*/
__global__ void generateRayFromCamera(Camera cam, int iter, int traceDepth, PathSegment* pathSegments, int* alive_paths)
{
	int x = (blockIdx.x * blockDim.x) + threadIdx.x;
	int y = (blockIdx.y * blockDim.y) + threadIdx.y;



	if (x < cam.resolution.x && y < cam.resolution.y) {
		int index = x + (y * cam.resolution.x);


		PathSegment & segment = pathSegments[index];
		alive_paths[index] = index;

		thrust::default_random_engine rng = makeSeededRandomEngine(iter, index, segment.remainingBounces);
		thrust::uniform_real_distribution<float> u01(-0.5f, 0.5f);

		segment.ray.origin = cam.position;
		segment.color = glm::vec3(1.0f, 1.0f, 1.0f);

		// antialiasing by jittering
		//float x_shift = x + u01(rng);
		//float y_shift = y + u01(rng);
		float x_shift = x;
		float y_shift = y;

#if ANTI_ALIAS
		x_shift = x + u01(rng);
		y_shift = y + u01(rng);
#endif
		

		segment.ray.direction = glm::normalize(cam.view
			- cam.right * cam.pixelLength.x * (x_shift - (float)cam.resolution.x * 0.5f)
			- cam.up * cam.pixelLength.y * (y_shift - (float)cam.resolution.y * 0.5f)
		);

		segment.pixelIndex = index;
		segment.remainingBounces = traceDepth;
	}
}

// computeIntersections handles generating ray intersections ONLY.
// Generating new rays is handled in your shader(s).
// Feel free to modify the code below.
__global__ void computeIntersections(
	int depth
	, int num_paths
	, PathSegment * pathSegments
	, int* alive_paths
	, Geom * geoms
	, int geoms_size
	, ShadeableIntersection * intersections,
	int iter)
{
	int alive_idx = blockIdx.x * blockDim.x + threadIdx.x;

	if (alive_idx < num_paths)
	{
		int path_index = alive_paths[alive_idx];
		if (path_index < 0) return;
		PathSegment pathSegment = pathSegments[path_index];

		float t;
		glm::vec3 intersect_point;
		glm::vec3 normal;
		float t_min = FLT_MAX;
		int hit_geom_index = -1;
		bool outside = true;

		glm::vec3 tmp_intersect;
		glm::vec3 tmp_normal;

		// naive parse through global geoms

		for (int i = 0; i < geoms_size; i++)
		{
			Geom & geom = geoms[i];

			if (geom.type == CUBE)
			{
				t = boxIntersectionTest(geom, pathSegment.ray, tmp_intersect, tmp_normal, outside);
			}
			else if (geom.type == SPHERE)
			{
				// Motion Blur atttempt
				//float vel = 0.0001;
				//geom.transform = 0.9f*geom.initialTransform + 0.1f*(float)iter * glm::mat4(
				//	1.0, 0.0, 0.0, 0.0,
				//	0.0, 1.0, 0.0, vel * iter,
				//	0.0, 0.0, 1.0, 0.0,
				//	0.0, 0.0, 0.0, 1.0) * geom.initialTransform;

				//geom.inverseTransform = glm::inverse(geom.transform);
				//geom.invTranspose = glm::inverseTranspose(geom.transform);

			
				t = sphereIntersectionTest(geom, pathSegment.ray, tmp_intersect, tmp_normal, outside);

			}
			// TODO: add more intersection tests here... triangle? metaball? CSG?

			// Compute the minimum t from the intersection tests to determine what
			// scene geometry object was hit first.
			if (t > 0.0f && t_min > t)
			{
				t_min = t;
				hit_geom_index = i;
				intersect_point = tmp_intersect;
				normal = tmp_normal;
			}
		}

		if (hit_geom_index == -1)
		{
			intersections[path_index].t = -1.0f;
		}
		else
		{
			//The ray hits something
			intersections[path_index].t = t_min;
			intersections[path_index].materialId = geoms[hit_geom_index].materialid;
			intersections[path_index].surfaceNormal = normal;
			intersections[path_index].intersectPoint = intersect_point;
		}
	}
}

// LOOK: "fake" shader demonstrating what you might do with the info in
// a ShadeableIntersection, as well as how to use thrust's random number
// generator. Observe that since the thrust random number generator basically
// adds "noise" to the iteration, the image should start off noisy and get
// cleaner as more iterations are computed.
//
// Note that this shader does NOT do a BSDF evaluation!
// Your shaders should handle that - this can allow techniques such as
// bump mapping.
__global__ void shadeFakeMaterial (
  int iter
  , int num_paths
	, ShadeableIntersection * shadeableIntersections
	, PathSegment * pathSegments
	, Material * materials
	)
{
  int idx = blockIdx.x * blockDim.x + threadIdx.x;
  if (idx < num_paths)
  {
	ShadeableIntersection intersection = shadeableIntersections[idx];
	if (intersection.t > 0.0f) { // if the intersection exists...
	  // Set up the RNG
	  // LOOK: this is how you use thrust's RNG! Please look at
	  // makeSeededRandomEngine as well.
	  thrust::default_random_engine rng = makeSeededRandomEngine(iter, idx, 0);
	  thrust::uniform_real_distribution<float> u01(0, 1);

	  Material material = materials[intersection.materialId];
	  glm::vec3 materialColor = material.color;

	  // If the material indicates that the object was a light, "light" the ray
	  if (material.emittance > 0.0f) {
		pathSegments[idx].color *= (materialColor * material.emittance);
	  }
	  // Otherwise, do some pseudo-lighting computation. This is actually more
	  // like what you would expect from shading in a rasterizer like OpenGL.
	  // TODO: replace this! you should be able to start with basically a one-liner
	  else {
		float lightTerm = glm::dot(intersection.surfaceNormal, glm::vec3(0.0f, 1.0f, 0.0f));
		pathSegments[idx].color *= (materialColor * lightTerm) * 0.3f + ((1.0f - intersection.t * 0.02f) * materialColor) * 0.7f;
		pathSegments[idx].color *= u01(rng); // apply some noise because why not
	  }
	// If there was no intersection, color the ray black.
	// Lots of renderers use 4 channel color, RGBA, where A = alpha, often
	// used for opacity, in which case they can indicate "no opacity".
	// This can be useful for post-processing and image compositing.
	} else {
	  pathSegments[idx].color = glm::vec3(0.0f);
	}
  }
}


__global__ void diffuseShader(
	int iter
	, int num_paths
	, ShadeableIntersection * shadeableIntersections
	, PathSegment * pathSegments
	, int* alive_paths
	, Material * materials, int depth
)
{
	int alive_idx = blockIdx.x * blockDim.x + threadIdx.x;
	if (alive_idx < num_paths)
	{
		int idx = alive_paths[alive_idx];
		if (idx < 0) return;

		ShadeableIntersection intersection = shadeableIntersections[idx];
		if (intersection.t > 0.0f) { // if the intersection exists...
		  // Set up the RNG
		  // LOOK: this is how you use thrust's RNG! Please look at
		  // makeSeededRandomEngine as well.
			thrust::default_random_engine rng = makeSeededRandomEngine(iter, idx, pathSegments[idx].remainingBounces);
			thrust::uniform_real_distribution<float> u01(0, 1);

			Material material = materials[intersection.materialId];
			glm::vec3 materialColor = material.color;
			glm::vec3 intersectionPoint = intersection.intersectPoint;

			// If the material indicates that the object was a light, "light" the ray
			if (material.emittance > 0.0f) {
				pathSegments[idx].color *= (materialColor * material.emittance);
				pathSegments[idx].remainingBounces = 0;
			}
			// Otherwise, do some pseudo-lighting computation. This is actually more
			// like what you would expect from shading in a rasterizer like OpenGL.
			else {
				if (pathSegments[idx].remainingBounces > 0) {
					scatterRay(pathSegments[idx], intersectionPoint, intersection.surfaceNormal, material, rng);
					pathSegments[idx].remainingBounces -= 1;
				}
			}
			// If there was no intersection, color the ray black.
			// Lots of renderers use 4 channel color, RGBA, where A = alpha, often
			// used for opacity, in which case they can indicate "no opacity".
			// This can be useful for post-processing and image compositing.
		}
		else {
			pathSegments[idx].color = glm::vec3(0.0f);
			pathSegments[idx].remainingBounces = 0;
		}

		if (pathSegments[idx].remainingBounces == 0) alive_paths[alive_idx] = -1;
	}
}

// Add the current iteration's output to the overall image
__global__ void finalGather(int nPaths, glm::vec3 * image, PathSegment * iterationPaths)
{
	int index = (blockIdx.x * blockDim.x) + threadIdx.x;

	if (index < nPaths)
	{
		PathSegment iterationPath = iterationPaths[index];
		image[iterationPath.pixelIndex] += iterationPath.color;
	}
}

/**
 * Wrapper for the __global__ call that sets up the kernel calls and does a ton
 * of memory management
 */
void pathtrace(uchar4 *pbo, int frame, int iter) {
	const int traceDepth = hst_scene->state.traceDepth;
	Camera &cam = hst_scene->state.camera;
	const int pixelcount = cam.resolution.x * cam.resolution.y;

	// 2D block for generating ray from camera
	const dim3 blockSize2d(8, 8);
	const dim3 blocksPerGrid2d(
			(cam.resolution.x + blockSize2d.x - 1) / blockSize2d.x,
			(cam.resolution.y + blockSize2d.y - 1) / blockSize2d.y);

	// 1D block for path tracing
	const int blockSize1d = 128;

	///////////////////////////////////////////////////////////////////////////

	// Recap:
	// * Initialize array of path rays (using rays that come out of the camera)
	//   * You can pass the Camera object to that kernel.
	//   * Each path ray must carry at minimum a (ray, color) pair,
	//   * where color starts as the multiplicative identity, white = (1, 1, 1).
	//   * This has already been done for you.
	// * For each depth:
	//   * Compute an intersection in the scene for each path ray.
	//     A very naive version of this has been implemented for you, but feel
	//     free to add more primitives and/or a better algorithm.
	//     Currently, intersection distance is recorded as a parametric distance,
	//     t, or a "distance along the ray." t = -1.0 indicates no intersection.
	//     * Color is attenuated (multiplied) by reflections off of any object
	//   * TODO: Stream compact away all of the terminated paths.
	//     You may use either your implementation or `thrust::remove_if` or its
	//     cousins.
	//     * Note that you can't really use a 2D kernel launch any more - switch
	//       to 1D.
	//   * TODO: Shade the rays that intersected something or didn't bottom out.
	//     That is, color the ray by performing a color computation according
	//     to the shader, then generate a new ray to continue the ray path.
	//     We recommend just updating the ray's PathSegment in place.
	//     Note that this step may come before or after stream compaction,
	//     since some shaders you write may also cause a path to terminate.
	// * Finally, add this iteration's results to the image. This has been done
	//   for you.

	generateRayFromCamera <<<blocksPerGrid2d, blockSize2d >>>(cam, iter, traceDepth, dev_paths, dev_alive_paths);
	checkCUDAError("generate camera ray");

	int depth = 0;
	PathSegment* dev_path_end = dev_paths + pixelcount;
	int num_paths = dev_path_end - dev_paths;

	// --- PathSegment Tracing Stage ---
	// Shoot ray into scene, bounce between objects, push shading chunks

	// calculate first intersections
	dim3 numblocksPathSegmentTracing = (num_paths + blockSize1d - 1) / blockSize1d;
	if (iter == 1) {
		computeIntersections << <numblocksPathSegmentTracing, blockSize1d >> > (
			depth
			, num_paths
			, dev_paths
			, dev_alive_paths
			, dev_geoms
			, hst_scene->geoms.size()
			, dev_first_intersections
			,iter);
		checkCUDAError("trace one bounce");
		hipDeviceSynchronize();
	}

#if MOTION_BLUR
	kernMotionBlur << <1, hst_scene->geoms.size() >>> (hst_scene->geoms.size(), dev_geoms, iter);
#endif

	hipEvent_t start, stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);

	hipEventRecord(start);

	bool iterationComplete = false;
	while (!iterationComplete) {




		// clean shading chunks
		hipMemset(dev_intersections, 0, num_paths * sizeof(ShadeableIntersection));
		// tracing
		if (depth == 0) {
			hipMemcpy(dev_intersections, dev_first_intersections, num_paths * sizeof(ShadeableIntersection), hipMemcpyDeviceToDevice);
		}
		else {
			numblocksPathSegmentTracing = (num_paths + blockSize1d - 1) / blockSize1d;
			computeIntersections << <numblocksPathSegmentTracing, blockSize1d >> > (
				depth
				, num_paths
				, dev_paths
				, dev_alive_paths
				, dev_geoms
				, hst_scene->geoms.size()
				, dev_intersections
				,iter);
			checkCUDAError("trace one bounce");
			hipDeviceSynchronize();
		}

		depth++;


		// --- Shading Stage ---
		// Shade path segments based on intersections and generate new rays by
		// evaluating the BSDF.
		// Start off with just a big kernel that handles all the different
		// materials you have in the scenefile.

#if SORT_BY_MATERIAL
		thrust::sort_by_key(thrust::device, dev_intersections, dev_intersections + num_paths, dev_paths, compareIntersections());
#endif 

		diffuseShader<<<numblocksPathSegmentTracing, blockSize1d>>>(iter, 
			num_paths, dev_intersections, dev_paths, dev_alive_paths, dev_materials, depth);

#if STREAM_COMPACT
		num_paths = StreamCompaction::Efficient::compactShared(num_paths, dev_alive_paths);
#endif

#if STREAM_COMPACT_THRUST
		int* dev_alive_paths_end = thrust::partition(thrust::device, dev_alive_paths, dev_alive_paths + num_paths, isTerminated());
		num_paths = dev_alive_paths_end - dev_alive_paths;
#endif
		//if (iter == 1) {
		//	printf("%d\n", num_paths);
		//}

		iterationComplete = (num_paths == 0) || depth > traceDepth;
	}
	
	hipEventRecord(stop);
	hipEventSynchronize(stop);
	float milliseconds = 0;
	hipEventElapsedTime(&milliseconds, start, stop);
	
	//printf("%.4f\n", milliseconds);

	num_paths = pixelcount;

	// Assemble this iteration and apply it to the image
	dim3 numBlocksPixels = (pixelcount + blockSize1d - 1) / blockSize1d;
	finalGather << <numBlocksPixels, blockSize1d >> > (num_paths, dev_image, dev_paths);

	///////////////////////////////////////////////////////////////////////////

	// Send results to OpenGL buffer for rendering
	sendImageToPBO<<<blocksPerGrid2d, blockSize2d>>>(pbo, cam.resolution, iter, dev_image);

	// Retrieve image from GPU
	hipMemcpy(hst_scene->state.image.data(), dev_image,
			pixelcount * sizeof(glm::vec3), hipMemcpyDeviceToHost);

	checkCUDAError("pathtrace");
}
